/*
 * Copyright (c) 2023, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <benchmarks/fixture/rmm_pool_raii.hpp>
#include <nvbench/nvbench.cuh>

#include <cuspatial_test/geometry_generator.cuh>

#include <cuspatial/distance.cuh>
#include <cuspatial/geometry/vec_2d.hpp>

#include <rmm/cuda_stream_view.hpp>
#include <rmm/exec_policy.hpp>

using namespace cuspatial;
using namespace cuspatial::test;

template <typename T>
void pairwise_point_polygon_distance_benchmark(nvbench::state& state, nvbench::type_list<T>)
{
  // TODO: to be replaced by nvbench fixture once it's ready
  cuspatial::rmm_pool_raii rmm_pool;
  rmm::cuda_stream_view stream{rmm::cuda_stream_default};

  auto const num_pairs{static_cast<std::size_t>(state.get_int64("num_pairs"))};

  auto const num_polygons_per_multipolygon{
    static_cast<std::size_t>(state.get_int64("num_polygons_per_multipolygon"))};
  auto const num_holes_per_polygon{
    static_cast<std::size_t>(state.get_int64("num_holes_per_polygon"))};
  auto const num_edges_per_ring{static_cast<std::size_t>(state.get_int64("num_edges_per_ring"))};

  auto const num_points_per_multipoint{
    static_cast<std::size_t>(state.get_int64("num_points_per_multipoint"))};

  auto mpoly_generator_param = multipolygon_generator_parameter<T>{
    num_pairs, num_polygons_per_multipolygon, num_holes_per_polygon, num_edges_per_ring};

  auto mpoint_generator_param = multipoint_generator_parameter<T>{
    num_pairs, num_points_per_multipoint, vec_2d<T>{-1, -1}, vec_2d<T>{0, 0}};

  auto multipolygons = generate_multipolygon_array<T>(mpoly_generator_param, stream);
  auto multipoints   = generate_multipoint_array<T>(mpoint_generator_param, stream);

  auto distances = rmm::device_vector<T>(num_pairs);
  auto out_it    = distances.begin();

  auto mpoly_view  = multipolygons.range();
  auto mpoint_view = multipoints.range();

  state.add_element_count(num_pairs, "NumPairs");
  state.add_element_count(mpoly_generator_param.num_polygons(), "NumPolygons");
  state.add_element_count(mpoly_generator_param.num_rings(), "NumRings");
  state.add_element_count(mpoly_generator_param.num_coords(), "NumPoints (in mpoly)");
  state.add_element_count(static_cast<std::size_t>(mpoly_generator_param.num_coords() *
                                                   mpoly_generator_param.num_rings() *
                                                   mpoly_generator_param.num_polygons()),
                          "Multipolygon Complexity");
  state.add_element_count(mpoint_generator_param.num_points(), "NumPoints (in multipoints)");

  state.add_global_memory_reads<T>(
    mpoly_generator_param.num_coords() + mpoint_generator_param.num_points(),
    "CoordinatesReadSize");
  state.add_global_memory_reads<std::size_t>(
    (mpoly_generator_param.num_rings() + 1) + (mpoly_generator_param.num_polygons() + 1) +
      (mpoly_generator_param.num_multipolygons + 1) + (mpoint_generator_param.num_multipoints + 1),
    "OffsetsDataSize");

  state.add_global_memory_writes<T>(num_pairs);

  state.exec(nvbench::exec_tag::sync,
             [&mpoly_view, &mpoint_view, &out_it, &stream](nvbench::launch& launch) {
               pairwise_point_polygon_distance(mpoint_view, mpoly_view, out_it, stream);
             });
}

using floating_point_types = nvbench::type_list<float, double>;

// Benchmark scalability with simple multipolygon (3 sides, 0 hole, 1 poly)
NVBENCH_BENCH_TYPES(pairwise_point_polygon_distance_benchmark,
                    NVBENCH_TYPE_AXES(floating_point_types))
  .set_type_axes_names({"CoordsType"})
  .add_int64_axis("num_pairs", {1, 1'00, 10'000, 1'000'000, 100'000'000})
  .add_int64_axis("num_polygons_per_multipolygon", {1})
  .add_int64_axis("num_holes_per_polygon", {0})
  .add_int64_axis("num_edges_per_ring", {3})
  .add_int64_axis("num_points_per_multipoint", {1})
  .set_name("point_polygon_distance_benchmark_simple_polygon");

// Benchmark scalability with complex multipolygon (100 sides, 10 holes, 3 polys)
NVBENCH_BENCH_TYPES(pairwise_point_polygon_distance_benchmark,
                    NVBENCH_TYPE_AXES(floating_point_types))
  .set_type_axes_names({"CoordsType"})
  .add_int64_axis("num_pairs", {1'000, 10'000, 100'000, 1'000'000})
  .add_int64_axis("num_polygons_per_multipolygon", {2})
  .add_int64_axis("num_holes_per_polygon", {3})
  .add_int64_axis("num_edges_per_ring", {50})
  .add_int64_axis("num_points_per_multipoint", {1})
  .set_name("point_polygon_distance_benchmark_complex_polygon");

// // Benchmark impact of rings (100K pairs, 1 polygon, 3 sides)
NVBENCH_BENCH_TYPES(pairwise_point_polygon_distance_benchmark,
                    NVBENCH_TYPE_AXES(floating_point_types))
  .set_type_axes_names({"CoordsType"})
  .add_int64_axis("num_pairs", {10'000})
  .add_int64_axis("num_polygons_per_multipolygon", {1})
  .add_int64_axis("num_holes_per_polygon", {0, 10, 100, 1000})
  .add_int64_axis("num_edges_per_ring", {3})
  .add_int64_axis("num_points_per_multipoint", {1})
  .set_name("point_polygon_distance_benchmark_ring_numbers");

// Benchmark impact of rings (1M pairs, 1 polygon, 0 holes, 3 sides)
NVBENCH_BENCH_TYPES(pairwise_point_polygon_distance_benchmark,
                    NVBENCH_TYPE_AXES(floating_point_types))
  .set_type_axes_names({"CoordsType"})
  .add_int64_axis("num_pairs", {100})
  .add_int64_axis("num_polygons_per_multipolygon", {1})
  .add_int64_axis("num_holes_per_polygon", {0})
  .add_int64_axis("num_edges_per_ring", {3})
  .add_int64_axis("num_points_per_multipoint", {50, 5'00, 5'000, 50'000, 500'000})
  .set_name("point_polygon_distance_benchmark_points_in_multipoint");
