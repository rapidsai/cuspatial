#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2022, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <benchmarks/fixture/rmm_pool_raii.hpp>
#include <nvbench/nvbench.cuh>

#include <cuspatial/detail/iterator.hpp>
#include <cuspatial/experimental/iterator_factory.cuh>
#include <cuspatial/experimental/linestring_distance.cuh>
#include <cuspatial/experimental/ranges/multilinestring_range.cuh>
#include <cuspatial/vec_2d.hpp>

#include <rmm/device_vector.hpp>
#include <rmm/exec_policy.hpp>

#include <thrust/execution_policy.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/scan.h>

#include <memory>

using namespace cuspatial;

/**
 * @brief Helper to generate linestrings used for benchmarks.
 *
 * The generator adopts a walking algorithm. The ith point is computed by
 * walking (cos(i) * segment_length, sin(i) * segment_length) from the `i-1`
 * point. The initial point of the linestring is at `(init_xy, init_xy)`.
 * Since equidistance sampling on a sinusoid will result in random values,
 * the shape of the linestring is random.
 *
 * The number of line segments per linestring is constrolled by
 * `num_segment_per_string`.
 *
 * Since the outreach upper bound of the linestring group is
 * `(init_xy + num_strings * num_segments_per_string * segment_length)`,
 * user may control the locality of the linestring group via these four
 * arguments. It's important to control the locality between pairs of
 * the linestrings. Linestrings pair that do not intersect will take
 * the longest compute path in the kernel and will benchmark the worst
 * case performance of the API.
 *
 * @tparam T The floating point type for the coordinates
 * @param num_strings Total number of linestrings
 * @param num_segments_per_string Number of line segments per linestring
 * @param segment_length Length of each segment, or stride of walk
 * @param init_xy The initial coordinate to start the walk
 * @param stream The CUDA stream to use for device memory operations and kernel launches
 * @return A tuple of x and y coordinates of points and offsets to which the first point
 * of each linestring starts.
 *
 */
template <typename T>
std::tuple<rmm::device_vector<vec_2d<T>>, rmm::device_vector<int32_t>> generate_linestring(
  int32_t num_strings, int32_t num_segments_per_string, T segment_length, vec_2d<T> init_xy)
{
  int32_t num_points = num_strings * (num_segments_per_string + 1);

  auto offset_iter = detail::make_counting_transform_iterator(
    0, [num_segments_per_string](auto i) { return i * num_segments_per_string; });
  auto rads_iter = detail::make_counting_transform_iterator(0, [](auto i) {
    return vec_2d<T>{cos(static_cast<T>(i)), sin(static_cast<T>(i))};
  });

  std::vector<int32_t> offsets(offset_iter, offset_iter + num_strings);
  std::vector<vec_2d<T>> rads(rads_iter, rads_iter + num_points);
  std::vector<vec_2d<T>> points(num_points);

  auto random_walk_func = [segment_length](vec_2d<T> const& prev, vec_2d<T> const& rad) {
    return prev + segment_length * rad;
  };

  thrust::exclusive_scan(
    thrust::host, points.begin(), points.end(), points.begin(), init_xy, random_walk_func);

  // Implicitly constructing a device vector from host vector.
  return std::tuple(std::move(points), std::move(offsets));
}

template <typename T>
void pairwise_linestring_distance_benchmark(nvbench::state& state, nvbench::type_list<T>)
{
  // TODO: to be replaced by nvbench fixture once it's ready
  cuspatial::rmm_pool_raii rmm_pool;

  auto const num_string_pairs{state.get_int64("NumStrings")};
  auto const num_segments_per_string{state.get_int64("NumSegmentsPerString")};

  auto [ls1, ls1_offset] =
    generate_linestring<T>(num_string_pairs, num_segments_per_string, 1, {0, 0});
  auto [ls2, ls2_offset] =
    generate_linestring<T>(num_string_pairs, num_segments_per_string, 1, {100, 100});

  auto distances = rmm::device_vector<T>(ls1.size());
  auto out_it    = distances.begin();

  auto multilinestrings1  = make_multilinestring_range(1,
                                                      thrust::make_counting_iterator(0),
                                                      num_string_pairs,
                                                      ls1_offset.begin(),
                                                      ls1.size(),
                                                      ls1.begin());
  auto multilinestrings2  = make_multilinestring_range(1,
                                                      thrust::make_counting_iterator(0),
                                                      num_string_pairs,
                                                      ls2_offset.begin(),
                                                      ls2.size(),
                                                      ls2.begin());
  auto const total_points = ls1.size() + ls2.size();

  state.add_element_count(num_string_pairs, "LineStringPairs");
  state.add_element_count(total_points, "NumPoints");
  state.add_global_memory_reads<T>(total_points * 2, "CoordinatesDataSize");
  state.add_global_memory_reads<int32_t>(num_string_pairs * 2, "OffsetsDataSize");
  state.add_global_memory_writes<T>(num_string_pairs);

  state.exec(nvbench::exec_tag::sync,
             [&multilinestrings1, &multilinestrings2, &out_it](nvbench::launch& launch) {
               pairwise_linestring_distance(multilinestrings1, multilinestrings2, out_it);
             });
}

using floating_point_types = nvbench::type_list<float, double>;
NVBENCH_BENCH_TYPES(pairwise_linestring_distance_benchmark, NVBENCH_TYPE_AXES(floating_point_types))
  .set_type_axes_names({"CoordsType"})
  .add_int64_axis("NumStrings", {1'000, 10'000, 100'000})
  .add_int64_axis("NumSegmentsPerString", {10, 100, 1'000});
