#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2022, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <benchmarks/fixture/rmm_pool_raii.hpp>
#include <nvbench/nvbench.cuh>

#include <cuspatial/detail/iterator.hpp>
#include <cuspatial/experimental/linestring_distance.cuh>
#include <cuspatial/experimental/type_utils.hpp>

#include <rmm/cuda_stream_view.hpp>
#include <rmm/device_vector.hpp>
#include <rmm/exec_policy.hpp>

#include <thrust/iterator/counting_iterator.h>

#include <memory>

namespace cuspatial {

/**
 * @brief Helper to generate linestrings used for benchmarks.
 *
 * The generator adopts a walking algorithm. The ith point is computed by
 * walking (cos(i) * segment_length, sin(i) * segment_length) from the `i-1`
 * point. The initial point of the linestring is at `(init_xy, init_xy)`.
 * Since equidistance sampling on a sinusoid will result in random values,
 * the shape of the linestring is random.
 *
 * The number of line segments per linestring is constrolled by
 * `num_segment_per_string`.
 *
 * Since the outreach upper bound of the linestring group is
 * `(init_xy + num_strings * num_segments_per_string * segment_length)`,
 * user may control the locality of the linestring group via these four
 * arguments. It's important to control the locality between pairs of
 * the linestrings. Linestrings pair that do not intersect will take
 * the longest compute path in the kernel and will benchmark the worst
 * case performance of the API.
 *
 * @tparam T The floating point type for the coordinates
 * @param num_strings Total number of linestrings
 * @param num_segments_per_string Number of line segments per linestring
 * @param segment_length Length of each segment, or stride of walk
 * @param init_xy The initial coordinate to start the walk
 * @param stream The CUDA stream to use for device memory operations and kernel launches
 * @return A tuple of x and y coordinates of points and offsets to which the first point
 * of each linestring starts.
 *
 */
template <typename T>
std::tuple<rmm::device_vector<T>, rmm::device_vector<T>, rmm::device_vector<int32_t>>
generate_linestring(int32_t num_strings,
                    int32_t num_segments_per_string,
                    T segment_length,
                    T init_xy,
                    rmm::cuda_stream_view stream)
{
  int32_t num_points = num_strings * (num_segments_per_string + 1);

  auto offset_iter = detail::make_counting_transform_iterator(
    0, [num_segments_per_string] __device__(auto i) { return i * num_segments_per_string; });
  auto points_x_iter =
    detail::make_counting_transform_iterator(0, [] __device__(auto i) { return cos(i); });
  auto points_y_iter =
    detail::make_counting_transform_iterator(0, [] __device__(auto i) { return sin(i); });

  rmm::device_vector<int32_t> offsets(offset_iter, offset_iter + num_strings);
  rmm::device_vector<T> points_x(points_x_iter, points_x_iter + num_points);
  rmm::device_vector<T> points_y(points_y_iter, points_y_iter + num_points);

  auto random_walk_func = [segment_length] __device__(T prev, T rad) {
    return prev + segment_length * rad;
  };
  thrust::exclusive_scan(rmm::exec_policy(stream),
                         points_x.begin(),
                         points_x.end(),
                         points_x.begin(),
                         init_xy,
                         random_walk_func);

  thrust::exclusive_scan(rmm::exec_policy(stream),
                         points_y.begin(),
                         points_y.end(),
                         points_y.begin(),
                         init_xy,
                         random_walk_func);

  return std::tuple(std::move(points_x), std::move(points_y), std::move(offsets));
}

template <typename T>
void pairwise_linestring_distance_benchmark(nvbench::state& state, nvbench::type_list<T>)
{
  // TODO: to be replaced by nvbench fixture once it's ready
  cuspatial::rmm_pool_raii rmm_pool;

  auto const num_string_pairs{state.get_int64("NumStrings")},
    num_segments_per_string{state.get_int64("NumSegmentsPerString")};
  auto stream = rmm::cuda_stream_default;

  auto [ls1_x, ls1_y, ls1_offset] =
    generate_linestring<T>(num_string_pairs, num_segments_per_string, 1, 0, stream);
  auto [ls2_x, ls2_y, ls2_offset] =
    generate_linestring<T>(num_string_pairs, num_segments_per_string, 1, 100, stream);

  auto ls1_offset_begin = ls1_offset.begin();
  auto ls2_offset_begin = ls2_offset.begin();
  auto ls1_points_begin = cuspatial::make_cartesian_2d_iterator(ls1_x.begin(), ls1_y.begin());
  auto ls2_points_begin = cuspatial::make_cartesian_2d_iterator(ls2_x.begin(), ls2_y.begin());
  auto distances        = rmm::device_vector<T>(ls1_x.size());
  auto out_it           = distances.begin();

  hipStreamSynchronize(stream.value());

  auto const total_points = ls1_x.size() + ls2_x.size();

  state.add_element_count(num_string_pairs, "LineStringPairs");
  state.add_element_count(total_points, "NumPoints");
  state.add_global_memory_reads<T>(total_points * 2, "CoordinatesDataSize");
  state.add_global_memory_reads<int32_t>(num_string_pairs * 2, "OffsetsDataSize");
  state.add_global_memory_writes<T>(num_string_pairs);

  state.exec(nvbench::exec_tag::sync,
             [&ls1_offset_begin,
              &num_string_pairs,
              &ls1_points_begin,
              ls1_size = ls1_x.size(),
              &ls2_offset_begin,
              &ls2_points_begin,
              ls2_size = ls2_x.size(),
              &out_it](nvbench::launch& launch) {
               cuspatial::pairwise_linestring_distance(ls1_offset_begin,
                                                       ls1_offset_begin + num_string_pairs,
                                                       ls1_points_begin,
                                                       ls1_points_begin + ls1_size,
                                                       ls2_offset_begin,
                                                       ls2_points_begin,
                                                       ls2_points_begin + ls2_size,
                                                       out_it);
             });
}

using floating_point_types = nvbench::type_list<float, double>;
NVBENCH_BENCH_TYPES(pairwise_linestring_distance_benchmark, NVBENCH_TYPE_AXES(floating_point_types))
  .set_type_axes_names({"CoordsType"})
  .add_int64_axis("NumStrings", {1'000, 10'000, 100'000})
  .add_int64_axis("NumSegmentsPerString", {10, 100, 1'000});

}  // namespace cuspatial
