/*
 * Copyright (c) 2022, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "cuspatial/error.hpp"
#include <benchmarks/fixture/rmm_pool_raii.hpp>
#include <benchmarks/utility/random.cuh>

#include <cuspatial/detail/iterator.hpp>
#include <cuspatial/experimental/type_utils.hpp>
#include <cuspatial/spatial_window.hpp>
#include <cuspatial/vec_2d.hpp>

#include <rmm/device_uvector.hpp>
#include <rmm/exec_policy.hpp>

#include <nvbench/nvbench.cuh>

#include <thrust/iterator/counting_iterator.h>
#include <thrust/random/linear_congruential_engine.h>
#include <thrust/random/normal_distribution.h>
#include <thrust/random/uniform_int_distribution.h>

#include <memory>

using namespace cuspatial;

/**
 * @brief Helper to generate random points within a rectangular window
 *
 * @p begin and @p end must be iterators to device-accessible memory
 *
 * @tparam PointsIter The type of the iterator to the output points container
 * @tparam T The floating point type for the coordinates
 * @param begin The start of the range of points to generate
 * @param end The end of the range of points to generate
 *
 * @param window_min the lower left window corner
 * @param window_max the upper right window corner
 *
 */
template <class PointsIter, typename T>
void generate_points(PointsIter begin, PointsIter end, vec_2d<T> window_min, vec_2d<T> window_max)
{
  auto engine_x = deterministic_engine(std::distance(begin, end));
  auto engine_y = deterministic_engine(2 * std::distance(begin, end));

  auto x_dist = make_uniform_dist(window_min.x, window_max.x);
  auto y_dist = make_uniform_dist(window_min.y, window_max.y);

  auto x_gen = value_generator{window_min.x, window_max.x, engine_x, x_dist};
  auto y_gen = value_generator{window_min.y, window_max.y, engine_y, y_dist};

  thrust::tabulate(rmm::exec_policy(), begin, end, [x_gen, y_gen] __device__(size_t n) mutable {
    return vec_2d<T>{x_gen(n), y_gen(n)};
  });
}

template <typename T>
void points_in_spatial_window_benchmark(nvbench::state& state, nvbench::type_list<T>)
{
  // TODO: to be replaced by nvbench fixture once it's ready
  cuspatial::rmm_pool_raii rmm_pool;

  auto const num_points{state.get_int64("NumPoints")};

  auto window_min = vec_2d<T>{-100, -100};
  auto window_max = vec_2d<T>{100, 100};

  auto range_min = vec_2d<T>{-200, -200};
  auto range_max = vec_2d<T>{200, 200};

  auto d_x = rmm::device_uvector<T>(num_points, rmm::cuda_stream_default);
  auto d_y = rmm::device_uvector<T>(num_points, rmm::cuda_stream_default);

  auto d_points =
    cuspatial::make_zipped_vec_2d_output_iterator<cuspatial::vec_2d<T>>(d_x.begin(), d_y.begin());

  generate_points(d_points, d_points + num_points, range_min, range_max);

  auto xs = cudf::column(cudf::data_type{cudf::type_to_id<T>()}, num_points, d_x.release());
  auto ys = cudf::column(cudf::data_type{cudf::type_to_id<T>()}, num_points, d_y.release());

  CUSPATIAL_CUDA_TRY(hipDeviceSynchronize());

  state.add_element_count(num_points);

  state.exec(nvbench::exec_tag::sync, [&](nvbench::launch& launch) {
    auto points_in =
      points_in_spatial_window(window_min.x, window_max.x, window_min.y, window_max.y, xs, ys);
  });
}

using floating_point_types = nvbench::type_list<float, double>;
NVBENCH_BENCH_TYPES(points_in_spatial_window_benchmark, NVBENCH_TYPE_AXES(floating_point_types))
  .set_type_axes_names({"CoordsType"})
  .add_int64_axis("NumPoints", {100'000, 1'000'000, 10'000'000, 100'000'000});
