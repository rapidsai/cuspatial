#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <time.h>
#include <sys/time.h>
#include <vector>
#include <string>
#include <iostream>

#include <gtest/gtest.h>
#include <thrust/device_vector.h>
#include <utilities/legacy/error_utils.hpp>
#include <cuspatial/coordinate_transform.hpp>
#include <utility/utility.hpp>

#include <tests/utilities/legacy/column_wrapper.cuh>
#include <tests/utilities/legacy/cudf_test_utils.cuh>
#include <tests/utilities/legacy/cudf_test_fixtures.h>

struct CoordinateTransToy : public GdfTest 
{
};   
   
TEST_F(CoordinateTransToy, coordinatetranstest)
{

  gdf_scalar x0; 
  x0.data.fp64=-90.66511046;
  x0.dtype=GDF_FLOAT64;
  x0.is_valid=true;
  gdf_scalar y0;
  y0.data.fp64=42.49197018;
  y0.dtype=GDF_FLOAT64;
  y0.is_valid=true;
  
  double point_lon[]={-90.664973,-90.665393,-90.664976,-90.664537};
  double point_lat[]={42.493894,42.491520,42.491420,42.493823};
  
  int num_point=sizeof(point_lon)/sizeof(double);
  std::vector<double> point_lon_vec(point_lon,point_lon+num_point);
  std::vector<double> point_lat_vec(point_lat,point_lat+num_point);
  std::cout<<"using camera origin ("<<x0.data.fp64<<","<<y0.data.fp64<<")"<<std::endl;
  std::cout<<"points before query:"<<std::endl;
  std::cout<<"lon:"<<std::endl;
  std::copy(point_lon_vec.begin(),point_lon_vec.end(),std::ostream_iterator<double>(std::cout, " "));std::cout<<std::endl; 
  std::cout<<"lat:"<<std::endl;
  std::copy(point_lat_vec.begin(),point_lat_vec.end(),std::ostream_iterator<double>(std::cout, " "));std::cout<<std::endl; 
  
  cudf::test::column_wrapper<double> point_lon_wrapp{point_lon_vec};
  cudf::test::column_wrapper<double> point_lat_wrapp{point_lat_vec};
  
  std::cout<<"calling cuspatial::spatial_window_points"<<std::endl;
  std::pair<gdf_column,gdf_column> res_pair=cuspatial::lonlat_to_coord(
	x0,y0,*(point_lon_wrapp.get()),*(point_lat_wrapp.get()));
  	
  thrust::device_ptr<double> out_x_ptr= thrust::device_pointer_cast(static_cast<double*>(res_pair.first.data));
  thrust::device_ptr<double> out_y_ptr= thrust::device_pointer_cast(static_cast<double*>(res_pair.second.data));
  int num_print=res_pair.first.size;
  std::cout<<"x:"<<std::endl;
  thrust::copy(out_x_ptr,out_x_ptr+num_print,std::ostream_iterator<double>(std::cout, " "));std::cout<<std::endl; 
  std::cout<<"y:"<<std::endl;
  thrust::copy(out_y_ptr,out_y_ptr+num_print,std::ostream_iterator<double>(std::cout, " "));std::cout<<std::endl; 
  
}
