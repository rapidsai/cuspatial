/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <time.h>
#include <sys/time.h>
#include <vector>
#include <string>
#include <iostream>

#include <gtest/gtest.h>
#include <utilities/error_utils.hpp>
#include <cuspatial/hausdorff.hpp> 

#include <tests/utilities/legacy/column_wrapper.cuh>
#include <tests/utilities/legacy/cudf_test_utils.cuh>
#include <tests/utilities/legacy/cudf_test_fixtures.h>

struct HausdorffToy : public GdfTest 
{
};   
   
TEST_F(HausdorffToy, hausdorfftest)
{
  std::cout<<"in HausdorffToy"<<std::endl;
  cudf::test::column_wrapper<double> point_x_wrapp{0,-8,6};
  cudf::test::column_wrapper<double> point_y_wrapp{0,-8,6};
  cudf::test::column_wrapper<uint32_t> cnt_wrapp{1,2};
  gdf_column dist=cuspatial::directed_hausdorff_distance(
  	*(point_x_wrapp.get()), *(point_y_wrapp.get()),*(cnt_wrapp.get()));
  double *h_dist=new double[dist.size];
  CUDA_TRY(hipMemcpy(h_dist, dist.data, dist.size*sizeof(double), hipMemcpyDeviceToHost));
  CUDF_EXPECTS(h_dist[0]==0&&h_dist[3]==0,"distance between the same trajectoriy pair should be 0"); 
  std::cout<<"dist(0,1)="<<h_dist[1]<<std::endl;
  std::cout<<"dist(1,0)="<<h_dist[2]<<std::endl;
  delete[] h_dist;
}
