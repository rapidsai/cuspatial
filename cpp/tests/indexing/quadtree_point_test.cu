#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <time.h>
#include <sys/time.h>
#include <string>

#include <gtest/gtest.h>
#include <utilities/legacy/error_utils.hpp>
#include <tests/utilities/legacy/cudf_test_utils.cuh>
#include <tests/utilities/legacy/cudf_test_fixtures.h>
#include <cudf/column/column_view.hpp>
#include <cudf/table/table_view.hpp>
#include <cudf/table/table.hpp>
#include <cuspatial/quadtree.hpp>
#include <utility/helper_thrust.cuh>

struct QuadtreeOnPointIndexingTest : public GdfTest 
{    
};

TEST_F(QuadtreeOnPointIndexingTest, test_empty)
{
    const uint32_t num_levels=1;
    uint32_t min_size=1;
    double scale=1.0;
    double x1=0,x2=1,y1=0,y2=1;

    cudf::column x_col,y_col;
    cudf::mutable_column_view pnt_x_view=x_col.mutable_view();
    cudf::mutable_column_view pnt_y_view=y_col.mutable_view();
    
    EXPECT_THROW ( cuspatial::quadtree_on_points(pnt_x_view,pnt_y_view,
            x1,y1,x2,y2, scale,num_levels, min_size),cudf::logic_error );
}

TEST_F(QuadtreeOnPointIndexingTest, test_single)
{
    const uint32_t num_levels=1;
    uint32_t min_size=1;
    uint32_t point_len=1;
 
    double scale=1.0;
    double x1=0,x2=1,y1=0,y2=1;

    double xx[]={0.45};
    double yy[]={0.45};
    
    hipStream_t stream=0;
    rmm::mr::device_memory_resource* mr=rmm::mr::get_default_resource();
    
    //no need to delete db_pnt_x and db_pnt_y, as they are taken over by mutable_column_views x/y
    
    rmm::device_buffer *db_pnt_x=new rmm::device_buffer(point_len* sizeof(double),stream,mr);
    CUDF_EXPECTS(db_pnt_x!=nullptr, "Error allocating memory for x coordiantes of points");
    double *d_pnt_x=static_cast<double *>(db_pnt_x->data());

    rmm::device_buffer *db_pnt_y=new rmm::device_buffer(point_len* sizeof(double),stream,mr);
    CUDF_EXPECTS(db_pnt_y!=nullptr, "Error allocating memory for y coordiantes of points");
    double *d_pnt_y=static_cast<double *>(db_pnt_y->data());

    HANDLE_CUDA_ERROR( hipMemcpy( d_pnt_x, xx, point_len * sizeof(double), hipMemcpyHostToDevice ) );
    HANDLE_CUDA_ERROR( hipMemcpy( d_pnt_y, yy, point_len * sizeof(double), hipMemcpyHostToDevice ) );

    cudf::mutable_column_view x(cudf::data_type{cudf::FLOAT64},point_len,d_pnt_x);
    cudf::mutable_column_view y(cudf::data_type{cudf::FLOAT64},point_len,d_pnt_y);

    std::unique_ptr<cudf::experimental::table> quadtree= cuspatial::quadtree_on_points(x,y,x1,y1,x2,y2, scale,num_levels, min_size);
    cudf::table_view quad_view=quadtree->view();
    CUDF_EXPECTS(quad_view.num_columns()==5,"a quadtree table must have 5 columns (key,lev,sign,length,fpos)");
    uint32_t num_quad=quad_view.num_rows();
    std::cout<<"num_quad="<<num_quad<<std::endl;
    CUDF_EXPECTS(num_quad==1,"the resulting quadtree must have a single quadrant");

    const uint32_t *d_key=quad_view.column(0).data<uint32_t>();
    const uint8_t  *d_lev=quad_view.column(1).data<uint8_t>();
    const bool *d_sign=quad_view.column(2).data<bool>();
    const uint32_t *d_len=quad_view.column(3).data<uint32_t>();
    const uint32_t *d_fpos=quad_view.column(4).data<uint32_t>();
    
    uint32_t *h_key=new uint32_t[num_quad];
    uint8_t  *h_lev=new uint8_t[num_quad];
    bool     *h_sign=new bool[num_quad];
    uint32_t *h_len=new uint32_t[num_quad];
    uint32_t *h_fpos=new uint32_t[num_quad];
    assert(h_key!=nullptr && h_lev!=nullptr && h_sign!=nullptr && h_len!=nullptr && h_fpos!=nullptr);
 
    EXPECT_EQ(hipMemcpy(h_key,d_key,num_quad*sizeof(uint32_t),hipMemcpyDeviceToHost),hipSuccess);
    EXPECT_EQ(hipMemcpy(h_lev,d_lev,num_quad*sizeof(uint8_t),hipMemcpyDeviceToHost),hipSuccess);
    EXPECT_EQ(hipMemcpy(h_sign,d_sign,num_quad*sizeof(bool),hipMemcpyDeviceToHost),hipSuccess);
    EXPECT_EQ(hipMemcpy(h_len,d_len,num_quad*sizeof(uint32_t),hipMemcpyDeviceToHost),hipSuccess);
    EXPECT_EQ(hipMemcpy(h_fpos,d_fpos,num_quad*sizeof(uint32_t),hipMemcpyDeviceToHost),hipSuccess);
    
    //the single top level quadtree node is expected to have a value of (0,0,0,1,0)
    EXPECT_EQ(h_key[0],(uint32_t)0);
    EXPECT_EQ(h_lev[0],(uint32_t)0);
    EXPECT_EQ(h_sign[0],(uint32_t)0);
    EXPECT_EQ(h_len[0],(uint32_t)1);
    EXPECT_EQ(h_fpos[0],(uint32_t)0);
    
    delete [] h_key; h_key=nullptr;
    delete[] h_lev;  h_lev=nullptr;
    delete[] h_sign; h_sign=nullptr;
    delete[] h_len; h_len=nullptr;
    delete[] h_fpos; h_fpos=nullptr;
}

TEST_F(QuadtreeOnPointIndexingTest, test_two)
{
    const uint32_t num_levels=1;
    uint32_t min_size=1;
    uint32_t point_len=2;
 
    double scale=1.0;
    double x1=0,x2=2,y1=0,y2=2;

    double xx[]={0.45,1.45};
    double yy[]={0.45,1.45};
    
    hipStream_t stream=0;
    rmm::mr::device_memory_resource* mr=rmm::mr::get_default_resource();
    
    //no need to delete db_pnt_x and db_pnt_y, as they are taken over by mutable_column_views x/y
    
    rmm::device_buffer *db_pnt_x=new rmm::device_buffer(point_len* sizeof(double),stream,mr);
    CUDF_EXPECTS(db_pnt_x!=nullptr, "Error allocating memory for x coordiantes of points");
    double *d_pnt_x=static_cast<double *>(db_pnt_x->data());

    rmm::device_buffer *db_pnt_y=new rmm::device_buffer(point_len* sizeof(double),stream,mr);
    CUDF_EXPECTS(db_pnt_y!=nullptr, "Error allocating memory for y coordiantes of points");
    double *d_pnt_y=static_cast<double *>(db_pnt_y->data());

    HANDLE_CUDA_ERROR( hipMemcpy( d_pnt_x, xx, point_len * sizeof(double), hipMemcpyHostToDevice ) );
    HANDLE_CUDA_ERROR( hipMemcpy( d_pnt_y, yy, point_len * sizeof(double), hipMemcpyHostToDevice ) );

    cudf::mutable_column_view x(cudf::data_type{cudf::FLOAT64},point_len,d_pnt_x);
    cudf::mutable_column_view y(cudf::data_type{cudf::FLOAT64},point_len,d_pnt_y);

    std::unique_ptr<cudf::experimental::table> quadtree= cuspatial::quadtree_on_points(x,y,x1,y1,x2,y2, scale,num_levels, min_size);
    cudf::table_view quad_view=quadtree->view();
    CUDF_EXPECTS(quad_view.num_columns()==5,"a quadtree table must have 5 columns (key,lev,sign,length,fpos)");
    uint32_t num_quad=quad_view.num_rows();
    std::cout<<"num_quad="<<num_quad<<std::endl;
    CUDF_EXPECTS(num_quad==2,"the resulting quadtree must have 2 quadrants");

    const uint32_t *d_key=quad_view.column(0).data<uint32_t>();
    const uint8_t  *d_lev=quad_view.column(1).data<uint8_t>();
    const bool *d_sign=quad_view.column(2).data<bool>();
    const uint32_t *d_len=quad_view.column(3).data<uint32_t>();
    const uint32_t *d_fpos=quad_view.column(4).data<uint32_t>();
    
    uint32_t *h_key=new uint32_t[num_quad];
    uint8_t  *h_lev=new uint8_t[num_quad];
    bool     *h_sign=new bool[num_quad];
    uint32_t *h_len=new uint32_t[num_quad];
    uint32_t *h_fpos=new uint32_t[num_quad];
    assert(h_key!=nullptr && h_lev!=nullptr && h_sign!=nullptr && h_len!=nullptr && h_fpos!=nullptr);

    EXPECT_EQ(hipMemcpy(h_key,d_key,num_quad*sizeof(uint32_t),hipMemcpyDeviceToHost),hipSuccess);
    EXPECT_EQ(hipMemcpy(h_lev,d_lev,num_quad*sizeof(uint8_t),hipMemcpyDeviceToHost),hipSuccess);
    EXPECT_EQ(hipMemcpy(h_sign,d_sign,num_quad*sizeof(bool),hipMemcpyDeviceToHost),hipSuccess);
    EXPECT_EQ(hipMemcpy(h_len,d_len,num_quad*sizeof(uint32_t),hipMemcpyDeviceToHost),hipSuccess);
    EXPECT_EQ(hipMemcpy(h_fpos,d_fpos,num_quad*sizeof(uint32_t),hipMemcpyDeviceToHost),hipSuccess);
    
    //the single top level quadtree node is expected to have a value of (0,0,0,1,0)
    EXPECT_EQ(h_key[0],(uint32_t)0);
    EXPECT_EQ(h_lev[0],(uint32_t)0);
    EXPECT_EQ(h_sign[0],(uint32_t)0);
    EXPECT_EQ(h_len[0],(uint32_t)1);
    EXPECT_EQ(h_fpos[0],(uint32_t)0);

    EXPECT_EQ(h_key[1],(uint32_t)3);
    EXPECT_EQ(h_lev[1],(uint32_t)0);
    EXPECT_EQ(h_sign[1],(uint32_t)0);
    EXPECT_EQ(h_len[1],(uint32_t)1);
    EXPECT_EQ(h_fpos[1],(uint32_t)1);
    
    delete [] h_key; h_key=nullptr;
    delete[] h_lev;  h_lev=nullptr;
    delete[] h_sign; h_sign=nullptr;
    delete[] h_len; h_len=nullptr;
    delete[] h_fpos; h_fpos=nullptr;
}


TEST_F(QuadtreeOnPointIndexingTest, test_small)
{
    const uint32_t num_levels=3;
    uint32_t min_size=12;
    uint32_t point_len=71;

    double scale=1.0;
    double x1=0,x2=8,y1=0,y2=8;

    double xx[]={1.9804558865545805, 0.1895259128530169, 1.2591725716781235, 0.8178039499335275, 0.48171647380517046, 1.3890664414691907, 0.2536015260915061, 3.1907684812039956, 3.028362149164369, 3.918090468102582, 3.710910700915217, 3.0706987088385853, 3.572744183805594, 3.7080407833612004, 3.70669993057843, 3.3588457228653024, 2.0697434332621234, 2.5322042870739683, 2.175448214220591, 2.113652420701984, 2.520755151373394, 2.9909779614491687, 2.4613232527836137, 4.975578758530645, 4.07037627210835, 4.300706849071861, 4.5584381091040616, 4.822583857757069, 4.849847745942472, 4.75489831780737, 4.529792124514895, 4.732546857961497, 3.7622247877537456, 3.2648444465931474, 3.01954722322135, 3.7164018490892348, 3.7002781846945347, 2.493975723955388, 2.1807636574967466, 2.566986568683904, 2.2006520196663066, 2.5104987015171574, 2.8222482218882474, 2.241538022180476, 2.3007438625108882, 6.0821276168848994, 6.291790729917634, 6.109985464455084, 6.101327777646798, 6.325158445513714, 6.6793884701899, 6.4274219368674315, 6.444584786789386, 7.897735998643542, 7.079453687660189, 7.430677191305505, 7.5085184104988, 7.886010001346151, 7.250745898479374, 7.769497359206111, 1.8703303641352362, 1.7015273093278767, 2.7456295127617385, 2.2065031771469, 3.86008672302403, 1.9143371250907073, 3.7176098065039747, 0.059011873032214, 3.1162712022943757, 2.4264509160270813, 3.154282922203257};
    assert(sizeof(xx)/sizeof(double)==point_len);
    double yy[71]={1.3472225743317712, 0.5431061133894604, 0.1448705855995005, 0.8138440641113271, 1.9022922214961997, 1.5177694304735412, 1.8762161698642947, 0.2621847215928189, 0.027638405909631958, 0.3338651960183463, 0.9937713340192049, 0.9376313558467103, 0.33184908855075124, 0.09804238103130436, 0.7485845679979923, 0.2346381514128677, 1.1809465376402173, 1.419555755682142, 1.2372448404986038, 1.2774712415624014, 1.902015274420646, 1.2420487904041893, 1.0484414482621331, 0.9606291981013242, 1.9486902798139454, 0.021365525588281198, 1.8996548860019926, 0.3234041700489503, 1.9531893897409585, 0.7800065259479418, 1.942673409259531, 0.5659923375279095, 2.8709552313924487, 2.693039435509084, 2.57810040095543, 2.4612194182614333, 2.3345952955903906, 3.3999020934055837, 3.2296461832828114, 3.6607732238530897, 3.7672478678985257, 3.0668114607133137, 3.8159308233351266, 3.8812819070357545, 3.6045900851589048, 2.5470532680258002, 2.983311357415729, 2.2235950639628523, 2.5239201807166616, 2.8765450351723674, 2.5605928243991434, 2.9754616970668213, 2.174562817047202, 3.380784914178574, 3.063690547962938, 3.380489849365283, 3.623862886287816, 3.538128217886674, 3.4154469467473447, 3.253257011908445, 4.209727933188015, 7.478882372510933, 7.474216636277054, 6.896038613284851, 7.513564222799629, 6.885401350515916, 6.194330707468438, 5.823535317960799, 6.789029097334483, 5.188939408363776, 5.788316610960881};
    assert(sizeof(yy)/sizeof(double)==point_len);

    hipStream_t stream=0;
    rmm::mr::device_memory_resource* mr=rmm::mr::get_default_resource();
    
    //no need to delete db_pnt_x and db_pnt_y, as they are taken over by mutable_column_views x/y
    
    rmm::device_buffer *db_pnt_x=new rmm::device_buffer(point_len* sizeof(double),stream,mr);
    CUDF_EXPECTS(db_pnt_x!=nullptr, "Error allocating memory for x coordiantes of points");
    double *d_pnt_x=static_cast<double *>(db_pnt_x->data());

    rmm::device_buffer *db_pnt_y=new rmm::device_buffer(point_len* sizeof(double),stream,mr);
    CUDF_EXPECTS(db_pnt_y!=nullptr, "Error allocating memory for y coordiantes of points");
    double *d_pnt_y=static_cast<double *>(db_pnt_y->data());

    HANDLE_CUDA_ERROR( hipMemcpy( d_pnt_x, xx, point_len * sizeof(double), hipMemcpyHostToDevice ) );
    HANDLE_CUDA_ERROR( hipMemcpy( d_pnt_y, yy, point_len * sizeof(double), hipMemcpyHostToDevice ) );

    cudf::mutable_column_view x(cudf::data_type{cudf::FLOAT64},point_len,d_pnt_x);
    cudf::mutable_column_view y(cudf::data_type{cudf::FLOAT64},point_len,d_pnt_y);

    std::unique_ptr<cudf::experimental::table> quadtree= cuspatial::quadtree_on_points(x,y,x1,y1,x2,y2, scale,num_levels, min_size);

    cudf::table_view quad_view=quadtree->view();
    std::cout<<"num cols="<<quad_view.num_columns()<<" num rows="<<quad_view.num_rows()<<std::endl;
    CUDF_EXPECTS(quad_view.num_columns()==5,"a quadtree table must have 5 columns (key,lev,sign,length,fpos)");
    
    const uint32_t *d_key=quad_view.column(0).data<uint32_t>();
    const uint8_t  *d_lev=quad_view.column(1).data<uint8_t>();
    const bool *d_sign=quad_view.column(2).data<bool>();
    const uint32_t *d_len=quad_view.column(3).data<uint32_t>();
    const uint32_t *d_fpos=quad_view.column(4).data<uint32_t>();

if(0)
{

    thrust::device_ptr<const uint32_t> d_key_ptr=thrust::device_pointer_cast(d_key);
    thrust::device_ptr<const uint8_t> d_lev_ptr=thrust::device_pointer_cast(d_lev);
    thrust::device_ptr<const bool> d_sign_ptr=thrust::device_pointer_cast(d_sign);
    thrust::device_ptr<const uint32_t> d_len_ptr=thrust::device_pointer_cast(d_len);
    thrust::device_ptr<const uint32_t> d_fpos_ptr=thrust::device_pointer_cast(d_fpos);

    printf("key\n");
    thrust::copy(d_key_ptr,d_key_ptr+quad_view.num_rows(),std::ostream_iterator<const uint32_t>(std::cout, " "));std::cout<<std::endl;

    printf("lev\n");
    //change from uint8_t to uint32_t in ostream_iterator to output numbers instead of special chars
    thrust::copy(d_lev_ptr,d_lev_ptr+quad_view.num_rows(),std::ostream_iterator<const uint32_t>(std::cout, " "));std::cout<<std::endl;

    printf("sign\n");
    thrust::copy(d_sign_ptr,d_sign_ptr+quad_view.num_rows(),std::ostream_iterator<const bool>(std::cout, " "));std::cout<<std::endl;

    printf("length\n");
    thrust::copy(d_len_ptr,d_len_ptr+quad_view.num_rows(),std::ostream_iterator<const uint32_t>(std::cout, " "));std::cout<<std::endl;

    printf("fpos\n");
    thrust::copy(d_fpos_ptr,d_fpos_ptr+quad_view.num_rows(),std::ostream_iterator<const uint32_t>(std::cout, " "));std::cout<<std::endl;
}
    
    uint32_t c_key[]={0, 1, 2, 0 ,1 ,3, 4, 7, 5, 6 ,13, 14, 28, 31};
    uint8_t  c_lev[]={0, 0, 0, 1, 1, 1, 1, 1, 2, 2, 2, 2, 2, 2};
    bool c_sign[]={1, 1, 0, 0, 1, 1, 0, 1, 0, 0, 0, 0, 0, 0};
    uint32_t c_len[]={3, 2, 11, 7, 2, 2, 9, 2, 9, 7, 5, 8, 8, 7};
    uint32_t c_fpos[]={3, 6, 60, 0, 8, 10, 36, 12, 7, 16, 23, 28, 45, 53};
    
    uint32_t n_key=sizeof(c_key)/sizeof(uint32_t);
    uint32_t n_lev=sizeof(c_lev)/sizeof(uint8_t);
    uint32_t n_sign=sizeof(c_sign)/sizeof(bool);
    uint32_t n_len=sizeof(c_len)/sizeof(uint32_t);
    uint32_t n_fpos=sizeof(c_fpos)/sizeof(uint32_t);    
    
    CUDF_EXPECTS(n_key==n_lev&& n_lev==n_sign && n_sign==n_len && n_len==n_fpos,"quadtree columns must have the same sizes");
    CUDF_EXPECTS(n_key==(uint32_t)(quad_view.num_rows()),"CPU and GPU results must agree on column sizes");

    uint32_t *h_key=new uint32_t[n_key];
    uint8_t  *h_lev=new uint8_t[n_key];
    bool     *h_sign=new bool[n_key];
    uint32_t *h_len=new uint32_t[n_key];
    uint32_t *h_fpos=new uint32_t[n_key];
    assert(h_key!=nullptr && h_lev!=nullptr && h_sign!=nullptr && h_len!=nullptr && h_fpos!=nullptr);

    EXPECT_EQ(hipMemcpy(h_key,d_key,n_key*sizeof(uint32_t),hipMemcpyDeviceToHost),hipSuccess);
    EXPECT_EQ(hipMemcpy(h_lev,d_lev,n_key*sizeof(uint8_t),hipMemcpyDeviceToHost),hipSuccess);
    EXPECT_EQ(hipMemcpy(h_sign,d_sign,n_key*sizeof(bool),hipMemcpyDeviceToHost),hipSuccess);
    EXPECT_EQ(hipMemcpy(h_len,d_len,n_key*sizeof(uint32_t),hipMemcpyDeviceToHost),hipSuccess);
    EXPECT_EQ(hipMemcpy(h_fpos,d_fpos,n_key*sizeof(uint32_t),hipMemcpyDeviceToHost),hipSuccess);
    
    for(uint32_t i=0;i<n_key;i++)
    {
        EXPECT_EQ(h_key[i],c_key[i]);
        EXPECT_EQ(h_lev[i],c_lev[i]);
        EXPECT_EQ(h_sign[i],c_sign[i]);
        EXPECT_EQ(h_len[i],c_len[i]);
        EXPECT_EQ(h_fpos[i],c_fpos[i]);
    }

    delete [] h_key; h_key=nullptr;
    delete[] h_lev;  h_lev=nullptr;
    delete[] h_sign; h_sign=nullptr;
    delete[] h_len; h_len=nullptr;
    delete[] h_fpos; h_fpos=nullptr;
    
}


