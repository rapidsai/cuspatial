/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <time.h>
#include <sys/time.h>
#include <string>

#include <gtest/gtest.h>
#include <utilities/legacy/error_utils.hpp>
#include <tests/utilities/legacy/cudf_test_utils.cuh>
#include <tests/utilities/legacy/cudf_test_fixtures.h>
#include <cudf/column/column_view.hpp>
#include <cudf/table/table_view.hpp>
#include <cudf/table/table.hpp>
#include <utility/z_order.cuh>
#include <utility/quadtree_thrust.cuh>
#include <cuspatial/quadtree.hpp>

struct QuadtreeOnPointIndexingTest : public GdfTest 
{
 
};

TEST_F(QuadtreeOnPointIndexingTest, test1)
{
    const int M=3;
    int MINSIZE=12;
    int point_len=71;
    
    double xx[]={1.9804558865545805, 0.1895259128530169, 1.2591725716781235, 0.8178039499335275, 0.48171647380517046, 1.3890664414691907, 0.2536015260915061, 3.1907684812039956, 3.028362149164369, 3.918090468102582, 3.710910700915217, 3.0706987088385853, 3.572744183805594, 3.7080407833612004, 3.70669993057843, 3.3588457228653024, 2.0697434332621234, 2.5322042870739683, 2.175448214220591, 2.113652420701984, 2.520755151373394, 2.9909779614491687, 2.4613232527836137, 4.975578758530645, 4.07037627210835, 4.300706849071861, 4.5584381091040616, 4.822583857757069, 4.849847745942472, 4.75489831780737, 4.529792124514895, 4.732546857961497, 3.7622247877537456, 3.2648444465931474, 3.01954722322135, 3.7164018490892348, 3.7002781846945347, 2.493975723955388, 2.1807636574967466, 2.566986568683904, 2.2006520196663066, 2.5104987015171574, 2.8222482218882474, 2.241538022180476, 2.3007438625108882, 6.0821276168848994, 6.291790729917634, 6.109985464455084, 6.101327777646798, 6.325158445513714, 6.6793884701899, 6.4274219368674315, 6.444584786789386, 7.897735998643542, 7.079453687660189, 7.430677191305505, 7.5085184104988, 7.886010001346151, 7.250745898479374, 7.769497359206111, 1.8703303641352362, 1.7015273093278767, 2.7456295127617385, 2.2065031771469, 3.86008672302403, 1.9143371250907073, 3.7176098065039747, 0.059011873032214, 3.1162712022943757, 2.4264509160270813, 3.154282922203257};
    assert(sizeof(xx)/sizeof(double)==point_len);
    double yy[71]={1.3472225743317712, 0.5431061133894604, 0.1448705855995005, 0.8138440641113271, 1.9022922214961997, 1.5177694304735412, 1.8762161698642947, 0.2621847215928189, 0.027638405909631958, 0.3338651960183463, 0.9937713340192049, 0.9376313558467103, 0.33184908855075124, 0.09804238103130436, 0.7485845679979923, 0.2346381514128677, 1.1809465376402173, 1.419555755682142, 1.2372448404986038, 1.2774712415624014, 1.902015274420646, 1.2420487904041893, 1.0484414482621331, 0.9606291981013242, 1.9486902798139454, 0.021365525588281198, 1.8996548860019926, 0.3234041700489503, 1.9531893897409585, 0.7800065259479418, 1.942673409259531, 0.5659923375279095, 2.8709552313924487, 2.693039435509084, 2.57810040095543, 2.4612194182614333, 2.3345952955903906, 3.3999020934055837, 3.2296461832828114, 3.6607732238530897, 3.7672478678985257, 3.0668114607133137, 3.8159308233351266, 3.8812819070357545, 3.6045900851589048, 2.5470532680258002, 2.983311357415729, 2.2235950639628523, 2.5239201807166616, 2.8765450351723674, 2.5605928243991434, 2.9754616970668213, 2.174562817047202, 3.380784914178574, 3.063690547962938, 3.380489849365283, 3.623862886287816, 3.538128217886674, 3.4154469467473447, 3.253257011908445, 4.209727933188015, 7.478882372510933, 7.474216636277054, 6.896038613284851, 7.513564222799629, 6.885401350515916, 6.194330707468438, 5.823535317960799, 6.789029097334483, 5.188939408363776, 5.788316610960881};
    assert(sizeof(yy)/sizeof(double)==point_len);

    //RMM_TRY( rmmInitialize(0));        
    
    double *d_p_x=NULL,*d_p_y=NULL;
    RMM_TRY( RMM_ALLOC( &d_p_x,point_len* sizeof(double), 0));
    assert(d_p_x!=NULL);
    RMM_TRY( RMM_ALLOC( &d_p_y,point_len* sizeof(double), 0));
    assert(d_p_y!=NULL);    
 
    HANDLE_CUDA_ERROR( hipMemcpy( d_p_x, xx, point_len * sizeof(double), hipMemcpyHostToDevice ) );    
    HANDLE_CUDA_ERROR( hipMemcpy( d_p_y, yy, point_len * sizeof(double), hipMemcpyHostToDevice ) );     
    
    double scale=1.0;
    SBBox bbox(thrust::make_tuple(0,0),thrust::make_tuple(8,8));    
   
    /*column_view::column_view(data_type type, size_type size, void const* data,
                            bitmask_type const* null_mask, size_type null_count,
                            size_type offset,
                         std::vector<column_view> const& children)*/
   
    cudf::column_view x(cudf::data_type{cudf::FLOAT64},point_len,d_p_x);
    cudf::column_view y(cudf::data_type{cudf::FLOAT64},point_len,d_p_y);
    std::unique_ptr<cudf::experimental::table> qidx= cuspatial::quadtree_on_points(x,y,bbox, scale,M, MINSIZE);
    std::cout<<qidx->view().num_columns()<<std::endl;
    //rmmFinalize();
}


