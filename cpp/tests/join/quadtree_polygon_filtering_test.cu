/*
 * Copyright (c) 2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cuspatial/error.hpp>
#include <cuspatial/point_quadtree.hpp>
#include <cuspatial/polygon_bounding_box.hpp>
#include <cuspatial/spatial_join.hpp>

#include <cudf/table/table.hpp>
#include <cudf/table/table_view.hpp>

#include <tests/utilities/base_fixture.hpp>
#include <tests/utilities/column_utilities.hpp>
#include <tests/utilities/column_wrapper.hpp>
#include <tests/utilities/table_utilities.hpp>
#include <tests/utilities/type_lists.hpp>

template <typename T>
struct QuadtreePolygonFilteringTest : public cudf::test::BaseFixture {
};

TYPED_TEST_CASE(QuadtreePolygonFilteringTest, cudf::test::FloatingPointTypes);

TYPED_TEST(QuadtreePolygonFilteringTest, test_errors)
{
  using T = TypeParam;
  using namespace cudf::test;
  // bad table
  cudf::table_view bad_quadtree{};
  // bad bboxes
  cudf::table_view bad_bboxes{};
  // empty quadtree
  cudf::table_view empty_quadtree{{
    fixed_width_column_wrapper<int32_t>({}),
    fixed_width_column_wrapper<int8_t>({}),
    fixed_width_column_wrapper<bool>({}),
    fixed_width_column_wrapper<int32_t>({}),
    fixed_width_column_wrapper<int32_t>({}),
  }};
  // empty bboxes
  cudf::table_view empty_bboxes{{fixed_width_column_wrapper<T>({}),
                                 fixed_width_column_wrapper<T>({}),
                                 fixed_width_column_wrapper<T>({}),
                                 fixed_width_column_wrapper<T>({})}};

  // Test throws on bad quadtree
  EXPECT_THROW(cuspatial::join_quadtree_and_bounding_boxes(
                 bad_quadtree, empty_bboxes, 0, 1, 0, 1, 1, 1, this->mr()),
               cuspatial::logic_error);

  // Test throws on bad bboxes
  EXPECT_THROW(cuspatial::join_quadtree_and_bounding_boxes(
                 empty_quadtree, bad_bboxes, 0, 1, 0, 1, 1, 1, this->mr()),
               cuspatial::logic_error);

  // Test throws on bad scale
  EXPECT_THROW(cuspatial::join_quadtree_and_bounding_boxes(
                 empty_quadtree, empty_bboxes, 0, 1, 0, 1, 0, 1, this->mr()),
               cuspatial::logic_error);

  // Test throws on bad max_depth <= 0
  EXPECT_THROW(cuspatial::join_quadtree_and_bounding_boxes(
                 empty_quadtree, empty_bboxes, 0, 1, 0, 1, 1, 0, this->mr()),
               cuspatial::logic_error);

  // Test throws on bad max_depth >= 16
  EXPECT_THROW(cuspatial::join_quadtree_and_bounding_boxes(
                 empty_quadtree, empty_bboxes, 0, 1, 0, 1, 1, 16, this->mr()),
               cuspatial::logic_error);

  // Test throws on reversed area of interest bbox coordinates
  EXPECT_THROW(cuspatial::join_quadtree_and_bounding_boxes(
                 empty_quadtree, empty_bboxes, 1, 0, 1, 0, 1, 1, this->mr()),
               cuspatial::logic_error);
}

TYPED_TEST(QuadtreePolygonFilteringTest, test_empty)
{
  using T = TypeParam;
  using namespace cudf::test;

  double const x_min{0.0};
  double const x_max{1.0};
  double const y_min{0.0};
  double const y_max{1.0};
  double const scale{1.0};
  uint32_t const max_depth{1};

  // empty quadtree
  cudf::table_view quadtree{{
    fixed_width_column_wrapper<int32_t>({}),
    fixed_width_column_wrapper<int8_t>({}),
    fixed_width_column_wrapper<bool>({}),
    fixed_width_column_wrapper<int32_t>({}),
    fixed_width_column_wrapper<int32_t>({}),
  }};
  // empty bboxes
  cudf::table_view bboxes{{fixed_width_column_wrapper<T>({}),
                           fixed_width_column_wrapper<T>({}),
                           fixed_width_column_wrapper<T>({}),
                           fixed_width_column_wrapper<T>({})}};

  auto polygon_quadrant_pairs = cuspatial::join_quadtree_and_bounding_boxes(
    quadtree, bboxes, x_min, x_max, y_min, y_max, scale, max_depth, this->mr());

  expect_tables_equal(cudf::table_view{{fixed_width_column_wrapper<uint32_t>({}),
                                        fixed_width_column_wrapper<uint32_t>({})}},
                      *polygon_quadrant_pairs);
}

TYPED_TEST(QuadtreePolygonFilteringTest, test_small)
{
  using T = TypeParam;
  using namespace cudf::test;

  double const x_min{0.0};
  double const x_max{8.0};
  double const y_min{0.0};
  double const y_max{8.0};
  double const scale{1.0};
  uint32_t const max_depth{3};
  uint32_t const min_size{12};

  fixed_width_column_wrapper<T> x(
    {1.9804558865545805,  0.1895259128530169, 1.2591725716781235, 0.8178039499335275,
     0.48171647380517046, 1.3890664414691907, 0.2536015260915061, 3.1907684812039956,
     3.028362149164369,   3.918090468102582,  3.710910700915217,  3.0706987088385853,
     3.572744183805594,   3.7080407833612004, 3.70669993057843,   3.3588457228653024,
     2.0697434332621234,  2.5322042870739683, 2.175448214220591,  2.113652420701984,
     2.520755151373394,   2.9909779614491687, 2.4613232527836137, 4.975578758530645,
     4.07037627210835,    4.300706849071861,  4.5584381091040616, 4.822583857757069,
     4.849847745942472,   4.75489831780737,   4.529792124514895,  4.732546857961497,
     3.7622247877537456,  3.2648444465931474, 3.01954722322135,   3.7164018490892348,
     3.7002781846945347,  2.493975723955388,  2.1807636574967466, 2.566986568683904,
     2.2006520196663066,  2.5104987015171574, 2.8222482218882474, 2.241538022180476,
     2.3007438625108882,  6.0821276168848994, 6.291790729917634,  6.109985464455084,
     6.101327777646798,   6.325158445513714,  6.6793884701899,    6.4274219368674315,
     6.444584786789386,   7.897735998643542,  7.079453687660189,  7.430677191305505,
     7.5085184104988,     7.886010001346151,  7.250745898479374,  7.769497359206111,
     1.8703303641352362,  1.7015273093278767, 2.7456295127617385, 2.2065031771469,
     3.86008672302403,    1.9143371250907073, 3.7176098065039747, 0.059011873032214,
     3.1162712022943757,  2.4264509160270813, 3.154282922203257});

  fixed_width_column_wrapper<T> y(
    {1.3472225743317712,   0.5431061133894604,   0.1448705855995005, 0.8138440641113271,
     1.9022922214961997,   1.5177694304735412,   1.8762161698642947, 0.2621847215928189,
     0.027638405909631958, 0.3338651960183463,   0.9937713340192049, 0.9376313558467103,
     0.33184908855075124,  0.09804238103130436,  0.7485845679979923, 0.2346381514128677,
     1.1809465376402173,   1.419555755682142,    1.2372448404986038, 1.2774712415624014,
     1.902015274420646,    1.2420487904041893,   1.0484414482621331, 0.9606291981013242,
     1.9486902798139454,   0.021365525588281198, 1.8996548860019926, 0.3234041700489503,
     1.9531893897409585,   0.7800065259479418,   1.942673409259531,  0.5659923375279095,
     2.8709552313924487,   2.693039435509084,    2.57810040095543,   2.4612194182614333,
     2.3345952955903906,   3.3999020934055837,   3.2296461832828114, 3.6607732238530897,
     3.7672478678985257,   3.0668114607133137,   3.8159308233351266, 3.8812819070357545,
     3.6045900851589048,   2.5470532680258002,   2.983311357415729,  2.2235950639628523,
     2.5239201807166616,   2.8765450351723674,   2.5605928243991434, 2.9754616970668213,
     2.174562817047202,    3.380784914178574,    3.063690547962938,  3.380489849365283,
     3.623862886287816,    3.538128217886674,    3.4154469467473447, 3.253257011908445,
     4.209727933188015,    7.478882372510933,    7.474216636277054,  6.896038613284851,
     7.513564222799629,    6.885401350515916,    6.194330707468438,  5.823535317960799,
     6.789029097334483,    5.188939408363776,    5.788316610960881});

  auto pair = cuspatial::quadtree_on_points(
    x, y, x_min, x_max, y_min, y_max, scale, max_depth, min_size, this->mr());

  auto &quadtree = std::get<1>(pair);

  fixed_width_column_wrapper<int32_t> poly_offsets({0, 1, 2, 3});
  fixed_width_column_wrapper<int32_t> ring_offsets({0, 3, 8, 12});
  fixed_width_column_wrapper<T> poly_x({// ring 1
                                        2.488450,
                                        1.333584,
                                        3.460720,
                                        // ring 2
                                        5.039823,
                                        5.561707,
                                        7.103516,
                                        7.190674,
                                        5.998939,
                                        // ring 3
                                        5.998939,
                                        5.573720,
                                        6.703534,
                                        5.998939,
                                        // ring 4
                                        2.088115,
                                        1.034892,
                                        2.415080,
                                        3.208660,
                                        2.088115});
  fixed_width_column_wrapper<T> poly_y({// ring 1
                                        5.856625,
                                        5.008840,
                                        4.586599,
                                        // ring 2
                                        4.229242,
                                        1.825073,
                                        1.503906,
                                        4.025879,
                                        5.653384,
                                        // ring 3
                                        1.235638,
                                        0.197808,
                                        0.086693,
                                        1.235638,
                                        // ring 4
                                        4.541529,
                                        3.530299,
                                        2.896937,
                                        3.745936,
                                        4.541529});

  auto polygon_bboxes =
    cuspatial::polygon_bounding_boxes(poly_offsets, ring_offsets, poly_x, poly_y, this->mr());

  auto polygon_quadrant_pairs = cuspatial::join_quadtree_and_bounding_boxes(
    *quadtree, *polygon_bboxes, x_min, x_max, y_min, y_max, scale, max_depth, this->mr());

  CUSPATIAL_EXPECTS(
    polygon_quadrant_pairs->num_columns() == 2,
    "a polygon-quadrant pair table must have 2 columns (polygon-index, quadrant-index)");

  expect_tables_equal(
    cudf::table_view{{fixed_width_column_wrapper<uint32_t>({0, 3, 1, 2, 1, 1, 3, 3}),
                      fixed_width_column_wrapper<uint32_t>({2, 2, 6, 6, 12, 13, 10, 11})}},
    *polygon_quadrant_pairs);
}
