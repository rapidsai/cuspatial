/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <time.h>
#include <sys/time.h>
#include <string>

#include <gtest/gtest.h>
#include <utilities/legacy/error_utils.hpp>
#include <tests/utilities/legacy/cudf_test_utils.cuh>
#include <tests/utilities/legacy/cudf_test_fixtures.h>
#include <cuspatial/shapefile_readers.hpp>
#include <utility/utility.hpp> 


struct ReadShapefilePolygonTest : public GdfTest 
{
    bool check_polygon(const cuspatial::polygons<double>&  h_polygon,const gdf_column& f_pos,
        const gdf_column& r_pos,const gdf_column& poly_x,const gdf_column& poly_y)
   {
        CUDF_EXPECTS(h_polygon.num_feature==(uint32_t)f_pos.size,"number of features/polygons mismatches expected");
        CUDF_EXPECTS(h_polygon.num_ring==(uint32_t)r_pos.size,"number of rings mismatches expected");
        CUDF_EXPECTS(h_polygon.num_vertex==(uint32_t)poly_x.size,"number of vertices mismatches expected");
        CUDF_EXPECTS(poly_x.size==poly_y.size,"numbers of vertices in x and y vectors mismatch");

        std::vector<uint32_t> h_f_pos(h_polygon.num_feature);
        EXPECT_EQ(hipMemcpy(h_f_pos.data(),f_pos.data,
                             h_polygon.num_feature* sizeof(uint32_t),
                             hipMemcpyDeviceToHost),
                  hipSuccess);
        for(size_t i = 0; i<h_polygon.num_feature;i++)
            EXPECT_EQ(h_polygon.feature_position[i],h_f_pos[i]);

        std::vector<uint32_t> h_r_pos(h_polygon.num_ring);
        EXPECT_EQ(hipMemcpy(h_r_pos.data(),r_pos.data,
                             h_polygon.num_ring* sizeof(uint32_t),
                             hipMemcpyDeviceToHost),
                  hipSuccess);
        for(size_t i = 0; i<h_polygon.num_ring;i++)
            EXPECT_EQ(h_polygon.ring_position[i],h_r_pos[i]);

        std::vector<double> h_x(h_polygon.num_vertex);
        EXPECT_EQ(hipMemcpy(h_x.data(),poly_x.data,
                             h_polygon.num_vertex* sizeof(double),
                             hipMemcpyDeviceToHost),
                  hipSuccess);
        for(size_t i = 0; i<h_polygon.num_vertex;i++)
           EXPECT_NEAR(h_polygon.x[i],h_x[i],1e-9);

        std::vector<double> h_y(h_polygon.num_vertex);
        EXPECT_EQ(hipMemcpy(h_y.data(),poly_y.data,
                             h_polygon.num_vertex* sizeof(double),
                             hipMemcpyDeviceToHost),
                  hipSuccess);
        for(size_t i = 0; i<h_polygon.num_vertex;i++)
            EXPECT_NEAR(h_polygon.y[i],h_y[i],1e-9);
        return true;
    }
};

#if  1 // disable until data files are checked in

TEST_F(ReadShapefilePolygonTest, testNonExist)
{
    const char* env_p = std::getenv("CUSPATIAL_HOME");
    CUDF_EXPECTS(env_p!=NULL,"CUSPATIAL_HOME environmental variable must be set");
    std::string shape_filename=std::string(env_p)+std::string("/test_fixtures/shapefiles/non_exist.shp"); 
    std::cout<<"Using shapefile "<<shape_filename<<std::endl;
    
    gdf_column f_pos,r_pos,poly_x,poly_y;
    EXPECT_THROW(cuspatial::read_polygon_shapefile(shape_filename.c_str(),&f_pos,&r_pos,&poly_x,&poly_y),cudf::logic_error);
}

TEST_F(ReadShapefilePolygonTest, testZero)
{

    const char* env_p = std::getenv("CUSPATIAL_HOME");
    CUDF_EXPECTS(env_p!=NULL,"CUSPATIAL_HOME environmental variable must be set");
    std::string shape_filename=std::string(env_p)+std::string("/test_fixtures/shapefiles/empty_poly.shp"); 
    std::cout<<"Using shapefile "<<shape_filename<<std::endl;

    gdf_column f_pos,r_pos,poly_x,poly_y;
    EXPECT_THROW(cuspatial::read_polygon_shapefile(shape_filename.c_str(),&f_pos,&r_pos,&poly_x,&poly_y),cudf::logic_error);
}

TEST_F(ReadShapefilePolygonTest, testOne)
{
    const char* env_p = std::getenv("CUSPATIAL_HOME");
    CUDF_EXPECTS(env_p!=NULL,"CUSPATIAL_HOME environmental variable must be set");
    std::string shape_filename=std::string(env_p)+std::string("/test_fixtures/shapefiles/one_poly.shp"); 
    std::cout<<"Using shapefile "<<shape_filename<<std::endl;
    
    cuspatial::polygons<double> h_polygon;
    h_polygon.num_group=1;
    h_polygon.num_feature=1;
    h_polygon.num_ring=1;
    h_polygon.num_vertex=5;
    h_polygon.feature_position=new uint32_t[h_polygon.num_feature]{1};
    h_polygon.ring_position=new uint32_t[h_polygon.num_ring]{5};
    h_polygon.x=new double[h_polygon.num_vertex]{-10,   5, 5, -10, -10};
    h_polygon.y=new double[h_polygon.num_vertex]{-10, -10, 5,   5, -10};

    gdf_column f_pos,r_pos,poly_x,poly_y;
    cuspatial::read_polygon_shapefile(shape_filename.c_str(),&f_pos,&r_pos,&poly_x,&poly_y);
    CUDF_EXPECTS(this->check_polygon(h_polygon,f_pos,r_pos,poly_x,poly_y),"polygon readout mismatches expected");
}

TEST_F(ReadShapefilePolygonTest, testTwo)
{
    const char* env_p = std::getenv("CUSPATIAL_HOME");
    CUDF_EXPECTS(env_p!=NULL,"CUSPATIAL_HOME environmental variable must be set");
    std::string shape_filename=std::string(env_p)+std::string("/test_fixtures/shapefiles/two_polys.shp"); 
    std::cout<<"Using shapefile "<<shape_filename<<std::endl;

    cuspatial::polygons<double> h_polygon;
    h_polygon.num_group=1;
    h_polygon.num_feature=2;
    h_polygon.num_ring=2;
    h_polygon.num_vertex=10;
    h_polygon.feature_position=new uint32_t[h_polygon.num_feature]{1,2};
    h_polygon.ring_position=new uint32_t[h_polygon.num_ring]{5,10};
    h_polygon.x=new double[h_polygon.num_vertex]{-10,   5, 5, -10, -10,0, 10, 10,  0, 0};
    h_polygon.y=new double[h_polygon.num_vertex]{-10, -10, 5,   5, -10,0,  0, 10, 10, 0};

    gdf_column f_pos,r_pos,poly_x,poly_y;
    cuspatial::read_polygon_shapefile(shape_filename.c_str(),&f_pos,&r_pos,&poly_x,&poly_y);

    CUDF_EXPECTS(this->check_polygon(h_polygon,f_pos,r_pos,poly_x,poly_y),"polygon readout mismatches expected");
}

#endif
