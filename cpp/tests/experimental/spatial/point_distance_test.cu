#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2022, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cuspatial_test/vector_equality.hpp>

#include <cuspatial_test/random.cuh>

#include <cuspatial/detail/iterator.hpp>
#include <cuspatial/error.hpp>
#include <cuspatial/experimental/iterator_factory.cuh>
#include <cuspatial/experimental/point_distance.cuh>
#include <cuspatial/experimental/ranges/multipoint_range.cuh>
#include <cuspatial/vec_2d.hpp>

#include <rmm/cuda_stream_view.hpp>
#include <rmm/device_uvector.hpp>
#include <rmm/device_vector.hpp>
#include <rmm/exec_policy.hpp>

#include <thrust/generate.h>
#include <thrust/host_vector.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/random/linear_congruential_engine.h>
#include <thrust/random/normal_distribution.h>
#include <thrust/transform.h>
#include <thrust/tuple.h>

#include <gmock/gmock.h>
#include <gtest/gtest.h>

#include <numeric>

namespace cuspatial {

/**
 * @brief Generate `num_points` points on device
 */
template <typename T>
struct PairwisePointDistanceTest : public ::testing::Test {
  rmm::device_vector<vec_2d<T>> generate_random_points(
    std::size_t num_points,
    std::size_t seed,
    rmm::cuda_stream_view stream = rmm::cuda_stream_default)
  {
    auto engine  = deterministic_engine(0);
    auto uniform = make_normal_dist<T>(0.0, 1.0);
    auto pgen    = point_generator(T{0.0}, T{1.0}, engine, uniform);
    rmm::device_vector<vec_2d<T>> points(num_points);
    auto counting_iter = thrust::make_counting_iterator(seed);
    thrust::transform(
      rmm::exec_policy(stream), counting_iter, counting_iter + num_points, points.begin(), pgen);
    return points;
  }

  /**
   * @brief Generate `num_multipoints` multipoint, returns offset and point vectors on device
   */
  std::pair<rmm::device_vector<std::size_t>, rmm::device_vector<vec_2d<T>>>
  generate_random_multipoints(std::size_t num_multipoints,
                              std::size_t max_points_per_multipoint,
                              std::size_t seed,
                              rmm::cuda_stream_view stream = rmm::cuda_stream_default)
  {
    std::vector<std::size_t> offset(num_multipoints + 1, 0);
    std::generate_n(offset.begin() + 1, num_multipoints, [max_points_per_multipoint]() {
      return std::rand() % max_points_per_multipoint;
    });
    std::inclusive_scan(offset.begin(), offset.end(), offset.begin());
    std::size_t num_points = offset.back();
    auto points            = generate_random_points(num_points, seed, stream);
    return {offset, points};
  }
};

/**
 * @brief Computes point distances on host
 *
 * @note Implicitly copies input vectors to host
 */
template <typename Cart2DVec>
auto compute_point_distance_host(Cart2DVec const& point1, Cart2DVec const& point2)
{
  using Cart2D = typename Cart2DVec::value_type;
  using T      = typename Cart2D::value_type;
  thrust::host_vector<Cart2D> h_point1(point1);
  thrust::host_vector<Cart2D> h_point2(point2);
  auto pair_iter =
    thrust::make_zip_iterator(thrust::make_tuple(h_point1.begin(), h_point2.begin()));
  auto result_iter = thrust::make_transform_iterator(pair_iter, [](auto p) {
    auto p0 = thrust::get<0>(p);
    auto p1 = thrust::get<1>(p);
    return std::sqrt(dot(p0 - p1, p0 - p1));
  });

  return thrust::host_vector<T>(result_iter, result_iter + point1.size());
}

/**
 * @brief Computes multipoint distances on host.
 *
 * @note Implicitly copies input vectors to host.
 * @note This function also tests the compatibility of `multipoint_range` on host.
 */
template <typename OffsetVec, typename Cart2DVec>
auto compute_multipoint_distance_host(OffsetVec const& lhs_offset,
                                      Cart2DVec const& lhs_points,
                                      OffsetVec const& rhs_offset,
                                      Cart2DVec const& rhs_points)
{
  using Cart2D    = typename Cart2DVec::value_type;
  using IndexType = typename OffsetVec::value_type;
  using T         = typename Cart2D::value_type;

  auto num_results = lhs_offset.size() - 1;
  thrust::host_vector<IndexType> h_offset1(lhs_offset);
  thrust::host_vector<Cart2D> h_point1(lhs_points);
  thrust::host_vector<IndexType> h_offset2(rhs_offset);
  thrust::host_vector<Cart2D> h_point2(rhs_points);

  auto h_multipoint_array1 =
    multipoint_range{h_offset1.begin(), h_offset1.end(), h_point1.begin(), h_point1.end()};
  auto h_multipoint_array2 =
    multipoint_range{h_offset2.begin(), h_offset2.end(), h_point2.begin(), h_point2.end()};

  std::vector<T> result(num_results, 0);

  std::transform(h_multipoint_array1.multipoint_begin(),
                 h_multipoint_array1.multipoint_end(),
                 h_multipoint_array2.multipoint_begin(),
                 result.begin(),
                 [](auto const& mp1, auto const& mp2) {
                   T min_distance_squared = std::numeric_limits<T>::max();
                   for (vec_2d<T> const& p1 : mp1)
                     for (vec_2d<T> const& p2 : mp2) {
                       T distance_squared   = dot((p1 - p2), (p1 - p2));
                       min_distance_squared = min(min_distance_squared, distance_squared);
                     }

                   return std::sqrt(min_distance_squared);
                 });
  return result;
}

using TestTypes = ::testing::Types<float, double>;

TYPED_TEST_CASE(PairwisePointDistanceTest, TestTypes);

TYPED_TEST(PairwisePointDistanceTest, Empty)
{
  using T         = TypeParam;
  using Cart2D    = vec_2d<T>;
  using Cart2DVec = std::vector<Cart2D>;

  rmm::device_vector<int32_t> multipoint_geom1(std::vector<int32_t>{0});
  rmm::device_vector<Cart2D> points1{};
  rmm::device_vector<int32_t> multipoint_geom2(std::vector<int32_t>{0});
  rmm::device_vector<Cart2D> points2{};

  rmm::device_vector<T> expected{};
  rmm::device_vector<T> got(points1.size());

  auto multipoint_1 = multipoint_range{
    multipoint_geom1.begin(), multipoint_geom1.end(), points1.begin(), points1.end()};
  auto multipoint_2 = multipoint_range{
    multipoint_geom2.begin(), multipoint_geom2.end(), points2.begin(), points2.end()};

  auto ret_it = pairwise_point_distance(multipoint_1, multipoint_2, got.begin());

  test::expect_vector_equivalent(expected, got);
  EXPECT_EQ(expected.size(), std::distance(got.begin(), ret_it));
}

TYPED_TEST(PairwisePointDistanceTest, OnePairSingleComponent)
{
  using T         = TypeParam;
  using Cart2D    = vec_2d<T>;
  using Cart2DVec = std::vector<Cart2D>;

  std::size_t constexpr num_pairs = 1;
  auto multipoint_geom1           = thrust::make_counting_iterator(0);
  rmm::device_vector<Cart2D> points1{Cart2DVec{{1.0, 1.0}}};
  auto multipoint_geom2 = thrust::make_counting_iterator(0);
  rmm::device_vector<Cart2D> points2{Cart2DVec{{0.0, 0.0}}};

  rmm::device_vector<T> expected{std::vector<T>{std::sqrt(T{2.0})}};
  rmm::device_vector<T> got(points1.size());

  auto multipoint_1 = multipoint_range{
    multipoint_geom1, multipoint_geom1 + num_pairs + 1, points1.begin(), points1.end()};
  auto multipoint_2 = multipoint_range{
    multipoint_geom2, multipoint_geom2 + num_pairs + 1, points2.begin(), points2.end()};

  auto ret_it = pairwise_point_distance(multipoint_1, multipoint_2, got.begin());

  test::expect_vector_equivalent(expected, got);
  EXPECT_EQ(expected.size(), std::distance(got.begin(), ret_it));
}

TYPED_TEST(PairwisePointDistanceTest, SingleComponentManyRandom)
{
  using T         = TypeParam;
  using Cart2D    = vec_2d<T>;
  using Cart2DVec = std::vector<Cart2D>;

  std::size_t constexpr num_pairs = 1000;

  auto multipoint_geom1 = thrust::make_counting_iterator(0);
  auto points1          = this->generate_random_points(num_pairs, 0);
  auto multipoint_geom2 = thrust::make_counting_iterator(0);
  auto points2          = this->generate_random_points(num_pairs, num_pairs);

  auto expected = compute_point_distance_host(points1, points2);
  rmm::device_vector<T> got(points1.size());

  auto multipoint_1 =
    make_multipoint_range(num_pairs, multipoint_geom1, points1.size(), points1.begin());
  auto multipoint_2 =
    make_multipoint_range(num_pairs, multipoint_geom2, points2.size(), points2.begin());

  auto ret_it = pairwise_point_distance(multipoint_1, multipoint_2, got.begin());
  thrust::host_vector<T> hgot(got);

  test::expect_vector_equivalent(hgot, expected);
  EXPECT_EQ(expected.size(), std::distance(got.begin(), ret_it));
}

TYPED_TEST(PairwisePointDistanceTest, SingleComponentCompareWithShapely)
{
  using T         = TypeParam;
  using Cart2D    = vec_2d<T>;
  using Cart2DVec = std::vector<Cart2D>;

  std::vector<T> x1{
    -12.309831056315302, -7.927059559371418,  -49.95705839647165,  -1.0512464476733485,
    -89.39777525663895,  -32.460148393873666, -20.64749623324501,  74.88373211296442,
    -3.566633537053898,  -91.4320392524529,   1.68283845329249,    30.90993923507801,
    2.5208716416609267,  -47.13990142514067,  -89.60387010381702,  15.799301259524867,
    -22.8887289692815,   81.6430657985936,    28.324072604115624,  -43.3201792789866,
    31.15072850958005,   -90.9256331022774,   -17.077973750390452, -88.54243712973691,
    -83.67679143413889,  -78.86701538797912,  60.11416346218348,   38.38679261335849,
    86.29202143733288,   90.51425714428673,   -72.13954336543273,  -29.909309579787713,
    -72.27943372189681,  49.182311914851205,  -84.50393600760954,  -94.33250533960667,
    -9.932568319346647,  36.99556837875937,   -24.20862704113279,  -50.442042319693705,
    -59.14098804172897,  30.673225738449172,  48.67403790478693,   -63.207315558126545,
    29.52859942242645,   26.173903500998197,  47.79243983907904,   -99.38850933058964,
    -83.31147453301942,  5.8413331217636255,  -47.87029604603307,  95.82254403897923,
    -55.52829900834991,  74.87973107553039,   -84.05457104705182,  -95.87736100367613,
    -6.480112617573386,  -78.09036923042659,  62.14707651291427,   -43.34499838125344,
    77.42752654240155,   12.530763429172254,  97.98997832862835,   -51.389571363066,
    59.66745813871337,   65.98475889051292,   30.40020778235388,   -49.595509308751595,
    9.930123176564942,   -19.283736893878867, -78.06236247946624,  63.68142858698178,
    79.46252260195803,   54.24426311960122,   30.458402886352822,  70.7820673095687,
    -15.306354680748024, 91.01665772140062,   -32.765892351019666, -72.46623073604916,
    58.863272100444334,  -41.35480445335994,  -61.06943341086172,  81.15104128608479,
    -77.69660768219927,  95.47462923834442,   -97.46155919360085,  -81.54704046899158,
    84.9228534190681,    -16.082575320922533, 52.509864091786355,  63.78396723518307,
    13.605239448412032,  -63.70301611378514,  -63.10763374202178,  -61.108649551804895,
    57.266357913172385,  -46.96569013769979,  -43.636365011489566, -29.306746287827558};
  std::vector<T> y1{
    -18.051875936208862, -72.61500308351708,  -23.919317289360777, 74.04449323147637,
    27.003656419402276,  5.131923603252009,   14.381495553187262,  -44.998590378882795,
    66.15308743061799,   31.82686362809011,   60.19621369406618,   36.02100660419922,
    -18.164297228344505, 23.06381468579426,   -34.39959102364766,  -80.65093614662105,
    -50.66614351982265,  30.696539385917852,  -62.06159838829518,  -55.67574678891346,
    2.2570284921564987,  49.260913129155036,  -69.70290379728544,  -14.168007892316037,
    87.743587998508,     -88.40683092249026,  -78.23312582934655,  18.950081576813904,
    -13.001178290210335, -88.72165572783072,  29.13236030074242,   0.9643364439866353,
    -58.14148269328302,  98.23977047259831,   87.65596263514071,   -68.42627074347195,
    -61.49539737381592,  95.22412232012014,   -71.3663413078797,   -87.93028627383005,
    -63.70741871892348,  1.83023166369769,    -44.184879390345245, -29.212266921068498,
    36.63070498793903,   90.55120945758097,   35.40957933073132,   -53.484664102448285,
    85.05271776288717,   80.18938384135001,   -21.313832146230382, -64.49346600820266,
    -72.18007667511924,  50.73463806168728,   7.319811593578507,   -56.54419097667299,
    -80.58912509276239,  6.9148441008914485,  -22.67913193215382,  75.95466324740005,
    69.60650343179027,   27.61785095385285,   -17.798865714702472, -78.36406107867042,
    6.59132839160077,    64.32222103875719,   55.24725933014744,   -53.49018275541756,
    -71.57964472201111,  -9.671216230543001,  -29.999576747551593, -54.15829040618368,
    29.253521698849028,  57.83102910157538,   76.77316185511351,   -54.755703196886174,
    58.71741301597688,   -89.00648352439477,  -62.572264098389354, 55.118081589496626,
    -72.80219811987917,  56.12298345685937,   -9.073644079329679,  87.3857422229443,
    16.65929971566098,   -91.77505633845232,  -99.4775802747735,   6.657482305470497,
    19.82536215719839,   -22.918311016363912, 30.170484267010387,  83.6666865961853,
    -91.70882742463144,  78.70726479431833,   86.04667133973348,   -83.58460594914955,
    84.27888264842167,   6.374228239422575,   62.58260784755962,   -87.64421055779096};

  std::vector<T> x2{
    -69.89840831561355,   78.8460456024616,    39.85341596822734,   -24.391223974913235,
    13.303395979112231,   -12.113621295331923, 65.76955972393912,   32.88000233887396,
    75.15679902070009,    70.42968479275325,   -70.48373074669782,  -67.41906709787041,
    24.0317463752441,     15.6825064869063,    22.786346338534358,  -20.418849974209763,
    34.82105661248487,    38.24867453316148,   -25.835471974453984, -99.8181927392706,
    89.84785718125181,    92.62449528299297,   -15.692938009982782, 42.32594734729251,
    -60.14762773795758,   74.97034158301297,   49.83345296858048,   -8.799811548418369,
    35.12809596314472,    93.18344995215058,   -94.67426883200939,  52.863378156989384,
    80.55592370229223,    -9.708518300250157,  58.19902373613033,   94.71328595396487,
    -41.956496383879006,  -99.23900353260521,  -96.8820547539014,   -61.540850851797046,
    10.60351610840815,    -86.06663137958869,  -19.76183018904282,  -52.98140516951296,
    -60.77170988936312,   -67.64765557651907,  45.61193823583003,   56.92515530750559,
    -33.35973933318071,   -51.94527984432248,  -14.582250347543601, -96.83073470861669,
    -47.25698648583708,   48.904375839188006,  14.554162511314495,  38.237373081363344,
    -32.7325518620032,    57.537241341535015,  -70.50257367880944,  -83.11435173667108,
    1.3843207970826832,   -61.35647094743536,  43.70708320820875,   -81.93488230360825,
    -53.098660448910465,  70.16656087048054,   0.7197864636628637,  92.59459361315123,
    -77.37226816319428,   -32.66885376463454,  34.32370196646004,   71.72963476414482,
    1.5234779242439433,   3.0626652169396085,  -1.600973288116736,  -1.875116500268692,
    24.115900341387686,   -6.818007491235834,  -37.57206985691543,  46.48919986671669,
    99.81587509298548,    26.961573147884856,  -57.411420876126954, -78.90146907605978,
    37.2322492476274,     67.99231943510561,   64.95985406157519,   -21.195261701977287,
    78.89518238318205,    -95.50952525706322,  76.75637507677297,   -63.30961059551444,
    88.07294705390709,    12.963110252847354,  -59.3400766172247,   18.016669829562915,
    0.024732013514316975, -47.68463698812436,  -16.12846919710843,  57.85570255646779};

  std::vector<T> y2{
    96.98573446222625,   -58.675433421313485, -15.58533007526851,  -14.697644147821276,
    85.96236693008059,   38.92770099339309,   19.791693980620906,  27.483461653596166,
    53.91447892576453,   75.83100042363395,   17.73746513670771,   51.50105094020323,
    33.83904611309756,   -9.59805189545494,   27.567402061211244,  33.72816965802343,
    48.98821930718205,   -14.861794980690213, 0.13287706149869294, 35.05682115680253,
    88.14369170856402,   -20.655621067301244, -36.15962607484525,  23.463908856814932,
    95.93206680397306,   10.936188747304243,  -76.64604957338365,  -44.27118733203363,
    -17.066191002518682, 51.827990165726675,  -55.472330987826744, 82.31391457552668,
    -99.25207116240846,  -8.9622361202783,    -14.764596152666753, 35.51101965248979,
    -7.515215371057382,  -12.734669471901016, -76.18168200736743,  -58.82174033449078,
    -64.55998759489724,  -66.29491004534883,  96.90488209719925,   -42.97997451919843,
    -31.865981559056365, -96.36343702487376,  -84.20827193890962,  26.79428452012931,
    62.912038904465774,  -87.227673692568,    11.2934368901489,    -65.442146916886,
    85.68799018964843,   61.94678236143925,   83.46238187197174,   21.333768673112008,
    61.8718601660381,    -35.70805034839669,  68.43167377857928,   -18.400251392936294,
    25.277688476279536,  -74.94714347783905,  2.391028130810602,   -78.06742777647494,
    73.16329191776757,   -5.425513550228256,  -17.11543472509981,  -21.571671681683625,
    60.95981137578463,   -87.30779120172515,  46.07464276698177,   -26.735186694206213,
    77.34113840661823,   -10.89097657623882,  -7.483005212073712,  -24.163324686785494,
    66.03877277717585,   46.514678630068175,  86.52324722682492,   23.88758093704468,
    32.70460360118328,   47.3873043949026,    -40.72743971179719,  96.60257606822059,
    -93.1284937647867,   -70.26297209791194,  94.52718104748459,   68.27804048047095,
    -74.27404656785302,  -21.16650114972075,  -34.93847763736745,  66.55335171298651,
    -88.44856487882186,  -23.53818606503958,  -29.02780534888051,  -29.346481830318815,
    74.28318391238213,   -38.37789665677865,  56.28623833724116,   -81.09317815145866};

  std::vector<T> expected{
    128.64717656028176, 87.88562670763609,  90.19632281028372,  91.76013021796666,
    118.4215357030851,  39.44788631062081,  86.58624490836462,  83.77327247860025,
    79.6690804001798,   167.7366440763836,  83.73027552297903,  99.54006861093508,
    56.276686562837135, 70.80573751073386,  128.34122090714868, 119.97639069191793,
    115.15820154183437, 62.91768450568626,  82.47065566268454,  106.88509910638807,
    104.02822613477268, 196.4153033352887,  33.57186030542483,  136.17156536458378,
    24.91330426477482,  183.12555244130633, 10.402491013960068, 78.8891909881514,
    51.325155608916646, 140.57498906651185, 87.55436962189877,  116.056329846112,
    158.26789618636312, 122.3127143880106,  175.65336769339257, 215.7342613973661,
    62.764576137605516, 173.82450721651924, 72.83278521088664,  31.152704497923047,
    69.74971493014701,  135.16371248533227, 156.8113160405468,  17.14989841904493,
    113.33993969348232, 209.1400727201678,  119.63772368951071, 175.72328059917774,
    54.63868144260578,  177.1094683844075,  46.59751045319419,  192.6556145241176,
    158.08460123131488, 28.29189388239395,  124.5848038188644,  155.08622923365692,
    144.85966618486546, 142.16736573734084, 160.92578604203126, 102.39361006963875,
    88.02052587541618,  126.40767969785988, 57.91601260573419,  30.546751247397513,
    130.95046326396607, 69.87298439379285,  78.21308650467851,  145.7285720718672,
    158.70858501146054, 78.78197209323828,  135.71261679147338, 28.579717281106852,
    91.58009372078648,  85.68704702725647,  90.14934991196503,  78.83501748640491,
    40.09634022929284,  167.14546691120552, 149.17295612658907, 122.98674172809133,
    113.17597316247064, 68.87263271597341,  31.86446035391618,  160.31767244865998,
    158.94024585517036, 34.900531808173085, 253.01889830507722, 86.25213267010419,
    94.2922665997649,   79.44626620532313,  69.47712008431841,  128.24056985459816,
    74.53904203761351,  127.79603731531678, 115.13613538697125, 95.93013225849919,
    58.10781125509778,  44.75789949605465,  28.21929483784659,  87.40828630126103};

  auto p1_geom = thrust::make_counting_iterator(0);
  auto p2_geom = thrust::make_counting_iterator(0);

  rmm::device_vector<T> dx1(x1), dy1(y1), dx2(x2), dy2(y2);
  rmm::device_vector<T> got(dx1.size());

  auto p1_begin = make_vec_2d_iterator(dx1.begin(), dy1.begin());
  auto p2_begin = make_vec_2d_iterator(dx2.begin(), dy2.begin());

  auto multipoints_1 = make_multipoint_range(dx1.size(), p1_geom, dx1.size(), p1_begin);
  auto multipoints_2 = make_multipoint_range(dx2.size(), p2_geom, dx2.size(), p2_begin);

  auto ret_it = pairwise_point_distance(multipoints_1, multipoints_2, got.begin());

  thrust::host_vector<T> hgot(got);
  test::expect_vector_equivalent(hgot, expected);
  EXPECT_EQ(expected.size(), std::distance(got.begin(), ret_it));
}

TYPED_TEST(PairwisePointDistanceTest, MultiComponentSinglePair)
{
  using T         = TypeParam;
  using Cart2D    = vec_2d<T>;
  using Cart2DVec = std::vector<Cart2D>;

  rmm::device_vector<int32_t> multipoint_geom1(std::vector<int32_t>{0, 3});
  rmm::device_vector<Cart2D> points1(Cart2DVec{{1.0, 1.0}, {2.5, 1.5}, {-0.1, -0.7}});
  rmm::device_vector<int32_t> multipoint_geom2(std::vector<int32_t>{0, 2});
  rmm::device_vector<Cart2D> points2(Cart2DVec{{1.8, 1.3}, {0.3, 0.6}});

  rmm::device_vector<T> expected{std::vector<T>{T{0.7280109889280517}}};
  rmm::device_vector<T> got(multipoint_geom1.size() - 1);

  auto multipoint_1 = multipoint_range{
    multipoint_geom1.begin(), multipoint_geom1.end(), points1.begin(), points1.end()};
  auto multipoint_2 = multipoint_range{
    multipoint_geom2.begin(), multipoint_geom2.end(), points2.begin(), points2.end()};

  auto ret_it = pairwise_point_distance(multipoint_1, multipoint_2, got.begin());

  test::expect_vector_equivalent(expected, got);
  EXPECT_EQ(expected.size(), std::distance(got.begin(), ret_it));
}

TYPED_TEST(PairwisePointDistanceTest, MultiComponentRandom)
{
  using T         = TypeParam;
  using Cart2D    = vec_2d<T>;
  using Cart2DVec = std::vector<Cart2D>;

  std::size_t constexpr num_pairs                 = 1000;
  std::size_t constexpr max_points_per_multipoint = 10;
  auto [mp0_offset, mp0_points] =
    this->generate_random_multipoints(num_pairs, max_points_per_multipoint, 0);
  auto [mp1_offset, mp1_points] =
    this->generate_random_multipoints(num_pairs, max_points_per_multipoint, num_pairs);

  auto expected = compute_multipoint_distance_host(mp0_offset, mp0_points, mp1_offset, mp1_points);
  auto got      = rmm::device_vector<T>(num_pairs);

  auto multipoint_1 =
    multipoint_range{mp0_offset.begin(), mp0_offset.end(), mp0_points.begin(), mp0_points.end()};
  auto multipoint_2 =
    multipoint_range{mp1_offset.begin(), mp1_offset.end(), mp1_points.begin(), mp1_points.end()};

  auto ret_it = pairwise_point_distance(multipoint_1, multipoint_2, got.begin());

  test::expect_vector_equivalent(expected, got);
  EXPECT_EQ(expected.size(), std::distance(got.begin(), ret_it));
}

}  // namespace cuspatial
