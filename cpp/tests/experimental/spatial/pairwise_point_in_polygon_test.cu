#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2022, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cuspatial/error.hpp>
#include <cuspatial/experimental/iterator_factory.cuh>
#include <cuspatial/experimental/pairwise_point_in_polygon.cuh>
#include <cuspatial/vec_2d.hpp>

#include <rmm/device_vector.hpp>

#include <thrust/host_vector.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/transform_iterator.h>

#include <gtest/gtest.h>

using namespace cuspatial;

template <typename T>
struct PairwisePointInPolygonTest : public ::testing::Test {
 public:
  rmm::device_vector<vec_2d<T>> make_device_points(std::initializer_list<vec_2d<T>> pts)
  {
    return rmm::device_vector<vec_2d<T>>(pts.begin(), pts.end());
  }

  rmm::device_vector<std::size_t> make_device_offsets(std::initializer_list<std::size_t> pts)
  {
    return rmm::device_vector<std::size_t>(pts.begin(), pts.end());
  }
};

// float and double are logically the same but would require separate tests due to precision.
using TestTypes = ::testing::Types<float, double>;
TYPED_TEST_CASE(PairwisePointInPolygonTest, TestTypes);

TYPED_TEST(PairwisePointInPolygonTest, OnePolygonOneRing)
{
  using T                = TypeParam;
  auto point_list        = std::vector<std::vector<T>>{{-2.0, 0.0},
                                                {2.0, 0.0},
                                                {0.0, -2.0},
                                                {0.0, 2.0},
                                                {-0.5, 0.0},
                                                {0.5, 0.0},
                                                {0.0, -0.5},
                                                {0.0, 0.5}};
  auto poly_offsets      = this->make_device_offsets({0});
  auto poly_ring_offsets = this->make_device_offsets({0});
  auto poly_point =
    this->make_device_points({{-1.0, -1.0}, {1.0, -1.0}, {1.0, 1.0}, {-1.0, 1.0}, {-1.0, -1.0}});

  auto got      = rmm::device_vector<int32_t>(1);
  auto expected = std::vector<int>{false, false, false, false, true, true, true, true};

  for (size_t i = 0; i < point_list.size(); ++i) {
    auto point = this->make_device_points({{point_list[i][0], point_list[i][1]}});
    auto ret   = pairwise_point_in_polygon(point.begin(),
                                         point.end(),
                                         poly_offsets.begin(),
                                         poly_offsets.end(),
                                         poly_ring_offsets.begin(),
                                         poly_ring_offsets.end(),
                                         poly_point.begin(),
                                         poly_point.end(),
                                         got.begin());
    EXPECT_EQ(got, std::vector<int>({expected[i]}));
    EXPECT_EQ(ret, got.end());
  }
}

TYPED_TEST(PairwisePointInPolygonTest, TwoPolygonsOneRingEach)
{
  using T         = TypeParam;
  auto point_list = std::vector<std::vector<T>>{{-2.0, 0.0},
                                                {2.0, 0.0},
                                                {0.0, -2.0},
                                                {0.0, 2.0},
                                                {-0.5, 0.0},
                                                {0.5, 0.0},
                                                {0.0, -0.5},
                                                {0.0, 0.5}};

  auto poly_offsets      = this->make_device_offsets({0, 1});
  auto poly_ring_offsets = this->make_device_offsets({0, 5});
  auto poly_point        = this->make_device_points({{-1.0, -1.0},
                                              {-1.0, 1.0},
                                              {1.0, 1.0},
                                              {1.0, -1.0},
                                              {-1.0, -1.0},
                                              {0.0, 1.0},
                                              {1.0, 0.0},
                                              {0.0, -1.0},
                                              {-1.0, 0.0},
                                              {0.0, 1.0}});

  auto got      = rmm::device_vector<int32_t>(2);
  auto expected = std::vector<int>({false, false, false, false, true, true, true, true});

  for (size_t i = 0; i < point_list.size() / 2; i = i + 2) {
    auto points = this->make_device_points(
      {{point_list[i][0], point_list[i][1]}, {point_list[i + 1][0], point_list[i + 1][1]}});
    auto ret = pairwise_point_in_polygon(points.begin(),
                                         points.end(),
                                         poly_offsets.begin(),
                                         poly_offsets.end(),
                                         poly_ring_offsets.begin(),
                                         poly_ring_offsets.end(),
                                         poly_point.begin(),
                                         poly_point.end(),
                                         got.begin());

    EXPECT_EQ(got, std::vector<int>({expected[i], expected[i + 1]}));
    EXPECT_EQ(ret, got.end());
  }
}

TYPED_TEST(PairwisePointInPolygonTest, OnePolygonTwoRings)
{
  using T = TypeParam;
  auto point_list =
    std::vector<std::vector<T>>{{0.0, 0.0}, {-0.4, 0.0}, {-0.6, 0.0}, {0.0, 0.4}, {0.0, -0.6}};
  auto poly_offsets      = this->make_device_offsets({0});
  auto poly_ring_offsets = this->make_device_offsets({0, 5});
  auto poly_point        = this->make_device_points({{-1.0, -1.0},
                                              {1.0, -1.0},
                                              {1.0, 1.0},
                                              {-1.0, 1.0},
                                              {-1.0, -1.0},
                                              {-0.5, -0.5},
                                              {-0.5, 0.5},
                                              {0.5, 0.5},
                                              {0.5, -0.5},
                                              {-0.5, -0.5}});

  auto got      = rmm::device_vector<int32_t>(1);
  auto expected = std::vector<int>{0b0, 0b0, 0b1, 0b0, 0b1};

  for (size_t i = 0; i < point_list.size(); ++i) {
    auto point = this->make_device_points({{point_list[i][0], point_list[i][1]}});
    auto ret   = pairwise_point_in_polygon(point.begin(),
                                         point.end(),
                                         poly_offsets.begin(),
                                         poly_offsets.end(),
                                         poly_ring_offsets.begin(),
                                         poly_ring_offsets.end(),
                                         poly_point.begin(),
                                         poly_point.end(),
                                         got.begin());

    EXPECT_EQ(got, std::vector<int>{expected[i]});
    EXPECT_EQ(ret, got.end());
  }
}

TYPED_TEST(PairwisePointInPolygonTest, EdgesOfSquare)
{
  auto test_point   = this->make_device_points({{0.0, 0.0}, {0.0, 0.0}, {0.0, 0.0}, {0.0, 0.0}});
  auto poly_offsets = this->make_device_offsets({0, 1, 2, 3});
  auto poly_ring_offsets = this->make_device_offsets({0, 5, 10, 15});

  // 0: rect on min x side
  // 1: rect on max x side
  // 2: rect on min y side
  // 3: rect on max y side
  auto poly_point = this->make_device_points(
    {{-1.0, -1.0}, {0.0, -1.0}, {0.0, 1.0},  {-1.0, 1.0},  {-1.0, -1.0}, {0.0, -1.0}, {1.0, -1.0},
     {1.0, 1.0},   {0.0, 1.0},  {0.0, -1.0}, {-1.0, -1.0}, {-1.0, 0.0},  {1.0, 0.0},  {1.0, -1.0},
     {-1.0, 1.0},  {-1.0, 0.0}, {-1.0, 1.0}, {1.0, 1.0},   {1.0, 0.0},   {-1.0, 0.0}});

  auto expected = std::vector<int>{0b0, 0b0, 0b0, 0b0};
  auto got      = rmm::device_vector<int32_t>(test_point.size());

  auto ret = pairwise_point_in_polygon(test_point.begin(),
                                       test_point.end(),
                                       poly_offsets.begin(),
                                       poly_offsets.end(),
                                       poly_ring_offsets.begin(),
                                       poly_ring_offsets.end(),
                                       poly_point.begin(),
                                       poly_point.end(),
                                       got.begin());

  EXPECT_EQ(got, expected);
  EXPECT_EQ(ret, got.end());
}

TYPED_TEST(PairwisePointInPolygonTest, CornersOfSquare)
{
  auto test_point   = this->make_device_points({{0.0, 0.0}, {0.0, 0.0}, {0.0, 0.0}, {0.0, 0.0}});
  auto poly_offsets = this->make_device_offsets({0, 1, 2, 3});
  auto poly_ring_offsets = this->make_device_offsets({0, 5, 10, 15});

  // 0: min x min y corner
  // 1: min x max y corner
  // 2: max x min y corner
  // 3: max x max y corner
  auto poly_point = this->make_device_points(
    {{-1.0, -1.0}, {-1.0, 0.0}, {0.0, 0.0},  {0.0, -1.0}, {-1.0, -1.0}, {-1.0, 0.0}, {-1.0, 1.0},
     {0.0, 1.0},   {-1.0, 0.0}, {-1.0, 0.0}, {0.0, -1.0}, {0.0, 0.0},   {1.0, 0.0},  {1.0, -1.0},
     {0.0, -1.0},  {0.0, 0.0},  {0.0, 1.0},  {1.0, 1.0},  {1.0, 0.0},   {0.0, 0.0}});

  auto expected = std::vector<int>{0b0, 0b0, 0b0, 0b0};
  auto got      = rmm::device_vector<int32_t>(test_point.size());

  auto ret = pairwise_point_in_polygon(test_point.begin(),
                                       test_point.end(),
                                       poly_offsets.begin(),
                                       poly_offsets.end(),
                                       poly_ring_offsets.begin(),
                                       poly_ring_offsets.end(),
                                       poly_point.begin(),
                                       poly_point.end(),
                                       got.begin());

  EXPECT_EQ(got, expected);
  EXPECT_EQ(ret, got.end());
}

struct OffsetIteratorFunctor {
  std::size_t __device__ operator()(std::size_t idx) { return idx * 5; }
};

template <typename T>
struct PolyPointIteratorFunctorA {
  T __device__ operator()(std::size_t idx)
  {
    switch (idx % 5) {
      case 0:
      case 1: return -1.0;
      case 2:
      case 3: return 1.0;
      case 4:
      default: return -1.0;
    }
  }
};

template <typename T>
struct PolyPointIteratorFunctorB {
  T __device__ operator()(std::size_t idx)
  {
    switch (idx % 5) {
      case 0: return -1.0;
      case 1:
      case 2: return 1.0;
      case 3:
      case 4:
      default: return -1.0;
    }
  }
};

TYPED_TEST(PairwisePointInPolygonTest, 32PolygonSupport)
{
  using T = TypeParam;

  auto constexpr num_polys       = 32;
  auto constexpr num_poly_points = num_polys * 5;

  auto test_point = this->make_device_points(
    {{0.0, 0.0}, {2.0, 0.0}, {0.0, 0.0}, {2.0, 0.0}, {0.0, 0.0}, {2.0, 0.0}, {0.0, 0.0},
     {2.0, 0.0}, {0.0, 0.0}, {2.0, 0.0}, {0.0, 0.0}, {2.0, 0.0}, {0.0, 0.0}, {2.0, 0.0},
     {0.0, 0.0}, {2.0, 0.0}, {0.0, 0.0}, {2.0, 0.0}, {0.0, 0.0}, {2.0, 0.0}, {0.0, 0.0},
     {2.0, 0.0}, {0.0, 0.0}, {2.0, 0.0}, {0.0, 0.0}, {2.0, 0.0}, {0.0, 0.0}, {2.0, 0.0},
     {0.0, 0.0}, {2.0, 0.0}, {0.0, 0.0}, {2.0, 0.0}});
  auto offsets_iter = thrust::make_counting_iterator<std::size_t>(0);
  auto poly_ring_offsets_iter =
    thrust::make_transform_iterator(offsets_iter, OffsetIteratorFunctor{});
  auto poly_point_xs_iter =
    thrust::make_transform_iterator(offsets_iter, PolyPointIteratorFunctorA<T>{});
  auto poly_point_ys_iter =
    thrust::make_transform_iterator(offsets_iter, PolyPointIteratorFunctorB<T>{});
  auto poly_point_iter = make_vec_2d_iterator(poly_point_xs_iter, poly_point_ys_iter);

  auto expected = std::vector<int>({1, 0, 1, 0, 1, 0, 1, 0, 1, 0, 1, 0, 1, 0, 1, 0,
                                    1, 0, 1, 0, 1, 0, 1, 0, 1, 0, 1, 0, 1, 0, 1, 0});
  auto got      = rmm::device_vector<int32_t>(test_point.size());

  auto ret = pairwise_point_in_polygon(test_point.begin(),
                                       test_point.end(),
                                       offsets_iter,
                                       offsets_iter + num_polys,
                                       poly_ring_offsets_iter,
                                       poly_ring_offsets_iter + num_polys,
                                       poly_point_iter,
                                       poly_point_iter + num_poly_points,
                                       got.begin());

  EXPECT_EQ(got, expected);
  EXPECT_EQ(ret, got.end());
}

struct PairwisePointInPolygonErrorTest : public PairwisePointInPolygonTest<double> {
};

TEST_F(PairwisePointInPolygonErrorTest, MismatchPolyPointXYLength)
{
  using T = double;

  auto test_point        = this->make_device_points({{0.0, 0.0}, {0.0, 0.0}});
  auto poly_offsets      = this->make_device_offsets({0});
  auto poly_ring_offsets = this->make_device_offsets({0});
  auto poly_point        = this->make_device_points({{0.0, 1.0}, {1.0, 0.0}, {0.0, -1.0}});
  auto got               = rmm::device_vector<int32_t>(test_point.size());

  EXPECT_THROW(pairwise_point_in_polygon(test_point.begin(),
                                         test_point.end(),
                                         poly_offsets.begin(),
                                         poly_offsets.end(),
                                         poly_ring_offsets.begin(),
                                         poly_ring_offsets.end(),
                                         poly_point.begin(),
                                         poly_point.end(),
                                         got.begin()),
               cuspatial::logic_error);
}

TYPED_TEST(PairwisePointInPolygonTest, SelfClosingLoopLeftEdgeMissing)
{
  using T                = TypeParam;
  auto point_list        = std::vector<std::vector<T>>{{-2.0, 0.0}, {0.0, 0.0}, {2.0, 0.0}};
  auto poly_offsets      = this->make_device_offsets({0});
  auto poly_ring_offsets = this->make_device_offsets({0});
  // "left" edge missing
  auto poly_point = this->make_device_points({{-1, 1}, {1, 1}, {1, -1}, {-1, -1}});
  auto expected   = std::vector<int>{0b0, 0b1, 0b0};
  auto got        = rmm::device_vector<int32_t>(1);

  for (size_t i = 0; i < point_list.size(); ++i) {
    auto point = this->make_device_points({{point_list[i][0], point_list[i][1]}});
    auto ret   = pairwise_point_in_polygon(point.begin(),
                                         point.end(),
                                         poly_offsets.begin(),
                                         poly_offsets.end(),
                                         poly_ring_offsets.begin(),
                                         poly_ring_offsets.end(),
                                         poly_point.begin(),
                                         poly_point.end(),
                                         got.begin());

    EXPECT_EQ(std::vector<int>{expected[i]}, got);
    EXPECT_EQ(got.end(), ret);
  }
}

TYPED_TEST(PairwisePointInPolygonTest, SelfClosingLoopRightEdgeMissing)
{
  using T                = TypeParam;
  auto point_list        = std::vector<std::vector<T>>{{-2.0, 0.0}, {0.0, 0.0}, {2.0, 0.0}};
  auto poly_offsets      = this->make_device_offsets({0});
  auto poly_ring_offsets = this->make_device_offsets({0});
  // "right" edge missing
  auto poly_point = this->make_device_points({{1, -1}, {-1, -1}, {-1, 1}, {1, 1}});
  auto expected   = std::vector<int>{0b0, 0b1, 0b0};
  auto got        = rmm::device_vector<int32_t>(1);
  for (size_t i = 0; i < point_list.size(); ++i) {
    auto point = this->make_device_points({{point_list[i][0], point_list[i][1]}});
    auto ret   = pairwise_point_in_polygon(point.begin(),
                                         point.end(),
                                         poly_offsets.begin(),
                                         poly_offsets.end(),
                                         poly_ring_offsets.begin(),
                                         poly_ring_offsets.end(),
                                         poly_point.begin(),
                                         poly_point.end(),
                                         got.begin());

    EXPECT_EQ(std::vector<int>{expected[i]}, got);
    EXPECT_EQ(got.end(), ret);
  }
}
