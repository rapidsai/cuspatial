/*
 * Copyright (c) 2022, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cuspatial/error.hpp>
#include <cuspatial/experimental/linestring_distance.cuh>
#include <cuspatial/experimental/type_utils.hpp>
#include <cuspatial/vec_2d.hpp>

#include <rmm/device_uvector.hpp>
#include <rmm/device_vector.hpp>

#include <cudf_test/base_fixture.hpp>
#include <cudf_test/column_utilities.hpp>
#include <cudf_test/column_wrapper.hpp>
#include <cudf_test/type_lists.hpp>

#include <thrust/iterator/constant_iterator.h>
#include <thrust/iterator/counting_iterator.h>

namespace cuspatial {
namespace test {

using namespace cudf;
using namespace cudf::test;

template <typename T>
struct PairwiseLinestringDistanceTest : public BaseFixture {
};

// float and double are logically the same but would require seperate tests due to precision.
using TestTypes = ::testing::Types<float, double>;
TYPED_TEST_CASE(PairwiseLinestringDistanceTest, TestTypes);

TYPED_TEST(PairwiseLinestringDistanceTest, FromSeparateArrayInputs)
{
  using T       = TypeParam;
  using CartVec = std::vector<vec_2d<T>>;

  auto a_cart2d = rmm::device_vector<vec_2d<T>>{
    CartVec({{0.0f, 0.0f}, {1.0f, 0.0f}, {2.0f, 0.0f}, {3.0f, 0.0f}, {4.0f, 0.0f}})};
  auto b_cart2d = rmm::device_vector<vec_2d<T>>{
    CartVec({{0.0f, 1.0f}, {1.0f, 1.0f}, {2.0f, 1.0f}, {3.0f, 1.0f}, {4.0f, 1.0f}})};
  auto offset = rmm::device_vector<int32_t>{std::vector<int32_t>{0}};

  auto distance = rmm::device_vector<T>{1};
  auto expected = rmm::device_vector<T>{std::vector<T>{1.0}};

  pairwise_linestring_distance(offset.begin(),
                               offset.end(),
                               a_cart2d.begin(),
                               a_cart2d.end(),
                               offset.begin(),
                               b_cart2d.begin(),
                               b_cart2d.end(),
                               distance.begin());

  EXPECT_EQ(distance, expected);
}

TYPED_TEST(PairwiseLinestringDistanceTest, FromSamePointArrayInput)
{
  using T       = TypeParam;
  using CartVec = std::vector<vec_2d<T>>;

  auto cart2ds = rmm::device_vector<vec_2d<T>>{
    CartVec({{0.0f, 0.0f}, {1.0f, 0.0f}, {2.0f, 0.0f}, {3.0f, 0.0f}, {4.0f, 0.0f}})};
  auto offset = rmm::device_vector<int32_t>{std::vector<int32_t>{0}};

  auto a_begin = cart2ds.begin();
  auto a_end   = cart2ds.begin() + 3;
  auto b_begin = cart2ds.begin() + 1;
  auto b_end   = cart2ds.end();

  auto distance = rmm::device_vector<T>{1};
  auto expected = rmm::device_vector<T>{std::vector<T>{0.0}};

  pairwise_linestring_distance(
    offset.begin(), offset.end(), a_begin, a_end, offset.begin(), b_begin, b_end, distance.begin());

  EXPECT_EQ(distance, expected);
}

TYPED_TEST(PairwiseLinestringDistanceTest, FromTransformIterator)
{
  using T       = TypeParam;
  using CartVec = std::vector<vec_2d<T>>;

  auto a_cart2d_x = rmm::device_vector<T>{std::vector<T>{0.0, 1.0, 2.0, 3.0, 4.0}};
  auto a_cart2d_y = rmm::device_vector<T>(5, 0.0);

  auto a_begin = make_vec_2d_iterator(a_cart2d_x.begin(), a_cart2d_y.begin());
  auto a_end   = a_begin + a_cart2d_x.size();

  auto b_cart2d_x = rmm::device_vector<T>{std::vector<T>{0.0, 1.0, 2.0, 3.0, 4.0}};
  auto b_cart2d_y = rmm::device_vector<T>(5, 1.0);

  auto b_begin = make_vec_2d_iterator(b_cart2d_x.begin(), b_cart2d_y.begin());
  auto b_end   = b_begin + b_cart2d_x.size();

  auto offset = rmm::device_vector<int32_t>{std::vector<int32_t>{0}};

  auto distance = rmm::device_vector<T>{1};
  auto expected = rmm::device_vector<T>{std::vector<T>{1.0}};

  pairwise_linestring_distance(
    offset.begin(), offset.end(), a_begin, a_end, offset.begin(), b_begin, b_end, distance.begin());

  EXPECT_EQ(distance, expected);
}

TYPED_TEST(PairwiseLinestringDistanceTest, FromMixedIterator)
{
  using T       = TypeParam;
  using CartVec = std::vector<vec_2d<T>>;

  auto a_cart2d = rmm::device_vector<vec_2d<T>>{
    CartVec({{0.0f, 0.0f}, {1.0f, 0.0f}, {2.0f, 0.0f}, {3.0f, 0.0f}, {4.0f, 0.0f}})};

  auto b_cart2d_x = rmm::device_vector<T>{std::vector<T>{0.0, 1.0, 2.0, 3.0, 4.0}};
  auto b_cart2d_y = rmm::device_vector<T>(5, 1.0);

  auto b_begin = make_vec_2d_iterator(b_cart2d_x.begin(), b_cart2d_y.begin());
  auto b_end   = b_begin + b_cart2d_x.size();

  auto offset = rmm::device_vector<int32_t>{std::vector<int32_t>{0}};

  auto distance = rmm::device_vector<T>{1};
  auto expected = rmm::device_vector<T>{std::vector<T>{1.0}};

  pairwise_linestring_distance(offset.begin(),
                               offset.end(),
                               a_cart2d.begin(),
                               a_cart2d.end(),
                               offset.begin(),
                               b_begin,
                               b_end,
                               distance.begin());

  EXPECT_EQ(distance, expected);
}

TYPED_TEST(PairwiseLinestringDistanceTest, FromLongInputs)
{
  using T       = TypeParam;
  using CartVec = std::vector<vec_2d<T>>;

  auto num_points = 1000;

  auto a_cart2d_x_begin = thrust::make_constant_iterator(T{0.0});
  auto a_cart2d_y_begin = thrust::make_counting_iterator(T{0.0});
  auto a_cart2d_begin   = make_vec_2d_iterator(a_cart2d_x_begin, a_cart2d_y_begin);
  auto a_cart2d_end     = a_cart2d_begin + num_points;

  auto b_cart2d_x_begin = thrust::make_constant_iterator(T{42.0});
  auto b_cart2d_y_begin = thrust::make_counting_iterator(T{0.0});
  auto b_cart2d_begin   = make_vec_2d_iterator(b_cart2d_x_begin, b_cart2d_y_begin);
  auto b_cart2d_end     = b_cart2d_begin + num_points;

  auto offset = rmm::device_vector<int32_t>{std::vector<int32_t>{0, 100, 200, 300, 400}};

  auto distance = rmm::device_vector<T>{5};
  auto expected = rmm::device_vector<T>{std::vector<T>{42.0, 42.0, 42.0, 42.0, 42.0}};

  pairwise_linestring_distance(offset.begin(),
                               offset.end(),
                               a_cart2d_begin,
                               a_cart2d_end,
                               offset.begin(),
                               b_cart2d_begin,
                               b_cart2d_end,
                               distance.begin());

  EXPECT_EQ(distance, expected);
}

}  // namespace test
}  // namespace cuspatial
