#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2022, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cuspatial_test/base_fixture.hpp>
#include <cuspatial_test/vector_equality.hpp>

#include <cuspatial/cuda_utils.hpp>
#include <cuspatial/detail/utility/linestring.cuh>
#include <cuspatial/error.hpp>
#include <cuspatial/vec_2d.hpp>

#include <rmm/device_vector.hpp>

#include <thrust/execution_policy.h>
#include <thrust/optional.h>
#include <thrust/pair.h>

#include <gtest/gtest.h>

#include <optional>

using namespace cuspatial;
using namespace cuspatial::detail;
using namespace cuspatial::test;

template <typename T>
using optional_vec2d = thrust::optional<vec_2d<T>>;

template <typename T>
struct SegmentIntersectionTest : public BaseFixture {
};

using TestTypes = ::testing::Types<float, double>;
TYPED_TEST_CASE(SegmentIntersectionTest, TestTypes);

template <typename T>
segment<T> __device__ order_end_points(segment<T> const& seg)
{
  auto [a, b] = seg;
  return a < b ? segment<T>{a, b} : segment<T>{b, a};
}

template <typename T, typename Point, typename Segment>
void __global__
compute_intersection(segment<T> ab, segment<T> cd, Point point_out, Segment segment_out)
{
  auto [p, s]    = detail::segment_intersection(ab, cd);
  point_out[0]   = p;
  segment_out[0] = s.has_value() ? thrust::optional(order_end_points(s.value())) : s;
}

template <typename T>
struct unpack_optional_segment {
  thrust::tuple<optional_vec2d<T>, optional_vec2d<T>> CUSPATIAL_HOST_DEVICE
  operator()(thrust::optional<segment<T>> segment)
  {
    if (segment.has_value())
      return thrust::make_tuple(segment.value().v1, segment.value().v2);
    else
      return thrust::tuple<optional_vec2d<T>, optional_vec2d<T>>{thrust::nullopt, thrust::nullopt};
  }
};

template <typename T>
void run_single_intersection_test(
  segment<T> const& ab,
  segment<T> const& cd,
  std::vector<thrust::optional<vec_2d<T>>> const& points_expected,
  std::vector<thrust::optional<segment<T>>> const& segments_expected)
{
  rmm::device_vector<thrust::optional<vec_2d<T>>> points_got(points_expected.size());
  rmm::device_vector<thrust::optional<segment<T>>> segments_got(segments_expected.size());

  compute_intersection<<<1, 1>>>(ab, cd, points_got.data(), segments_got.data());

  // Unpack the segment into two separate optional vec_2d column.
  rmm::device_vector<thrust::optional<vec_2d<T>>> first(segments_got.size());
  rmm::device_vector<thrust::optional<vec_2d<T>>> second(segments_got.size());
  auto outit = thrust::make_zip_iterator(first.begin(), second.begin());

  thrust::transform(segments_got.begin(), segments_got.end(), outit, unpack_optional_segment<T>{});

  std::vector<thrust::optional<vec_2d<T>>> expected_first(segments_expected.size());
  std::vector<thrust::optional<vec_2d<T>>> expected_second(segments_expected.size());
  auto h_outit = thrust::make_zip_iterator(expected_first.begin(), expected_second.begin());

  thrust::transform(thrust::host,
                    segments_expected.begin(),
                    segments_expected.end(),
                    h_outit,
                    unpack_optional_segment<T>{});

  CUSPATIAL_EXPECT_VECTORS_EQUIVALENT(points_got, points_expected);
  CUSPATIAL_EXPECT_VECTORS_EQUIVALENT(first, expected_first);
  CUSPATIAL_EXPECT_VECTORS_EQUIVALENT(second, expected_second);
}

TYPED_TEST(SegmentIntersectionTest, SimpleIntersect)
{
  using T = TypeParam;

  segment<T> ab{{0.0, 0.0}, {1.0, 1.0}};
  segment<T> cd{{0.0, 1.0}, {1.0, 0.0}};

  std::vector<thrust::optional<vec_2d<T>>> points_expected{vec_2d<T>{0.5, 0.5}};
  std::vector<thrust::optional<segment<T>>> segments_expected{thrust::nullopt};

  run_single_intersection_test(ab, cd, points_expected, segments_expected);
}

TYPED_TEST(SegmentIntersectionTest, IntersectAtEndPoint)
{
  using T = TypeParam;

  segment<T> ab{{0.0, 0.0}, {1.0, 1.0}};
  segment<T> cd{{1.0, 1.0}, {1.0, 0.0}};

  std::vector<thrust::optional<vec_2d<T>>> points_expected{vec_2d<T>{1.0, 1.0}};
  std::vector<thrust::optional<segment<T>>> segments_expected{thrust::nullopt};

  run_single_intersection_test(ab, cd, points_expected, segments_expected);
}

TYPED_TEST(SegmentIntersectionTest, UnparallelDisjoint1)
{
  using T = TypeParam;

  segment<T> ab{{0.0, 0.0}, {0.4, 1.0}};
  segment<T> cd{{1.0, 0.0}, {0.6, 1.0}};

  std::vector<thrust::optional<vec_2d<T>>> points_expected{thrust::nullopt};
  std::vector<thrust::optional<segment<T>>> segments_expected{thrust::nullopt};

  run_single_intersection_test(ab, cd, points_expected, segments_expected);
}

TYPED_TEST(SegmentIntersectionTest, UnparallelDisjoint2)
{
  using T = TypeParam;

  segment<T> ab{{0.0, 0.0}, {1.0, 0.0}};
  segment<T> cd{{2.0, 0.0}, {2.0, 1.0}};

  std::vector<thrust::optional<vec_2d<T>>> points_expected{thrust::nullopt};
  std::vector<thrust::optional<segment<T>>> segments_expected{thrust::nullopt};

  run_single_intersection_test(ab, cd, points_expected, segments_expected);
}

TYPED_TEST(SegmentIntersectionTest, ParallelDisjoint1)
{
  using T = TypeParam;

  segment<T> ab{{0.0, 0.0}, {0.0, 1.0}};
  segment<T> cd{{1.0, 0.0}, {1.0, 1.0}};

  std::vector<thrust::optional<vec_2d<T>>> points_expected{thrust::nullopt};
  std::vector<thrust::optional<segment<T>>> segments_expected{thrust::nullopt};

  run_single_intersection_test(ab, cd, points_expected, segments_expected);
}

TYPED_TEST(SegmentIntersectionTest, ParallelDisjoint2)
{
  using T = TypeParam;

  segment<T> ab{{0.0, 0.0}, {1.0, 0.0}};
  segment<T> cd{{0.0, 1.0}, {1.0, 1.0}};

  std::vector<thrust::optional<vec_2d<T>>> points_expected{thrust::nullopt};
  std::vector<thrust::optional<segment<T>>> segments_expected{thrust::nullopt};

  run_single_intersection_test(ab, cd, points_expected, segments_expected);
}

TYPED_TEST(SegmentIntersectionTest, ParallelDisjoint3)
{
  using T = TypeParam;

  segment<T> ab{{0.0, 0.0}, {1.0, 1.0}};
  segment<T> cd{{1.0, 0.0}, {2.0, 1.0}};

  std::vector<thrust::optional<vec_2d<T>>> points_expected{thrust::nullopt};
  std::vector<thrust::optional<segment<T>>> segments_expected{thrust::nullopt};

  run_single_intersection_test(ab, cd, points_expected, segments_expected);
}

TYPED_TEST(SegmentIntersectionTest, ParallelDisjoint4)
{
  using T = TypeParam;

  segment<T> ab{{0.0, 0.0}, {0.0, -1.0}};
  segment<T> cd{{1.0, 0.0}, {1.0, 1.0}};

  std::vector<thrust::optional<vec_2d<T>>> points_expected{thrust::nullopt};
  std::vector<thrust::optional<segment<T>>> segments_expected{thrust::nullopt};

  run_single_intersection_test(ab, cd, points_expected, segments_expected);
}

TYPED_TEST(SegmentIntersectionTest, CollinearDisjoint1)
{
  using T = TypeParam;

  segment<T> ab{{0.0, 0.0}, {1.0, 0.0}};
  segment<T> cd{{2.0, 0.0}, {3.0, 0.0}};

  std::vector<thrust::optional<vec_2d<T>>> points_expected{thrust::nullopt};
  std::vector<thrust::optional<segment<T>>> segments_expected{thrust::nullopt};

  run_single_intersection_test(ab, cd, points_expected, segments_expected);
}

TYPED_TEST(SegmentIntersectionTest, CollinearDisjoint2)
{
  using T = TypeParam;

  segment<T> ab{{0.0, 0.0}, {1.0, 0.0}};
  segment<T> cd{{-1.0, 0.0}, {-2.0, 0.0}};

  std::vector<thrust::optional<vec_2d<T>>> points_expected{thrust::nullopt};
  std::vector<thrust::optional<segment<T>>> segments_expected{thrust::nullopt};

  run_single_intersection_test(ab, cd, points_expected, segments_expected);
}

TYPED_TEST(SegmentIntersectionTest, CollinearDisjoint3)
{
  using T = TypeParam;

  segment<T> ab{{0.0, 0.0}, {0.0, 1.0}};
  segment<T> cd{{0.0, 2.0}, {0.0, 3.0}};

  std::vector<thrust::optional<vec_2d<T>>> points_expected{thrust::nullopt};
  std::vector<thrust::optional<segment<T>>> segments_expected{thrust::nullopt};

  run_single_intersection_test(ab, cd, points_expected, segments_expected);
}

TYPED_TEST(SegmentIntersectionTest, CollinearDisjoint4)
{
  using T = TypeParam;

  segment<T> ab{{0.0, 0.0}, {0.0, 1.0}};
  segment<T> cd{{0.0, -1.0}, {0.0, -2.0}};

  std::vector<thrust::optional<vec_2d<T>>> points_expected{thrust::nullopt};
  std::vector<thrust::optional<segment<T>>> segments_expected{thrust::nullopt};

  run_single_intersection_test(ab, cd, points_expected, segments_expected);
}

TYPED_TEST(SegmentIntersectionTest, CollinearDisjoint5)
{
  using T = TypeParam;

  segment<T> ab{{0.0, 0.0}, {1.0, 1.0}};
  segment<T> cd{{2.0, 2.0}, {3.0, 3.0}};

  std::vector<thrust::optional<vec_2d<T>>> points_expected{thrust::nullopt};
  std::vector<thrust::optional<segment<T>>> segments_expected{thrust::nullopt};

  run_single_intersection_test(ab, cd, points_expected, segments_expected);
}

TYPED_TEST(SegmentIntersectionTest, CollinearDisjoint6)
{
  using T = TypeParam;

  segment<T> ab{{0.0, 0.0}, {1.0, 1.0}};
  segment<T> cd{{-1.0, -1.0}, {-2.0, -2.0}};

  std::vector<thrust::optional<vec_2d<T>>> points_expected{thrust::nullopt};
  std::vector<thrust::optional<segment<T>>> segments_expected{thrust::nullopt};

  run_single_intersection_test(ab, cd, points_expected, segments_expected);
}

TYPED_TEST(SegmentIntersectionTest, Overlap1)
{
  using T = TypeParam;

  segment<T> ab{{0.0, 0.0}, {1.0, 1.0}};
  segment<T> cd{{0.5, 0.5}, {1.5, 1.5}};

  std::vector<thrust::optional<vec_2d<T>>> points_expected{thrust::nullopt};
  std::vector<thrust::optional<segment<T>>> segments_expected{segment<T>{{0.5, 0.5}, {1.0, 1.0}}};

  run_single_intersection_test(ab, cd, points_expected, segments_expected);
}

TYPED_TEST(SegmentIntersectionTest, Overlap2)
{
  using T = TypeParam;

  segment<T> ab{{0.0, 0.0}, {1.0, 1.0}};
  segment<T> cd{{0.5, 0.5}, {-1.5, -1.5}};

  std::vector<thrust::optional<vec_2d<T>>> points_expected{thrust::nullopt};
  std::vector<thrust::optional<segment<T>>> segments_expected{segment<T>{{0.0, 0.0}, {0.5, 0.5}}};

  run_single_intersection_test(ab, cd, points_expected, segments_expected);
}

TYPED_TEST(SegmentIntersectionTest, Overlap3)
{
  using T = TypeParam;

  segment<T> ab{{0.0, 0.0}, {1.0, 0.0}};
  segment<T> cd{{0.5, 0.0}, {2.0, 0.0}};

  std::vector<thrust::optional<vec_2d<T>>> points_expected{thrust::nullopt};
  std::vector<thrust::optional<segment<T>>> segments_expected{segment<T>{{0.5, 0.0}, {1.0, 0.0}}};

  run_single_intersection_test(ab, cd, points_expected, segments_expected);
}

TYPED_TEST(SegmentIntersectionTest, Overlap4)
{
  using T = TypeParam;

  segment<T> ab{{0.0, 0.0}, {1.0, 0.0}};
  segment<T> cd{{0.5, 0.0}, {-1.0, 0.0}};

  std::vector<thrust::optional<vec_2d<T>>> points_expected{thrust::nullopt};
  std::vector<thrust::optional<segment<T>>> segments_expected{segment<T>{{0.0, 0.0}, {0.5, 0.0}}};

  run_single_intersection_test(ab, cd, points_expected, segments_expected);
}

TYPED_TEST(SegmentIntersectionTest, Overlap5)
{
  using T = TypeParam;

  segment<T> ab{{0.0, 0.0}, {0.0, 1.0}};
  segment<T> cd{{0.0, 0.5}, {0.0, 2.0}};

  std::vector<thrust::optional<vec_2d<T>>> points_expected{thrust::nullopt};
  std::vector<thrust::optional<segment<T>>> segments_expected{segment<T>{{0.0, 0.5}, {0.0, 1.0}}};

  run_single_intersection_test(ab, cd, points_expected, segments_expected);
}

TYPED_TEST(SegmentIntersectionTest, Overlap6)
{
  using T = TypeParam;

  segment<T> ab{{0.0, 0.0}, {0.0, 1.0}};
  segment<T> cd{{0.0, 0.5}, {0.0, -2.0}};

  std::vector<thrust::optional<vec_2d<T>>> points_expected{thrust::nullopt};
  std::vector<thrust::optional<segment<T>>> segments_expected{segment<T>{{0.0, 0.0}, {0.0, 0.5}}};

  run_single_intersection_test(ab, cd, points_expected, segments_expected);
}

TYPED_TEST(SegmentIntersectionTest, Overlap7)
{
  using T = TypeParam;

  segment<T> ab{{0.0, 0.0}, {0.0, 1.0}};
  segment<T> cd{{0.0, 0.0}, {0.0, 0.5}};

  std::vector<thrust::optional<vec_2d<T>>> points_expected{thrust::nullopt};
  std::vector<thrust::optional<segment<T>>> segments_expected{segment<T>{{0.0, 0.0}, {0.0, 0.5}}};

  run_single_intersection_test(ab, cd, points_expected, segments_expected);
}

TYPED_TEST(SegmentIntersectionTest, Overlap8)
{
  using T = TypeParam;

  segment<T> ab{{0.0, 0.0}, {0.0, 1.0}};
  segment<T> cd{{0.0, 0.5}, {0.0, 1.0}};

  std::vector<thrust::optional<vec_2d<T>>> points_expected{thrust::nullopt};
  std::vector<thrust::optional<segment<T>>> segments_expected{segment<T>{{0.0, 0.5}, {0.0, 1.0}}};

  run_single_intersection_test(ab, cd, points_expected, segments_expected);
}

TYPED_TEST(SegmentIntersectionTest, Overlap9)
{
  using T = TypeParam;

  segment<T> ab{{0.0, 0.0}, {0.0, 1.0}};
  segment<T> cd{{0.0, 0.25}, {0.0, 0.75}};

  std::vector<thrust::optional<vec_2d<T>>> points_expected{thrust::nullopt};
  std::vector<thrust::optional<segment<T>>> segments_expected{cd};

  run_single_intersection_test(ab, cd, points_expected, segments_expected);
}

TYPED_TEST(SegmentIntersectionTest, Overlap10)
{
  using T = TypeParam;

  segment<T> ab{{0.0, 0.25}, {0.0, 0.75}};
  segment<T> cd{{0.0, 0.0}, {0.0, 1.0}};

  std::vector<thrust::optional<vec_2d<T>>> points_expected{thrust::nullopt};
  std::vector<thrust::optional<segment<T>>> segments_expected{ab};

  run_single_intersection_test(ab, cd, points_expected, segments_expected);
}
