/*
 * Copyright (c) 2023, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cuspatial_test/base_fixture.hpp>
#include <cuspatial_test/vector_equality.hpp>
#include <cuspatial_test/vector_factories.cuh>

#include <cuspatial/constants.hpp>
#include <cuspatial/error.hpp>
#include <cuspatial/geometry/vec_2d.hpp>
#include <cuspatial/pairwise_multipoint_equals_count.cuh>

using namespace cuspatial;
using namespace cuspatial::test;

template <typename T>
struct PairwiseMultipointEqualsCountTest : public BaseFixture {
  void run_single(std::initializer_list<std::initializer_list<vec_2d<T>>> lhs_coordinates,
                  std::initializer_list<std::initializer_list<vec_2d<T>>> rhs_coordinates,
                  std::initializer_list<uint32_t> expected)
  {
    auto larray = make_multipoints_array(lhs_coordinates);
    auto rarray = make_multipoints_array(rhs_coordinates);

    auto lhs = larray.range();
    auto rhs = rarray.range();

    auto got = rmm::device_uvector<uint32_t>(lhs.size(), stream());

    auto ret = pairwise_multipoint_equals_count(lhs, rhs, got.begin(), stream());

    auto d_expected = make_device_vector(expected);

    CUSPATIAL_EXPECT_VECTORS_EQUIVALENT(got, d_expected);
    EXPECT_EQ(ret, got.end());
  }
};

using TestTypes = ::testing::Types<float, double>;

TYPED_TEST_CASE(PairwiseMultipointEqualsCountTest, TestTypes);

TYPED_TEST(PairwiseMultipointEqualsCountTest, EmptyInput)
{
  using T = TypeParam;
  using P = vec_2d<T>;
  CUSPATIAL_RUN_TEST(this->run_single,
                     std::initializer_list<std::initializer_list<P>>{},
                     std::initializer_list<std::initializer_list<P>>{},
                     {});
}

TYPED_TEST(PairwiseMultipointEqualsCountTest, ExampleOne)
{
  CUSPATIAL_RUN_TEST(this->run_single, {{{0, 0}}}, {{{0, 0}, {1, 1}, {2, 2}, {3, 3}}}, {1});
}

TYPED_TEST(PairwiseMultipointEqualsCountTest, ExampleTwo)
{
  CUSPATIAL_RUN_TEST(this->run_single, {{{0, 0}, {1, 1}, {2, 2}, {3, 3}}}, {{{0, 0}}}, {1});
}

TYPED_TEST(PairwiseMultipointEqualsCountTest, ExampleThree)
{
  CUSPATIAL_RUN_TEST(this->run_single,
                     {{{3, 3}, {3, 3}, {0, 0}}, {{0, 0}, {1, 1}, {2, 2}}, {{0, 0}}},
                     {{{0, 0}, {2, 2}, {1, 1}}, {{2, 2}, {0, 0}, {1, 1}}, {{1, 1}}},
                     {1, 3, 0});
}

TYPED_TEST(PairwiseMultipointEqualsCountTest, OneOneEqual)
{
  CUSPATIAL_RUN_TEST(this->run_single, {{{0, 0}}}, {{{0, 0}}}, {1});
}

TYPED_TEST(PairwiseMultipointEqualsCountTest, OneOneNotEqual)
{
  CUSPATIAL_RUN_TEST(this->run_single, {{{0, 0}}}, {{{1, 0}}}, {0});
}

TYPED_TEST(PairwiseMultipointEqualsCountTest, OnePairWithTwoEachEqual)
{
  CUSPATIAL_RUN_TEST(this->run_single, {{{0, 0}, {1, 1}}}, {{{1, 1}, {0, 0}}}, {2});
}

TYPED_TEST(PairwiseMultipointEqualsCountTest, OnePairithTwoNotEqual)
{
  CUSPATIAL_RUN_TEST(this->run_single, {{{0, 0}, {2, 1}}}, {{{1, 1}, {0, 0}}}, {1});
}

TYPED_TEST(PairwiseMultipointEqualsCountTest, OnePairThreeOneEqual)
{
  CUSPATIAL_RUN_TEST(this->run_single, {{{0, 0}, {1, 1}, {2, 2}}}, {{{1, 1}, {1, 1}, {1, 1}}}, {1});
}

TYPED_TEST(PairwiseMultipointEqualsCountTest, OnePairFourOneEqual)
{
  CUSPATIAL_RUN_TEST(
    this->run_single, {{{0, 0}, {1, 1}, {1, 1}, {2, 2}}}, {{{1, 1}, {1, 1}, {1, 1}}}, {2});
}

TYPED_TEST(PairwiseMultipointEqualsCountTest, OnePair)
{
  CUSPATIAL_RUN_TEST(this->run_single, {{{0, 0}, {1, 1}, {2, 2}}}, {{{-1, -1}}}, {0});
}

TYPED_TEST(PairwiseMultipointEqualsCountTest, OneThreeEqual)
{
  CUSPATIAL_RUN_TEST(this->run_single, {{{1, 1}}}, {{{0, 0}, {1, 1}, {0, 0}}}, {1});
}

TYPED_TEST(PairwiseMultipointEqualsCountTest, OneThreeNotEqual)
{
  CUSPATIAL_RUN_TEST(this->run_single, {{{1, 1}}}, {{{0, 0}, {0, 0}, {1, 1}}}, {1});
}

TYPED_TEST(PairwiseMultipointEqualsCountTest, ThreeThreeEqualMiddle)
{
  CUSPATIAL_RUN_TEST(
    this->run_single, {{{0, 0}, {1, 1}, {2, 2}}}, {{{-1, -1}, {1, 1}, {-1, -1}}}, {1});
}

TYPED_TEST(PairwiseMultipointEqualsCountTest, ThreeThreeNotEqualMiddle)
{
  CUSPATIAL_RUN_TEST(
    this->run_single, {{{0, 0}, {1, 1}, {2, 2}}}, {{{0, 0}, {-1, -1}, {2, 2}}}, {2});
}

TYPED_TEST(PairwiseMultipointEqualsCountTest, ThreeThreeNeedRhsMultipoints)
{
  CUSPATIAL_RUN_TEST(this->run_single,
                     {
                       {{0, 0}},
                       {{1, 1}},
                       {{2, 2}},
                     },
                     {{{0, 0}, {1, 1}}, {{2, 2}, {3, 3}}, {{0, 0}, {1, 1}}},
                     {1, 0, 0});
}
