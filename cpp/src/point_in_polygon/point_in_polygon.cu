#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020-2023, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cuspatial/error.hpp>
#include <cuspatial/geometry/vec_2d.hpp>
#include <cuspatial/iterator_factory.cuh>
#include <cuspatial/point_in_polygon.cuh>
#include <cuspatial/range/multipoint_range.cuh>
#include <cuspatial/range/multipolygon_range.cuh>

#include <cudf/column/column.hpp>
#include <cudf/column/column_factories.hpp>
#include <cudf/column/column_view.hpp>
#include <cudf/utilities/type_dispatcher.hpp>

#include <rmm/cuda_stream_view.hpp>
#include <rmm/mr/device/device_memory_resource.hpp>

#include <memory>
#include <type_traits>

namespace {

struct point_in_polygon_functor {
  template <typename T>
  static constexpr bool is_supported()
  {
    return std::is_floating_point<T>::value;
  }

  template <typename T, std::enable_if_t<!is_supported<T>()>* = nullptr, typename... Args>
  std::unique_ptr<cudf::column> operator()(Args&&...)
  {
    CUSPATIAL_FAIL("Non-floating point operation is not supported");
  }

  template <typename T, std::enable_if_t<is_supported<T>()>* = nullptr>
  std::unique_ptr<cudf::column> operator()(cudf::column_view const& test_points_x,
                                           cudf::column_view const& test_points_y,
                                           cudf::column_view const& poly_offsets,
                                           cudf::column_view const& poly_ring_offsets,
                                           cudf::column_view const& poly_points_x,
                                           cudf::column_view const& poly_points_y,
                                           bool pairwise,
                                           rmm::cuda_stream_view stream,
                                           rmm::mr::device_memory_resource* mr)
  {
    auto size = test_points_x.size();
    auto tid  = pairwise ? cudf::type_to_id<uint8_t>() : cudf::type_to_id<int32_t>();
    auto type = cudf::data_type{tid};
    auto results =
      cudf::make_fixed_width_column(type, size, cudf::mask_state::UNALLOCATED, stream, mr);

    if (results->size() == 0) { return results; }

    auto points_begin =
      cuspatial::make_vec_2d_iterator(test_points_x.begin<T>(), test_points_y.begin<T>());
    auto polygon_offsets_begin = poly_offsets.begin<cudf::size_type>();
    auto ring_offsets_begin    = poly_ring_offsets.begin<cudf::size_type>();
    auto polygon_points_begin =
      cuspatial::make_vec_2d_iterator(poly_points_x.begin<T>(), poly_points_y.begin<T>());

    auto multipoints_range =
      make_multipoint_range(size, thrust::make_counting_iterator(0), size, points_begin);

    auto polygon_size       = poly_offsets.size() - 1;
    auto multipolygon_range = make_multipolygon_range(polygon_size,
                                                      thrust::make_counting_iterator(0),
                                                      polygon_size,
                                                      polygon_offsets_begin,
                                                      poly_ring_offsets.size() - 1,
                                                      ring_offsets_begin,
                                                      poly_points_x.size(),
                                                      polygon_points_begin);

    if (pairwise) {
      auto results_begin = results->mutable_view().begin<uint8_t>();
      cuspatial::pairwise_point_in_polygon(
        multipoints_range, multipolygon_range, results_begin, stream);
    } else {
      auto results_begin = results->mutable_view().begin<int32_t>();
      cuspatial::point_in_polygon(multipoints_range, multipolygon_range, results_begin, stream);
    }

    return results;
  }
};
}  // anonymous namespace

namespace cuspatial {

namespace detail {

std::unique_ptr<cudf::column> point_in_polygon(cudf::column_view const& test_points_x,
                                               cudf::column_view const& test_points_y,
                                               cudf::column_view const& poly_offsets,
                                               cudf::column_view const& poly_ring_offsets,
                                               cudf::column_view const& poly_points_x,
                                               cudf::column_view const& poly_points_y,
                                               bool pairwise,
                                               rmm::cuda_stream_view stream,
                                               rmm::mr::device_memory_resource* mr)
{
  CUSPATIAL_EXPECTS(
    test_points_x.size() == test_points_y.size() and poly_points_x.size() == poly_points_y.size(),
    "All points must have both x and y values");

  CUSPATIAL_EXPECTS(test_points_x.type() == test_points_y.type() and
                      test_points_x.type() == poly_points_x.type() and
                      test_points_x.type() == poly_points_y.type(),
                    "All points much have the same type for both x and y");

  CUSPATIAL_EXPECTS(not test_points_x.has_nulls() && not test_points_y.has_nulls(),
                    "Test points must not contain nulls");

  CUSPATIAL_EXPECTS(not poly_points_x.has_nulls() && not poly_points_y.has_nulls(),
                    "Polygon points must not contain nulls");

  if (pairwise) {
    CUSPATIAL_EXPECTS(test_points_x.size() == std::max(poly_offsets.size() - 1, 0),
                      "Must pass in the same number of points as polygons.");
  }

  return cudf::type_dispatcher(test_points_x.type(),
                               point_in_polygon_functor(),
                               test_points_x,
                               test_points_y,
                               poly_offsets,
                               poly_ring_offsets,
                               poly_points_x,
                               poly_points_y,
                               pairwise,
                               stream,
                               mr);
}

}  // namespace detail

std::unique_ptr<cudf::column> point_in_polygon(cudf::column_view const& test_points_x,
                                               cudf::column_view const& test_points_y,
                                               cudf::column_view const& poly_offsets,
                                               cudf::column_view const& poly_ring_offsets,
                                               cudf::column_view const& poly_points_x,
                                               cudf::column_view const& poly_points_y,
                                               rmm::mr::device_memory_resource* mr)
{
  return cuspatial::detail::point_in_polygon(test_points_x,
                                             test_points_y,
                                             poly_offsets,
                                             poly_ring_offsets,
                                             poly_points_x,
                                             poly_points_y,
                                             false,
                                             rmm::cuda_stream_default,
                                             mr);
}

std::unique_ptr<cudf::column> pairwise_point_in_polygon(cudf::column_view const& test_points_x,
                                                        cudf::column_view const& test_points_y,
                                                        cudf::column_view const& poly_offsets,
                                                        cudf::column_view const& poly_ring_offsets,
                                                        cudf::column_view const& poly_points_x,
                                                        cudf::column_view const& poly_points_y,
                                                        rmm::mr::device_memory_resource* mr)
{
  return cuspatial::detail::point_in_polygon(test_points_x,
                                             test_points_y,
                                             poly_offsets,
                                             poly_ring_offsets,
                                             poly_points_x,
                                             poly_points_y,
                                             true,
                                             rmm::cuda_stream_default,
                                             mr);
}

}  // namespace cuspatial
