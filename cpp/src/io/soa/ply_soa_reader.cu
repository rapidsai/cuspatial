#include <stdio.h>
#include <string.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/device_ptr.h>
#include <rmm/rmm.h>
#include <utilities/error_utils.hpp>
#include <cuspatial/soa_readers.hpp>
#include <cuspatial/shared_util.h>

namespace cuSpatial
{
	/**
	 * @Brief read poygon data from file in SoA format; data type of vertices is fixed to double (GDF_FLOAT64)

	 * @param[in] ply_fn: polygon data file name

	 * @param[out] ply_fpos: pointer/array to index polygons, i.e., prefix-sum of #of rings of all polygons

	 * @param[out] ply_rpos: pointer/array to index rings, i.e., prefix-sum of #of vertices of all rings

	 * @param[out] ply_x: pointer/array of x coordiantes of concatenated polygons

	 * @param[out] ply_y: pointer/array of x coordiantes of concatenated polygons
	*
	*/	
	
	void read_ply_soa(const char *poly_fn,gdf_column& ply_fpos, gdf_column& ply_rpos,
                                   gdf_column& ply_x,gdf_column& ply_y)
	{
	        struct PolyMeta<double> pm;
	        int num_p=read_polygon_soa<double>(poly_fn,pm);
	        if(num_p<=0) return;
                
  		ply_fpos.dtype=GDF_INT32;
  		ply_fpos.col_name=(char *)malloc(strlen("f_pos")+ 1);
		strcpy(ply_fpos.col_name,"f_pos");
		ply_fpos.data=NULL;
		RMM_TRY( RMM_ALLOC(&ply_fpos.data, pm.num_f * sizeof(uint), 0) );
		hipMemcpy(ply_fpos.data, pm.p_f_len,pm.num_f * sizeof(uint) , hipMemcpyHostToDevice);
		thrust::device_ptr<uint> d_pfp_ptr=thrust::device_pointer_cast((uint *)ply_fpos.data);
		//prefix-sum: len to pos
		thrust::inclusive_scan(d_pfp_ptr,d_pfp_ptr+pm.num_f,d_pfp_ptr);
		ply_fpos.size=pm.num_f;
		ply_fpos.valid=nullptr;
		ply_fpos.null_count=0;
		delete[] pm.p_f_len;

 		ply_rpos.dtype=GDF_INT32;
 		ply_rpos.col_name=(char *)malloc(strlen("r_pos")+ 1);
		strcpy(ply_rpos.col_name,"r_pos");
		ply_rpos.data=NULL;
		RMM_TRY( RMM_ALLOC(&ply_rpos.data, pm.num_r * sizeof(uint), 0) );
		hipMemcpy(ply_rpos.data, pm.p_r_len,pm.num_r * sizeof(uint) , hipMemcpyHostToDevice);
		thrust::device_ptr<uint> d_prp_ptr=thrust::device_pointer_cast((uint *)ply_rpos.data);
		//prefix-sum: len to pos
		thrust::inclusive_scan(d_prp_ptr,d_prp_ptr+pm.num_r,d_prp_ptr);
		ply_rpos.size=pm.num_r;
		ply_rpos.valid=nullptr;
		ply_rpos.null_count=0;
		delete[] pm.p_r_len;

 		ply_x.dtype= GDF_FLOAT64;
 		ply_x.col_name=(char *)malloc(strlen("x")+ 1);
		strcpy(ply_x.col_name,"x");
		RMM_TRY( RMM_ALLOC(&ply_x.data, pm.num_v * sizeof(double), 0) );
		hipMemcpy(ply_x.data, pm.p_x,pm.num_v * sizeof(double) , hipMemcpyHostToDevice);		
		ply_x.size=pm.num_v;
		ply_x.valid=nullptr;
		ply_x.null_count=0;		
		delete[] pm.p_x;

 		ply_y.dtype= GDF_FLOAT64;
 		ply_y.col_name=(char *)malloc(strlen("y")+ 1);
		strcpy(ply_y.col_name,"y");
		ply_y.data=NULL;
		RMM_TRY( RMM_ALLOC(&ply_y.data, pm.num_v * sizeof(double), 0) );
		hipMemcpy(ply_y.data, pm.p_y,pm.num_v * sizeof(double) , hipMemcpyHostToDevice);		
		ply_y.size=pm.num_v;
		ply_y.valid=nullptr;
		ply_y.null_count=0;
		delete[] pm.p_y;
		
		delete[] pm.p_g_len;
	}
}