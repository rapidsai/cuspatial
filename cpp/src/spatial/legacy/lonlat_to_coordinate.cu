#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/utilities/legacy/type_dispatcher.hpp>
#include <utilities/legacy/cuda_utils.hpp>
#include <type_traits>
#include <utility>
#include <thrust/device_vector.h>

#include <utility/legacy/utility.hpp>
#include <cuspatial/legacy/coordinate_transform.hpp>
#include <cuspatial/error.hpp>

#include <cudf/legacy/column.hpp>

namespace {

/**
 * @brief CUDA kernel for approximately transforming lon/lat to x/y (in km) relative to a camera origin
 *
 *Note: points in the third quadrant relative camera will be transformed into points in the first quadrant - use with caution
 */

template <typename T>
__global__ void coord_trans_kernel(gdf_size_type loc_size,
                                   double cam_lon, double cam_lat,
                                   const T* const __restrict__ in_lon,
                                   const T* const __restrict__ in_lat,
                                   T* const __restrict__ out_x,
                                   T* const __restrict__ out_y)
{
    //assuming 1D grid/block config
    uint32_t idx =blockIdx.x*blockDim.x+threadIdx.x;
    if(idx>=loc_size) return;
    out_x[idx]=((cam_lon - in_lon[idx]) * 40000.0 *cos((cam_lat + in_lat[idx]) * M_PI / 360) / 360);
    out_y[idx]=(cam_lat - in_lat[idx]) * 40000.0 / 360;
}

struct ll2coord_functor {
    template <typename T>
    static constexpr bool is_supported()
    {
        return std::is_floating_point<T>::value;
    }

    template <typename T, std::enable_if_t< is_supported<T>() >* = nullptr>
    std::pair<gdf_column,gdf_column> operator()(const gdf_scalar  & cam_lon,const gdf_scalar  & cam_lat,
    	 const gdf_column  & in_lon,const gdf_column  & in_lat)

    {
        gdf_column  out_x, out_y;
        memset(&out_x,0,sizeof(gdf_column));
        memset(&out_y,0,sizeof(gdf_column));

        hipStream_t stream{0};
        T* temp_x{nullptr};
        T* temp_y{nullptr};
        RMM_TRY( RMM_ALLOC(&temp_x, in_lon.size * sizeof(T), stream) );
        RMM_TRY( RMM_ALLOC(&temp_y, in_lat.size * sizeof(T), stream) );

        gdf_column_view_augmented(&out_x, temp_x, nullptr, in_lon.size,
                                in_lon.dtype, 0,
                                gdf_dtype_extra_info{TIME_UNIT_NONE}, "x");
        gdf_column_view_augmented(&out_y, temp_y, nullptr, in_lat.size,
                                in_lat.dtype, 0,
                                gdf_dtype_extra_info{TIME_UNIT_NONE}, "y");

        gdf_size_type min_grid_size = 0, block_size = 0;
        CUDA_TRY( hipOccupancyMaxPotentialBlockSize(&min_grid_size, &block_size, coord_trans_kernel<T>) );
        cudf::util::cuda::grid_config_1d grid{in_lon.size, block_size, 1};

        coord_trans_kernel<T> <<< grid.num_blocks, block_size >>> (in_lon.size,
        	*((double*)(&(cam_lon.data))),*((double*)(&(cam_lat.data))),
   	    	static_cast<T*>(in_lon.data), static_cast<T*>(in_lat.data),
   	    	temp_x, temp_y );
        CUDA_TRY( hipDeviceSynchronize() );

 	return std::make_pair(out_x,out_y);
    }

    template <typename T, std::enable_if_t< !is_supported<T>() >* = nullptr>
    std::pair<gdf_column,gdf_column> operator()(const gdf_scalar  & cam_lon,const gdf_scalar  & cam_lat,
    	const gdf_column  & in_lon,const gdf_column  & in_lat)
    {
        CUSPATIAL_FAIL("Non-floating point operation is not supported");
    }
};

} // namespace anonymous

namespace cuspatial {

/**
 * @brief transforming in_lon/in_lat (lon/lat defined in coord_2d) to out_x/out_y relative to a camera origiin
 * see coordinate_transform.hpp
*/

std::pair<gdf_column,gdf_column> lonlat_to_coord(const gdf_scalar& cam_lon, const gdf_scalar& cam_lat,
	const gdf_column& in_lon, const gdf_column  & in_lat)

{

    double cx=*((double*)(&(cam_lon.data)));
    double cy=*((double*)(&(cam_lat.data)));
    CUSPATIAL_EXPECTS(cx >=-180 && cx <=180 && cy >=-90 && cy <=90,
    	"camera origin must have valid lat/lon values [-180,-90,180,90]");
    CUSPATIAL_EXPECTS(in_lon.data != nullptr &&in_lat.data!=nullptr, "input point cannot be empty");
    CUSPATIAL_EXPECTS(in_lon.size == in_lat.size, "input x and y arrays must have the same length");

    //future versions might allow in_(x/y) have null_count>0, which might be useful for taking query results as inputs
    CUSPATIAL_EXPECTS(in_lon.null_count == 0 && in_lat.null_count == 0, "this version does not support point in_lon/in_lat contains nulls");

    auto res=cudf::type_dispatcher(in_lon.dtype, ll2coord_functor(), cam_lon,cam_lat,in_lon,in_lat);

    return res;

  }//lonlat_to_coord

}// namespace cuspatial
