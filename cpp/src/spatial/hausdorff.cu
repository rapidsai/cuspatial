#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required point_b_y applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "detail/cartesian_product_group_index_iterator.cuh"
#include "detail/hausdorff.cuh"

#include <utility/scatter_output_iterator.cuh>
#include <utility/size_from_offsets.cuh>

#include <cuspatial/error.hpp>

#include <cudf/column/column_device_view.cuh>
#include <cudf/column/column_factories.hpp>
#include <cudf/types.hpp>
#include <cudf/utilities/type_dispatcher.hpp>

#include <rmm/cuda_stream_view.hpp>
#include <rmm/device_uvector.hpp>
#include <rmm/exec_policy.hpp>

#include <thrust/binary_search.h>
#include <thrust/iterator/transform_iterator.h>

#include <limits>
#include <memory>

namespace cuspatial {
namespace detail {
namespace {

template <typename T>
struct hausdorff_accumulator_factory {
  cudf::column_device_view const xs;
  cudf::column_device_view const ys;

  hausdorff_acc<T> inline __device__ operator()(cartesian_product_group_index const& idx)
  {
    auto const a_idx = idx.group_a.offset + idx.element_a_idx;
    auto const b_idx = idx.group_b.offset + idx.element_b_idx;

    auto const distance = hypot(xs.element<T>(b_idx) - xs.element<T>(a_idx),
                                ys.element<T>(b_idx) - ys.element<T>(a_idx));

    return hausdorff_acc<T>{b_idx, b_idx, distance, distance, 0};
  }
};

struct hausdorff_functor {
  template <typename T, typename... Args>
  std::enable_if_t<not std::is_floating_point<T>::value, std::unique_ptr<cudf::column>> operator()(
    Args&&...)
  {
    CUSPATIAL_FAIL("Non-floating point operation is not supported");
  }

  template <typename T>
  std::enable_if_t<std::is_floating_point<T>::value, std::unique_ptr<cudf::column>> operator()(
    cudf::column_view const& xs,
    cudf::column_view const& ys,
    cudf::column_view const& space_offsets,
    rmm::cuda_stream_view stream,
    rmm::mr::device_memory_resource* mr)
  {
    auto const num_points  = static_cast<uint32_t>(xs.size());
    auto const num_spaces  = static_cast<uint32_t>(space_offsets.size());
    auto const num_results = static_cast<uint64_t>(num_spaces) * static_cast<uint64_t>(num_spaces);

    CUSPATIAL_EXPECTS(num_results < std::numeric_limits<cudf::size_type>::max(),
                      "Matrix of spaces must be less than 2^31");

    if (num_results == 0) {
      return cudf::make_empty_column(cudf::data_type{cudf::type_to_id<T>()});
    }

    // ===== Make Hausdorff Accumulator ============================================================

    auto gcp_iter = make_cartesian_product_group_index_iterator(
      num_points, num_spaces, space_offsets.begin<uint32_t>());

    auto d_xs = cudf::column_device_view::create(xs);
    auto d_ys = cudf::column_device_view::create(ys);

    auto hausdorff_acc_iter =
      thrust::make_transform_iterator(gcp_iter, hausdorff_accumulator_factory<T>{*d_xs, *d_ys});

    // ===== Materialize ===========================================================================

    auto result = cudf::make_fixed_width_column(cudf::data_type{cudf::type_to_id<T>()},
                                                static_cast<cudf::size_type>(num_results),
                                                cudf::mask_state::UNALLOCATED,
                                                stream,
                                                mr);

    auto result_temp = rmm::device_uvector<hausdorff_acc<T>>(num_results, stream);

    auto scatter_map = thrust::make_transform_iterator(
      gcp_iter, [num_spaces] __device__(cartesian_product_group_index const& idx) {
        // the given output is only a "result" if it is the last output for a given pair-of-spaces
        bool const is_result = idx.element_a_idx + 1 == idx.group_a.size &&  //
                               idx.element_b_idx + 1 == idx.group_b.size;

        if (not is_result) { return static_cast<uint32_t>(-1); }

        // the destination for the result is determined per- pair-of-spaces
        return idx.group_b.idx * num_spaces + idx.group_a.idx;
      });

    auto scatter_out = make_scatter_output_iterator(result_temp.begin(), scatter_map);

    auto gpc_key_iter = thrust::make_transform_iterator(
      gcp_iter, [] __device__(cartesian_product_group_index const& idx) {
        return thrust::make_pair(idx.group_a.idx, idx.group_b.idx);
      });

    // the following output iterator and `inclusive_scan_by_key` could be replaced by a
    // reduce_by_key, if it supported non-commutative operators.

    auto const num_cartesian =
      static_cast<uint64_t>(num_points) * static_cast<uint64_t>(num_points);

    //
    // `thrust::inclusive_scan_by_key` causes an out-of-memory
    // error on input sizes close to (but not exactly) INT_MAX.
    //
    // Doing the reduction in chunks is a temporary workaround until this is fixed.
    //
    // The magic number between OOM vs. no OOM is somewhere between:
    //                                             (1uL << 31uL) - 2048uL;
    auto const magic_inclusive_scan_by_key_limit = (1uL << 31uL) - 4096uL;

    for (auto itr = gpc_key_iter, end = gpc_key_iter + num_cartesian; itr < end;) {
      auto const len = static_cast<uint32_t>(std::min(
        static_cast<uint64_t>(thrust::distance(itr, end)), magic_inclusive_scan_by_key_limit));

      thrust::inclusive_scan_by_key(rmm::exec_policy(stream),
                                    itr,
                                    itr + len,
                                    hausdorff_acc_iter,
                                    scatter_out,
                                    thrust::equal_to<thrust::pair<uint32_t, uint32_t>>());
      thrust::advance(itr, len);
    }

    thrust::transform(rmm::exec_policy(stream),
                      result_temp.begin(),
                      result_temp.end(),
                      result->mutable_view().begin<T>(),
                      [] __device__(hausdorff_acc<T> const& a) { return static_cast<T>(a); });

    return result;
  }
};

}  // namespace
}  // namespace detail

std::unique_ptr<cudf::column> directed_hausdorff_distance(cudf::column_view const& xs,
                                                          cudf::column_view const& ys,
                                                          cudf::column_view const& space_offsets,
                                                          rmm::mr::device_memory_resource* mr)
{
  CUSPATIAL_EXPECTS(xs.type() == ys.type(), "Inputs `xs` and `ys` must have same type.");
  CUSPATIAL_EXPECTS(xs.size() == ys.size(), "Inputs `xs` and `ys` must have same length.");

  CUSPATIAL_EXPECTS(not xs.has_nulls() and not ys.has_nulls() and not space_offsets.has_nulls(),
                    "Inputs must not have nulls.");

  CUSPATIAL_EXPECTS(xs.size() >= space_offsets.size(),
                    "At least one point is required for each space");

  return cudf::type_dispatcher(
    xs.type(), detail::hausdorff_functor(), xs, ys, space_offsets, rmm::cuda_stream_default, mr);
}

}  // namespace cuspatial
