#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/column/column.hpp>
#include <cudf/column/column_factories.hpp>
#include <cudf/column/column_view.hpp>
#include <cudf/detail/utilities/cuda.cuh>
#include <memory>

#include <cuspatial/error.hpp>
#include <type_traits>

#include "cudf/utilities/type_dispatcher.hpp"
#include "rmm/mr/device/device_memory_resource.hpp"

namespace {

template <typename T>
__global__ void point_in_polygon_kernel(cudf::size_type num_test_points,
                                        const T* const __restrict__ test_points_x,
                                        const T* const __restrict__ test_points_y,
                                        cudf::size_type num_polys,
                                        const cudf::size_type* const __restrict__ poly_offsets,
                                        cudf::size_type num_rings,
                                        const cudf::size_type* const __restrict__ poly_ring_offsets,
                                        cudf::size_type num_points,
                                        const T* const __restrict__ poly_points_x,
                                        const T* const __restrict__ poly_points_y,
                                        int32_t* const __restrict__ result)
{
  auto idx = blockIdx.x * blockDim.x + threadIdx.x;

  if (idx > num_test_points) { return; }

  int32_t hit_mask = 0;

  T x = test_points_x[idx];
  T y = test_points_y[idx];

  // for each polygon
  for (auto poly_idx = 0; poly_idx < num_polys; poly_idx++) {
    auto poly_idx_next = poly_idx + 1;
    auto poly_begin    = poly_offsets[poly_idx];
    auto poly_end      = (poly_idx_next < num_polys) ? poly_offsets[poly_idx_next] : num_rings;

    bool point_is_within = false;

    // for each ring
    for (auto ring_idx = poly_begin; ring_idx < poly_end; ring_idx++) {
      auto ring_idx_next = ring_idx + 1;
      auto ring_begin    = poly_ring_offsets[ring_idx];
      auto ring_end = (ring_idx_next < num_rings) ? poly_ring_offsets[ring_idx_next] : num_points;

      // for each line segment
      for (auto point_idx = ring_begin; point_idx < ring_end - 1; point_idx++) {
        T ax = poly_points_x[point_idx];
        T ay = poly_points_y[point_idx];
        T bx = poly_points_x[point_idx + 1];
        T by = poly_points_y[point_idx + 1];

        bool y_between_ay_by = ay <= y && y < by;  // is y in range [ay, by) when ay < by?
        bool y_between_by_ay = by <= y && y < ay;  // is y in range [by, ay) when by < ay?
        bool y_in_bounds     = y_between_ay_by || y_between_by_ay;  // is y in range [by, ay]?
        T run                = bx - ax;
        T rise               = by - ay;
        T rise_to_point      = y - ay;

        if (y_in_bounds && x < (run / rise) * rise_to_point + ax) {
          point_is_within = not point_is_within;
        }
      }
    }

    hit_mask |= point_is_within << poly_idx;
  }

  result[idx] = hit_mask;
}

struct point_in_polygon_functor {
  template <typename T>
  static constexpr bool is_supported()
  {
    return std::is_floating_point<T>::value;
  }

  template <typename T, std::enable_if_t<!is_supported<T>()>* = nullptr, typename... Args>
  std::unique_ptr<cudf::column> operator()(Args&&...)
  {
    CUSPATIAL_FAIL("Non-floating point operation is not supported");
  }

  template <typename T, std::enable_if_t<is_supported<T>()>* = nullptr>
  std::unique_ptr<cudf::column> operator()(cudf::column_view const& test_points_x,
                                           cudf::column_view const& test_points_y,
                                           cudf::column_view const& poly_offsets,
                                           cudf::column_view const& poly_ring_offsets,
                                           cudf::column_view const& poly_points_x,
                                           cudf::column_view const& poly_points_y,
                                           rmm::mr::device_memory_resource* mr,
                                           hipStream_t stream)
  {
    auto size = test_points_x.size();
    auto tid  = cudf::type_to_id<int32_t>();
    auto type = cudf::data_type{tid};
    auto results =
      cudf::make_fixed_width_column(type, size, cudf::mask_state::UNALLOCATED, stream, mr);

    if (results->size() == 0) { return results; }

    constexpr cudf::size_type block_size = 256;

    cudf::detail::grid_1d grid{results->size(), block_size, 1};

    auto kernel = point_in_polygon_kernel<T>;

    kernel<<<grid.num_blocks, block_size, 0, stream>>>(test_points_x.size(),
                                                       test_points_x.begin<T>(),
                                                       test_points_y.begin<T>(),
                                                       poly_offsets.size(),
                                                       poly_offsets.begin<cudf::size_type>(),
                                                       poly_ring_offsets.size(),
                                                       poly_ring_offsets.begin<cudf::size_type>(),
                                                       poly_points_x.size(),
                                                       poly_points_x.begin<T>(),
                                                       poly_points_y.begin<T>(),
                                                       results->mutable_view().begin<int32_t>());

    return results;
  }
};

}  // anonymous namespace

namespace cuspatial {

namespace detail {

std::unique_ptr<cudf::column> point_in_polygon(cudf::column_view const& test_points_x,
                                               cudf::column_view const& test_points_y,
                                               cudf::column_view const& poly_offsets,
                                               cudf::column_view const& poly_ring_offsets,
                                               cudf::column_view const& poly_points_x,
                                               cudf::column_view const& poly_points_y,
                                               rmm::mr::device_memory_resource* mr,
                                               hipStream_t stream)
{
  return cudf::type_dispatcher(test_points_x.type(),
                               point_in_polygon_functor(),
                               test_points_x,
                               test_points_y,
                               poly_offsets,
                               poly_ring_offsets,
                               poly_points_x,
                               poly_points_y,
                               mr,
                               stream);
}

}  // namespace detail

std::unique_ptr<cudf::column> point_in_polygon(cudf::column_view const& test_points_x,
                                               cudf::column_view const& test_points_y,
                                               cudf::column_view const& poly_offsets,
                                               cudf::column_view const& poly_ring_offsets,
                                               cudf::column_view const& poly_points_x,
                                               cudf::column_view const& poly_points_y,
                                               rmm::mr::device_memory_resource* mr)
{
  CUSPATIAL_EXPECTS(
    test_points_x.size() == test_points_y.size() and poly_points_x.size() == poly_points_y.size(),
    "All points must have both x and y values");

  CUSPATIAL_EXPECTS(test_points_x.type() == test_points_y.type() and
                      test_points_x.type() == poly_points_x.type() and
                      test_points_x.type() == poly_points_y.type(),
                    "All points much have the same type for both x and y");

  CUSPATIAL_EXPECTS(not test_points_x.has_nulls() && not test_points_y.has_nulls(),
                    "Test points must not contain nulls");

  CUSPATIAL_EXPECTS(not poly_points_x.has_nulls() && not poly_points_y.has_nulls(),
                    "Polygon points must not contain nulls");

  CUSPATIAL_EXPECTS(poly_offsets.size() <= std::numeric_limits<int32_t>::digits,
                    "Number of polygons cannot exceed 31");

  CUSPATIAL_EXPECTS(poly_ring_offsets.size() >= poly_offsets.size(),
                    "Each polygon must have at least one ring");

  CUSPATIAL_EXPECTS(poly_points_x.size() >= poly_offsets.size() * 4,
                    "Each ring must have at least four vertices");

  return cuspatial::detail::point_in_polygon(test_points_x,
                                             test_points_y,
                                             poly_offsets,
                                             poly_ring_offsets,
                                             poly_points_x,
                                             poly_points_y,
                                             mr,
                                             0);
}

}  // namespace cuspatial
