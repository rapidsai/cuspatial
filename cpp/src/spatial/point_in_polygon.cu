#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/utilities/legacy/type_dispatcher.hpp>
#include <utilities/legacy/cuda_utils.hpp>
#include <type_traits>
#include <utility/utility.hpp>
#include <cuspatial/point_in_polygon.hpp>

#include <cudf/legacy/column.hpp>

namespace {

template <typename T>
__global__ void pip_kernel(gdf_size_type pnt_size,const T* const __restrict__ pnt_x,const T* const __restrict__ pnt_y,
        gdf_size_type ply_size,const uint32_t* const __restrict__ ply_fpos,const uint32_t* const __restrict__ ply_rpos,
        const T* const __restrict__ ply_x,const T* const __restrict__ ply_y,
        uint32_t* const __restrict__ res_bm)
{
    uint32_t mask=0;
    //assuming 1D grid/block config
    uint32_t idx =blockIdx.x*blockDim.x+threadIdx.x;
    if(idx>=pnt_size) return;

    T x = pnt_x[idx];
    T y = pnt_y[idx];
    for (uint32_t j = 0; j < ply_size; j++) //for each polygon
    {
       uint32_t r_f = (0 == j) ? 0 : ply_fpos[j-1];
       uint32_t r_t=ply_fpos[j];
       bool in_polygon = false;
       for (uint32_t k = r_f; k < r_t; k++) //for each ring
       {
           uint32_t m = (k==0)?0:ply_rpos[k-1];
           for (;m < ply_rpos[k]-1; m++) //for each line segment
           {
              T x0, x1, y0, y1;
              x0 = ply_x[m];
              y0 = ply_y[m];
              x1 = ply_x[m+1];
              y1 = ply_y[m+1];

              if ((((y0 <= y) && (y < y1)) ||
                   ((y1 <= y) && (y < y0))) &&
                       (x < (x1 - x0) * (y - y0) / (y1 - y0) + x0))
                 in_polygon = !in_polygon;
            }
      }
      if(in_polygon)
      	mask|=(0x01<<j);
   }
   res_bm[idx]=mask;
}

struct pip_functor {
    template <typename T>
    static constexpr bool is_supported()
    {
        return std::is_floating_point<T>::value;
    }

    template <typename T, std::enable_if_t< is_supported<T>() >* = nullptr>
    gdf_column operator()(gdf_column const & pnt_x,gdf_column const & pnt_y,
 			  gdf_column const & ply_fpos,gdf_column const & ply_rpos,
			  gdf_column const & ply_x,gdf_column const & ply_y)
    {
        gdf_column res_bm;
        uint32_t* temp_bitmap{nullptr};

        hipStream_t stream{0};
        RMM_TRY( RMM_ALLOC(&temp_bitmap, pnt_y.size * sizeof(uint32_t), stream) );

        gdf_size_type min_grid_size = 0, block_size = 0;
        CUDA_TRY( hipOccupancyMaxPotentialBlockSize(&min_grid_size, &block_size, pip_kernel<T>) );
        cudf::util::cuda::grid_config_1d grid{pnt_y.size, block_size, 1};

        pip_kernel<T> <<< grid.num_blocks, block_size >>> (pnt_x.size,
               	static_cast<T*>(pnt_x.data), static_cast<T*>(pnt_y.data),
        	ply_fpos.size,static_cast<uint32_t*>(ply_fpos.data),static_cast<uint32_t*>(ply_rpos.data),
        	static_cast<T*>(ply_x.data), static_cast<T*>(ply_y.data),
                temp_bitmap);
        CUDA_TRY( hipDeviceSynchronize() );

        gdf_column_view(&res_bm, temp_bitmap, nullptr, pnt_x.size, GDF_INT32);

        return res_bm;
    }

    template <typename T, std::enable_if_t< !is_supported<T>() >* = nullptr>
    gdf_column operator()(gdf_column const & pnt_x,gdf_column const & pnt_y,
 			  gdf_column const & ply_fpos,gdf_column const & ply_rpos,
			  gdf_column const & ply_x,gdf_column const & ply_y)

    {
        CUDF_FAIL("Non-floating point operation is not supported");
    }
};

} // namespace anonymous

namespace cuspatial {

/*
 * Point-in-Polygon (PIP) tests among a column of points and a column of
 * polygons. See point_in_polygon.hpp
 */
gdf_column point_in_polygon_bitmap(const gdf_column& points_x,
                                   const gdf_column& points_y,
                                   const gdf_column& poly_fpos,
                                   const gdf_column& poly_rpos,
                                   const gdf_column& poly_x,
                                   const gdf_column& poly_y)
{

    CUDF_EXPECTS(points_y.data != nullptr && points_x.data != nullptr, "query point data cannot be empty");
    CUDF_EXPECTS(points_y.dtype == points_x.dtype, "polygon vertex and point temp_bitmap type mismatch for x array ");

    //future versions might allow pnt_(x/y) have null_count>0, which might be useful for taking query results as inputs
    CUDF_EXPECTS(points_x.null_count == 0 && points_y.null_count == 0, "this version does not support points_x/points_y contains nulls");

    CUDF_EXPECTS(poly_fpos.data != nullptr &&poly_rpos.data!=nullptr, "polygon index cannot be empty");
    CUDF_EXPECTS(poly_fpos.size >0 && (size_t)poly_fpos.size<=sizeof(uint32_t)*8, "#polygon of polygons can not exceed bitmap capacity (32 for unsigned int)");
    CUDF_EXPECTS(poly_y.data != nullptr && poly_x.data != nullptr, "polygon temp_bitmap cannot be empty");
    CUDF_EXPECTS(poly_fpos.size <=poly_rpos.size,"#of polygons must be equal or less than # of rings (one polygon has at least one ring");
    CUDF_EXPECTS(poly_y.size == poly_x.size, "polygon vertice sizes mismatch between x/y arrays");
    CUDF_EXPECTS(points_y.size == points_x.size, "query points size mismatch from between x/y arrays");
    CUDF_EXPECTS(poly_y.dtype == poly_x.dtype, "polygon vertex temp_bitmap type mismatch between x/y arrays");
    CUDF_EXPECTS(poly_y.dtype == points_y.dtype, "polygon vertex and point temp_bitmap type mismatch for y array");
    CUDF_EXPECTS(poly_x.null_count == 0 && poly_y.null_count == 0, "polygon should not contain nulls");

    gdf_column res_bm = cudf::type_dispatcher(points_x.dtype, pip_functor(),
                                              points_x, points_y, poly_fpos,
                                              poly_rpos,poly_x,poly_y);

    return res_bm;
  }//point_in_polygon_bitmap

}// namespace cuspatial
