#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/column/column_device_view.cuh>
#include <cudf/column/column_view.hpp>
#include <cudf/copying.hpp>
#include <cudf/detail/copy_if.cuh>

#include <cuspatial/error.hpp>

#include <memory>
#include <type_traits>

namespace {

// Functor to filter out points that are not inside the query window
// This is passed to cudf::detail::copy_if
template <typename T>
struct spatial_window_filter {
  spatial_window_filter(T window_min_x,
                        T window_max_x,
                        T window_min_y,
                        T window_max_y,
                        cudf::column_device_view const& x,
                        cudf::column_device_view const& y)
    : min_x{std::min(window_min_x, window_max_x)},  // support mirrored rectangles
      max_x{std::max(window_min_x, window_max_x)},  // where specified min > max
      min_y{std::min(window_min_y, window_max_y)},
      max_y{std::max(window_min_y, window_max_y)},
      points_x{x},
      points_y{y}
  {
  }

  __device__ inline bool operator()(cudf::size_type i)
  {
    auto x = points_x.element<T>(i);
    auto y = points_y.element<T>(i);
    return x > min_x && x < max_x && y > min_y && y < max_y;
  }

 protected:
  T min_x;
  T max_x;
  T min_y;
  T max_y;
  cudf::column_device_view points_x;
  cudf::column_device_view points_y;
};

// Type-dispatch functor that creates the spatial window filter of the correct type.
// Only floating point types are supported.
struct spatial_window_dispatch {
  template <typename T, std::enable_if_t<std::is_floating_point<T>::value>* = nullptr>
  std::unique_ptr<cudf::table> operator()(double window_min_x,
                                          double window_max_x,
                                          double window_min_y,
                                          double window_max_y,
                                          cudf::column_view const& x,
                                          cudf::column_view const& y,
                                          hipStream_t stream,
                                          rmm::mr::device_memory_resource* mr)
  {
    auto device_x = cudf::column_device_view::create(x, stream);
    auto device_y = cudf::column_device_view::create(y, stream);
    return cudf::detail::copy_if(cudf::table_view{{x, y}},
                                 spatial_window_filter<T>{static_cast<T>(window_min_x),
                                                          static_cast<T>(window_max_x),
                                                          static_cast<T>(window_min_y),
                                                          static_cast<T>(window_max_y),
                                                          *device_x,
                                                          *device_y},
                                 mr,
                                 stream);
  }

  template <typename T,
            std::enable_if_t<not std::is_floating_point<T>::value>* = nullptr,
            typename... Args>
  std::unique_ptr<cudf::table> operator()(Args&&...)
  {
    CUSPATIAL_FAIL("Only floating-point types supported");
  }
};

}  // namespace

namespace cuspatial {

namespace detail {

/*
 * Return all points (x,y) that fall within a query window (x1,y1,x2,y2)
 * see query.hpp
 *
 * Detail version that takes a stream.
 */
std::unique_ptr<cudf::table> points_in_spatial_window(double window_min_x,
                                                      double window_max_x,
                                                      double window_min_y,
                                                      double window_max_y,
                                                      cudf::column_view const& x,
                                                      cudf::column_view const& y,
                                                      hipStream_t stream,
                                                      rmm::mr::device_memory_resource* mr)
{
  CUSPATIAL_EXPECTS(x.type() == y.type(), "Type mismatch between x and y arrays");
  CUSPATIAL_EXPECTS(x.size() == y.size(), "Size mismatch between x and y arrays");

  CUSPATIAL_EXPECTS(not(x.has_nulls() || y.has_nulls()), "NULL point data not supported");

  return cudf::type_dispatcher(x.type(),
                               spatial_window_dispatch(),
                               window_min_x,
                               window_max_x,
                               window_min_y,
                               window_max_y,
                               x,
                               y,
                               stream,
                               mr);
}

}  // namespace detail

/*
 * Return all points (x,y) that fall within a query window (x1,y1,x2,y2)
 * see query.hpp
 */
std::unique_ptr<cudf::table> points_in_spatial_window(double window_min_x,
                                                      double window_max_x,
                                                      double window_min_y,
                                                      double window_max_y,
                                                      cudf::column_view const& x,
                                                      cudf::column_view const& y,
                                                      rmm::mr::device_memory_resource* mr)
{
  return detail::points_in_spatial_window(
    window_min_x, window_max_x, window_min_y, window_max_y, x, y, 0, mr);
}

}  // namespace cuspatial
