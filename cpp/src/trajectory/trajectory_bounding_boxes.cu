#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <thrust/iterator/discard_iterator.h>

#include <cudf/column/column_device_view.cuh>
#include <cudf/column/column_factories.hpp>
#include <cudf/column/column_view.hpp>
#include <cudf/copying.hpp>
#include <cudf/detail/utilities/cuda.cuh>
#include <cudf/table/table.hpp>
#include <cudf/utilities/traits.hpp>
#include <cudf/utilities/type_dispatcher.hpp>

#include <cuspatial/error.hpp>
#include <cuspatial/trajectory.hpp>

namespace cuspatial {
namespace experimental {

namespace {

struct dispatch_element {
  template <typename Element>
  std::enable_if_t<std::is_floating_point<Element>::value, std::unique_ptr<cudf::table>> operator()(
    cudf::size_type num_trajectories,
    cudf::column_view const& object_id,
    cudf::column_view const& x,
    cudf::column_view const& y,
    rmm::mr::device_memory_resource* mr,
    hipStream_t stream)
  {
    auto policy = rmm::exec_policy(stream);

    // Construct output columns
    auto type = cudf::data_type{cudf::type_to_id<Element>()};
    std::vector<std::unique_ptr<cudf::column>> cols{};
    cols.reserve(4);
    // allocate bbox_x1 output column
    cols.push_back(
      cudf::make_numeric_column(type, num_trajectories, cudf::mask_state::UNALLOCATED, stream, mr));
    // allocate bbox_y1 output column
    cols.push_back(
      cudf::make_numeric_column(type, num_trajectories, cudf::mask_state::UNALLOCATED, stream, mr));
    // allocate bbox_x2 output column
    cols.push_back(
      cudf::make_numeric_column(type, num_trajectories, cudf::mask_state::UNALLOCATED, stream, mr));
    // allocate bbox_y2 output column
    cols.push_back(
      cudf::make_numeric_column(type, num_trajectories, cudf::mask_state::UNALLOCATED, stream, mr));

    auto points = thrust::make_zip_iterator(thrust::make_tuple(
      x.begin<Element>(), y.begin<Element>(), x.begin<Element>(), y.begin<Element>()));

    auto bboxes = thrust::make_zip_iterator(
      thrust::make_tuple(cols.at(0)->mutable_view().begin<Element>(),  // bbox_x1
                         cols.at(1)->mutable_view().begin<Element>(),  // bbox_y1
                         cols.at(2)->mutable_view().begin<Element>(),  // bbox_x2
                         cols.at(3)->mutable_view().begin<Element>())  // bbox_y2
    );

    thrust::fill(policy->on(stream),
                 bboxes,
                 bboxes + num_trajectories,
                 thrust::make_tuple(std::numeric_limits<Element>::max(),
                                    std::numeric_limits<Element>::max(),
                                    std::numeric_limits<Element>::min(),
                                    std::numeric_limits<Element>::min()));

    thrust::reduce_by_key(
      policy->on(stream),               // execution policy
      object_id.begin<int32_t>(),       // keys_first
      object_id.end<int32_t>(),         // keys_last
      points,                           // values_first
      thrust::make_discard_iterator(),  // keys_output
      bboxes,                           // values_output
      thrust::equal_to<int32_t>(),      // binary_pred
      [] __device__(auto a, auto b) {   // binary_op
        Element x1, y1, x2, y2, x3, y3, x4, y4;
        thrust::tie(x1, y1, x2, y2) = a;
        thrust::tie(x3, y3, x4, y4) = b;
        return thrust::make_tuple(
          min(min(x1, x2), x3), min(min(y1, y2), y3), max(max(x1, x2), x4), max(max(y1, y2), y4));
      });

    // check for errors
    CHECK_CUDA(stream);

    return std::make_unique<cudf::table>(std::move(cols));
  }

  template <typename Element>
  std::enable_if_t<not std::is_floating_point<Element>::value, std::unique_ptr<cudf::table>>
  operator()(cudf::size_type num_trajectories,
             cudf::column_view const& object_id,
             cudf::column_view const& x,
             cudf::column_view const& y,
             rmm::mr::device_memory_resource* mr,
             hipStream_t stream)
  {
    CUSPATIAL_FAIL("X and Y must be floating point types");
  }
};

}  // namespace

namespace detail {
std::unique_ptr<cudf::table> trajectory_bounding_boxes(cudf::size_type num_trajectories,
                                                       cudf::column_view const& object_id,
                                                       cudf::column_view const& x,
                                                       cudf::column_view const& y,
                                                       rmm::mr::device_memory_resource* mr,
                                                       hipStream_t stream)
{
  return cudf::type_dispatcher(
    x.type(), dispatch_element{}, num_trajectories, object_id, x, y, mr, stream);
}
}  // namespace detail

std::unique_ptr<cudf::table> trajectory_bounding_boxes(cudf::size_type num_trajectories,
                                                       cudf::column_view const& object_id,
                                                       cudf::column_view const& x,
                                                       cudf::column_view const& y,
                                                       rmm::mr::device_memory_resource* mr)
{
  CUSPATIAL_EXPECTS(object_id.size() == x.size() && x.size() == y.size(), "Data size mismatch");
  CUSPATIAL_EXPECTS(x.type().id() == y.type().id(), "Data type mismatch");
  CUSPATIAL_EXPECTS(object_id.type().id() == cudf::INT32, "Invalid object_id type");
  CUSPATIAL_EXPECTS(!(x.has_nulls() || y.has_nulls() || object_id.has_nulls()),
                    "NULL support unimplemented");

  if (num_trajectories == 0 || object_id.is_empty() || x.is_empty() || y.is_empty()) {
    std::vector<std::unique_ptr<cudf::column>> cols{};
    cols.reserve(4);
    cols.push_back(cudf::empty_like(x));
    cols.push_back(cudf::empty_like(y));
    cols.push_back(cudf::empty_like(x));
    cols.push_back(cudf::empty_like(y));
    return std::make_unique<cudf::table>(std::move(cols));
  }

  return detail::trajectory_bounding_boxes(num_trajectories, object_id, x, y, mr, 0);
}

}  // namespace experimental
}  // namespace cuspatial
