/*
 * Copyright (c) 2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <rmm/thrust_rmm_allocator.h>

#include <thrust/iterator/constant_iterator.h>
#include <thrust/iterator/discard_iterator.h>

#include <cudf/column/column_factories.hpp>
#include <cudf/copying.hpp>
#include <cudf/detail/sorting.hpp>
#include <cudf/table/table.hpp>
#include <cudf/table/table_view.hpp>

#include <cuspatial/error.hpp>
#include <cuspatial/trajectory.hpp>

#include <memory>
#include <vector>

namespace cuspatial {
namespace experimental {
namespace detail {

std::pair<std::unique_ptr<cudf::table>, std::unique_ptr<cudf::column>> derive_trajectories(
  cudf::column_view const& object_id,
  cudf::column_view const& x,
  cudf::column_view const& y,
  cudf::column_view const& timestamp,
  rmm::mr::device_memory_resource* mr,
  hipStream_t stream)
{
  auto sorted = cudf::detail::sort_by_key(cudf::table_view{{object_id, x, y, timestamp}},
                                          cudf::table_view{{object_id, timestamp}},
                                          {},
                                          {},
                                          mr,
                                          stream);

  auto policy    = rmm::exec_policy(stream);
  auto sorted_id = sorted->get_column(0).view();
  rmm::device_vector<int32_t> lengths(object_id.size());
  auto grouped = thrust::reduce_by_key(policy->on(stream),
                                       sorted_id.begin<int32_t>(),
                                       sorted_id.end<int32_t>(),
                                       thrust::make_constant_iterator(1),
                                       thrust::make_discard_iterator(),
                                       lengths.begin());

  auto offsets = cudf::make_numeric_column(cudf::data_type{cudf::INT32},
                                           thrust::distance(lengths.begin(), grouped.second),
                                           cudf::mask_state::UNALLOCATED,
                                           stream,
                                           mr);

  thrust::exclusive_scan(
    policy->on(stream), lengths.begin(), lengths.end(), offsets->mutable_view().begin<int32_t>());

  return std::make_pair(std::move(sorted), std::move(offsets));
}
}  // namespace detail

std::pair<std::unique_ptr<cudf::table>, std::unique_ptr<cudf::column>> derive_trajectories(
  cudf::column_view const& object_id,
  cudf::column_view const& x,
  cudf::column_view const& y,
  cudf::column_view const& timestamp,
  rmm::mr::device_memory_resource* mr)
{
  CUSPATIAL_EXPECTS(
    x.size() == y.size() && x.size() == object_id.size() && x.size() == timestamp.size(),
    "Data size mismatch");
  CUSPATIAL_EXPECTS(object_id.type().id() == cudf::INT32, "Invalid object_id datatype");
  CUSPATIAL_EXPECTS(cudf::is_timestamp(timestamp.type()), "Invalid timestamp datatype");
  CUSPATIAL_EXPECTS(
    !(x.has_nulls() || y.has_nulls() || object_id.has_nulls() || timestamp.has_nulls()),
    "NULL support unimplemented");
  if (object_id.is_empty() || x.is_empty() || y.is_empty() || timestamp.is_empty()) {
    std::vector<std::unique_ptr<cudf::column>> cols{};
    cols.reserve(4);
    cols.push_back(cudf::empty_like(object_id));
    cols.push_back(cudf::empty_like(x));
    cols.push_back(cudf::empty_like(y));
    cols.push_back(cudf::empty_like(timestamp));
    return std::make_pair(std::make_unique<cudf::table>(std::move(cols)),
                          cudf::make_empty_column(cudf::data_type{cudf::INT32}));
  }
  return detail::derive_trajectories(object_id, x, y, timestamp, mr, 0);
}
}  // namespace experimental
}  // namespace cuspatial
