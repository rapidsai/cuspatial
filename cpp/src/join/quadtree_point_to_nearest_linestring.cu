#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020-2022, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "detail/get_quad_and_local_point_indices.cuh"

#include <indexing/construction/detail/utilities.cuh>
#include <utility/point_to_nearest_linestring.cuh>

#include <cuspatial/error.hpp>
#include <cuspatial/spatial_join.hpp>

#include <cudf/column/column_device_view.cuh>
#include <cudf/column/column_factories.hpp>
#include <cudf/column/column_view.hpp>
#include <cudf/table/table.hpp>
#include <cudf/table/table_view.hpp>

#include <rmm/cuda_stream_view.hpp>
#include <rmm/device_uvector.hpp>
#include <rmm/exec_policy.hpp>

#include <thrust/binary_search.h>
#include <thrust/distance.h>
#include <thrust/execution_policy.h>
#include <thrust/fill.h>
#include <thrust/functional.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/discard_iterator.h>
#include <thrust/iterator/permutation_iterator.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/reduce.h>
#include <thrust/scan.h>
#include <thrust/scatter.h>
#include <thrust/tuple.h>

#include <limits>
#include <memory>

namespace cuspatial {
namespace detail {
namespace {

template <typename QuadOffsetsIter>
inline __device__ std::pair<uint32_t, uint32_t> get_local_linestring_index_and_count(
  uint32_t const linestring_index, QuadOffsetsIter quad_offsets, QuadOffsetsIter quad_offsets_end)
{
  auto const lhs_end     = quad_offsets;
  auto const rhs_end     = quad_offsets_end;
  auto const quad_offset = quad_offsets[linestring_index];
  auto const lhs =
    thrust::lower_bound(thrust::seq, lhs_end, quad_offsets + linestring_index, quad_offset);
  auto const rhs =
    thrust::upper_bound(thrust::seq, quad_offsets + linestring_index, rhs_end, quad_offset);

  return std::make_pair(
    // local_linestring_index
    static_cast<uint32_t>(thrust::distance(lhs, quad_offsets + linestring_index)),
    // num_linestrings_in_quad
    static_cast<uint32_t>(thrust::distance(lhs, rhs)));
}

template <typename QuadOffsetsIter, typename QuadLengthsIter>
inline __device__ std::pair<uint32_t, uint32_t> get_transposed_point_and_pair_index(
  uint32_t const global_index,
  uint32_t const* point_offsets,
  uint32_t const* point_offsets_end,
  QuadOffsetsIter quad_offsets,
  QuadOffsetsIter quad_offsets_end,
  QuadLengthsIter quad_lengths)
{
  auto const [quad_linestring_index, local_point_index] =
    get_quad_and_local_point_indices(global_index, point_offsets, point_offsets_end);

  auto const [local_linestring_index, num_linestrings_in_quad] =
    get_local_linestring_index_and_count(quad_linestring_index, quad_offsets, quad_offsets_end);

  auto const quad_point_offset           = quad_offsets[quad_linestring_index];
  auto const num_points_in_quad          = quad_lengths[quad_linestring_index];
  auto const quad_linestring_offset      = quad_linestring_index - local_linestring_index;
  auto const quad_linestring_point_start = local_linestring_index * num_points_in_quad;
  auto const transposed_point_start      = quad_linestring_point_start + local_point_index;

  return std::make_pair(
    // transposed point index
    (transposed_point_start / num_linestrings_in_quad) + quad_point_offset,
    // transposed linestring index
    (transposed_point_start % num_linestrings_in_quad) + quad_linestring_offset);
}

template <typename T, typename PointIter, typename QuadOffsetsIter, typename QuadLengthsIter>
struct compute_point_linestring_indices_and_distances {
  PointIter points;
  uint32_t const* point_offsets;
  uint32_t const* point_offsets_end;
  QuadOffsetsIter quad_offsets;
  QuadOffsetsIter quad_offsets_end;
  QuadLengthsIter quad_lengths;
  uint32_t const* linestring_indices;
  cudf::column_device_view const linestring_offsets;
  cudf::column_device_view const linestring_points_x;
  cudf::column_device_view const linestring_points_y;
  inline __device__ thrust::tuple<uint32_t, uint32_t, T> operator()(uint32_t const global_index)
  {
    auto const [point_id, linestring_id] = get_transposed_point_and_pair_index(
      global_index, point_offsets, point_offsets_end, quad_offsets, quad_offsets_end, quad_lengths);

    T x{}, y{};
    thrust::tie(x, y)         = points[point_id];
    auto const linestring_idx = linestring_indices[linestring_id];
    auto const distance       = point_to_linestring_distance<T>(
      x, y, linestring_idx, linestring_offsets, linestring_points_x, linestring_points_y);

    return thrust::make_tuple(point_id, linestring_idx, distance);
  }
};

struct compute_quadtree_point_to_nearest_linestring {
  template <typename T, typename... Args>
  std::enable_if_t<!std::is_floating_point<T>::value, std::unique_ptr<cudf::table>> operator()(
    Args&&...)
  {
    CUDF_FAIL("Non-floating point operation is not supported");
  }

  template <typename T>
  std::enable_if_t<std::is_floating_point<T>::value, std::unique_ptr<cudf::table>> operator()(
    cudf::table_view const& linestring_quad_pairs,
    cudf::table_view const& quadtree,
    cudf::column_view const& point_indices,
    cudf::column_view const& point_x,
    cudf::column_view const& point_y,
    cudf::column_view const& linestring_offsets,
    cudf::column_view const& linestring_points_x,
    cudf::column_view const& linestring_points_y,
    rmm::cuda_stream_view stream,
    rmm::mr::device_memory_resource* mr)
  {
    // Wrapped in an IIFE so `local_point_offsets` is freed on return
    auto const [point_idxs, linestring_idxs, distances, num_distances] = [&]() {
      auto num_linestring_quad_pairs = linestring_quad_pairs.num_rows();
      auto linestring_indices        = linestring_quad_pairs.column(0).begin<uint32_t>();
      auto quad_lengths              = thrust::make_permutation_iterator(
        quadtree.column(3).begin<uint32_t>(), linestring_quad_pairs.column(1).begin<uint32_t>());
      auto quad_offsets = thrust::make_permutation_iterator(
        quadtree.column(4).begin<uint32_t>(), linestring_quad_pairs.column(1).begin<uint32_t>());

      // Compute a "local" set of zero-based point offsets from number of points in each quadrant
      // Use `num_linestring_quad_pairs + 1` as the length so that the last element produced by
      // `inclusive_scan` is the total number of points to be tested against any linestring.
      rmm::device_uvector<uint32_t> local_point_offsets(num_linestring_quad_pairs + 1, stream);

      thrust::inclusive_scan(rmm::exec_policy(stream),
                             quad_lengths,
                             quad_lengths + num_linestring_quad_pairs,
                             local_point_offsets.begin() + 1);

      // Ensure local point offsets starts at 0
      uint32_t init{0};
      local_point_offsets.set_element_async(0, init, stream);

      // The last element is the total number of points to test against any linestring.
      auto num_point_linestring_pairs = local_point_offsets.back_element(stream);

      // Enumerate the point X/Ys using the sorted `point_indices` (from quadtree construction)
      auto point_xys_iter = thrust::make_permutation_iterator(
        thrust::make_zip_iterator(point_x.begin<T>(), point_y.begin<T>()),
        point_indices.begin<uint32_t>());

      //
      // Compute the combination of point and linestring index pairs. For each linestring / quadrant
      // pair, enumerate pairs of (point_index, linestring_index) for each point in each quadrant,
      // and calculate the minimum distance between each point / linestring pair.
      //
      // In Python pseudocode:
      // ```
      // pl_pairs_and_dist = []
      // for linestring, quadrant in lq_pairs:
      //   for point in quadrant:
      //     pl_pairs_and_dist.append((point, linestring, min_distance(point, linestring)))
      // ```
      //
      // However, the above psuedocode produces values in an order such that the distance
      // from a point to each linestring cannot be reduced with `thrust::reduce_by_key`:
      // ```
      //   point | linestring | distance
      //       0 |          0 |     10.0
      //       1 |          0 |     30.0
      //       2 |          0 |     20.0
      //       0 |          1 |     30.0
      //       1 |          1 |     20.0
      //       2 |          1 |     10.0
      // ```
      //
      // In order to use `thrust::reduce_by_key` to compute the minimum distance from a point to
      // the linestrings in its quadrant, the above table needs to be sorted by `point` instead of
      // `linestring`:
      // ```
      //   point | linestring | distance
      //       0 |          0 |     10.0
      //       0 |          1 |     30.0
      //       1 |          0 |     30.0
      //       1 |          1 |     20.0
      //       2 |          0 |     20.0
      //       2 |          1 |     10.0
      // ```
      //
      // A naive approach would be to allocate memory for the above three columns, sort the
      // columns by `point`, then use `thrust::reduce_by_key` to compute the min distances.
      //
      // The sizes of the intermediate buffers required can easily grow beyond available
      // device memory, so a better approach is to use a Thrust iterator to yield values
      // in the sorted order as we do here.
      //

      auto all_point_linestring_indices_and_distances = thrust::make_transform_iterator(
        thrust::make_counting_iterator(0u),
        compute_point_linestring_indices_and_distances<T,
                                                       decltype(point_xys_iter),
                                                       decltype(quad_offsets),
                                                       decltype(quad_lengths)>{
          point_xys_iter,
          local_point_offsets.begin(),
          local_point_offsets.end(),
          quad_offsets,
          quad_offsets + num_linestring_quad_pairs,
          quad_lengths,
          linestring_indices,
          *cudf::column_device_view::create(linestring_offsets, stream),
          *cudf::column_device_view::create(linestring_points_x, stream),
          *cudf::column_device_view::create(linestring_points_y, stream)});

      auto all_point_indices =
        thrust::make_transform_iterator(all_point_linestring_indices_and_distances,
                                        [] __device__(auto const& x) { return thrust::get<0>(x); });

      // Allocate vectors for the distances min reduction
      rmm::device_uvector<uint32_t> point_idxs(point_x.size(), stream);
      rmm::device_uvector<uint32_t> linestring_idxs(point_x.size(), stream);
      rmm::device_uvector<T> distances(point_x.size(), stream);

      // Fill distances with 0
      CUSPATIAL_CUDA_TRY(
        hipMemsetAsync(distances.data(), 0, distances.size() * sizeof(T), stream.value()));

      // Reduce the intermediate point/linestring indices to lists of point/linestring index pairs
      // and distances, selecting the linestring index closest to each point.
      auto point_idxs_end = thrust::reduce_by_key(
        rmm::exec_policy(stream),
        all_point_indices,  // point indices in
        all_point_indices + num_point_linestring_pairs,
        all_point_linestring_indices_and_distances,
        point_idxs.begin(),  // point indices out
        // point/linestring indices and distances out
        thrust::make_zip_iterator(
          thrust::make_discard_iterator(), linestring_idxs.begin(), distances.begin()),
        thrust::equal_to<uint32_t>(),  // comparator
        // binop to select the point/linestring pair with the smallest distance
        [] __device__(auto const& lhs, auto const& rhs) {
          T const& d_lhs = thrust::get<2>(lhs);
          T const& d_rhs = thrust::get<2>(rhs);
          // If lhs distance is 0, choose rhs
          if (d_lhs == T{0}) { return rhs; }
          // if rhs distance is 0, choose lhs
          if (d_rhs == T{0}) { return lhs; }
          // If distances to lhs/rhs are the same, choose linestring with smallest id
          if (d_lhs == d_rhs) {
            auto const& i_lhs = thrust::get<1>(lhs);
            auto const& i_rhs = thrust::get<1>(rhs);
            return i_lhs < i_rhs ? lhs : rhs;
          }
          // Otherwise choose linestring with smallest distance
          return d_lhs < d_rhs ? lhs : rhs;
        });

      auto const num_distances = thrust::distance(point_idxs.begin(), point_idxs_end.first);

      return std::make_tuple(
        std::move(point_idxs), std::move(linestring_idxs), std::move(distances), num_distances);
    }();

    // Allocate output columns for the point and linestring index pairs and their distances
    auto point_index_col      = make_fixed_width_column<uint32_t>(point_x.size(), stream, mr);
    auto linestring_index_col = make_fixed_width_column<uint32_t>(point_x.size(), stream, mr);
    auto distance_col         = make_fixed_width_column<T>(point_x.size(), stream, mr);

    // Note: no need to resize `point_idxs`, `linestring_idxs`, or `distances` if we set the end
    // iterator to `point_linestring_idxs_and_distances + num_distances`.

    auto point_linestring_idxs_and_distances =
      thrust::make_zip_iterator(point_idxs.begin(), linestring_idxs.begin(), distances.begin());

    // scatter the values from their positions after reduction into their output positions
    thrust::scatter(
      rmm::exec_policy(stream),
      point_linestring_idxs_and_distances,
      point_linestring_idxs_and_distances + num_distances,
      point_idxs.begin(),
      thrust::make_zip_iterator(point_index_col->mutable_view().begin<uint32_t>(),
                                linestring_index_col->mutable_view().begin<uint32_t>(),
                                distance_col->mutable_view().template begin<T>()));

    std::vector<std::unique_ptr<cudf::column>> cols{};
    cols.reserve(3);
    cols.push_back(std::move(point_index_col));
    cols.push_back(std::move(linestring_index_col));
    cols.push_back(std::move(distance_col));
    return std::make_unique<cudf::table>(std::move(cols));
  }
};

}  // namespace

std::unique_ptr<cudf::table> quadtree_point_to_nearest_linestring(
  cudf::table_view const& linestring_quad_pairs,
  cudf::table_view const& quadtree,
  cudf::column_view const& point_indices,
  cudf::column_view const& point_x,
  cudf::column_view const& point_y,
  cudf::column_view const& linestring_offsets,
  cudf::column_view const& linestring_points_x,
  cudf::column_view const& linestring_points_y,
  rmm::cuda_stream_view stream,
  rmm::mr::device_memory_resource* mr)
{
  return cudf::type_dispatcher(point_x.type(),
                               compute_quadtree_point_to_nearest_linestring{},
                               linestring_quad_pairs,
                               quadtree,
                               point_indices,
                               point_x,
                               point_y,
                               linestring_offsets,
                               linestring_points_x,
                               linestring_points_y,
                               stream,
                               mr);
}

}  // namespace detail

std::unique_ptr<cudf::table> quadtree_point_to_nearest_linestring(
  cudf::table_view const& linestring_quad_pairs,
  cudf::table_view const& quadtree,
  cudf::column_view const& point_indices,
  cudf::column_view const& point_x,
  cudf::column_view const& point_y,
  cudf::column_view const& linestring_offsets,
  cudf::column_view const& linestring_points_x,
  cudf::column_view const& linestring_points_y,
  rmm::mr::device_memory_resource* mr)
{
  CUSPATIAL_EXPECTS(linestring_quad_pairs.num_columns() == 2,
                    "a quadrant-linestring table must have 2 columns");
  CUSPATIAL_EXPECTS(quadtree.num_columns() == 5, "a quadtree table must have 5 columns");
  CUSPATIAL_EXPECTS(point_indices.size() == point_x.size() && point_x.size() == point_y.size(),
                    "number of points must be the same for both x and y columns");
  CUSPATIAL_EXPECTS(linestring_points_x.size() == linestring_points_y.size(),
                    "numbers of vertices must be the same for both x and y columns");
  CUSPATIAL_EXPECTS(linestring_points_x.size() >= 2 * linestring_offsets.size(),
                    "all linestrings must have at least two vertices");
  CUSPATIAL_EXPECTS(linestring_points_x.type() == linestring_points_y.type(),
                    "linestring columns must have the same data type");
  CUSPATIAL_EXPECTS(point_x.type() == point_y.type(), "point columns must have the same data type");
  CUSPATIAL_EXPECTS(point_x.type() == linestring_points_x.type(),
                    "points and linestrings must have the same data type");

  if (linestring_quad_pairs.num_rows() == 0 || quadtree.num_rows() == 0 ||
      point_indices.size() == 0 || linestring_offsets.size() == 0) {
    std::vector<std::unique_ptr<cudf::column>> cols{};
    cols.reserve(3);
    cols.push_back(cudf::make_empty_column(cudf::data_type{cudf::type_id::UINT32}));
    cols.push_back(cudf::make_empty_column(cudf::data_type{cudf::type_id::UINT32}));
    cols.push_back(cudf::make_empty_column(point_x.type()));
    return std::make_unique<cudf::table>(std::move(cols));
  }

  return detail::quadtree_point_to_nearest_linestring(linestring_quad_pairs,
                                                      quadtree,
                                                      point_indices,
                                                      point_x,
                                                      point_y,
                                                      linestring_offsets,
                                                      linestring_points_x,
                                                      linestring_points_y,
                                                      rmm::cuda_stream_default,
                                                      mr);
}

}  // namespace cuspatial
