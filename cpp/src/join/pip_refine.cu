#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <thrust/gather.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/iterator/discard_iterator.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/reduce.h>

#include <vector>
#include <cudf/column/column.hpp>
#include <cudf/column/column_view.hpp>
#include <cudf/column/column_factories.hpp>
#include <cudf/column/column_device_view.cuh>
#include <cudf/table/table_view.hpp>
#include <cudf/table/table.hpp>

#include <utility/helper_thrust.cuh>
#include <utility/quadtree_thrust.cuh>
#include <utility/bbox_thrust.cuh>
#include <cuspatial/bounding_box.hpp>
#include <cuspatial/spatial_jion.hpp>


namespace
{

const uint8_t max_warps_per_block=32;
const uint8_t num_threads_per_warp=32;

template <typename T>
__global__ void quad_pip_phase1_kernel(const uint32_t * pq_poly_id,const uint32_t *pq_quad_id,
	const uint32_t *pnt_length,const uint32_t *pnt_fpos, const T*  pnt_x,const T*  pnt_y, 
	const uint32_t*  poly_fpos,const uint32_t*  poly_rpos,const T*  poly_x,const T*  poly_y,
        uint32_t* num_hits)        
{
    __shared__ uint32_t qid,pid,num_point,first_pos,num_adjusted;
    
    //assume #of points/threads no more than num_threads_per_warp*max_warps_per_block (32*32)
    __shared__ uint32_t data[max_warps_per_block];
    //assuming 1d 
    if(threadIdx.x==0)
    {
    	qid=pq_quad_id[blockIdx.x];
    	pid=pq_poly_id[blockIdx.x];
    	num_point=pnt_length[qid];
    	first_pos=pnt_fpos[qid]; 
    	num_adjusted=((num_point-1)/num_threads_per_warp+1)*num_threads_per_warp;
       	//printf("block=%d qid=%d pid=%d num_point=%d first_pos=%d\n",
    	//	blockIdx.x,qid,pid,num_point,first_pos);	
    }
     __syncthreads();
     
    if((threadIdx.x>=max_warps_per_block)&&(threadIdx.x>=num_adjusted))
    	return;
    __syncthreads();
    
    if(threadIdx.x<max_warps_per_block)
        data[threadIdx.x]=0;
    __syncthreads();
    
    uint32_t tid = first_pos+threadIdx.x;
    bool in_polygon = false;
    if(threadIdx.x<num_point)
    {
       T x = pnt_x[tid];
       T y = pnt_y[tid];
      
       uint32_t r_f = (0 == pid) ? 0 : poly_fpos[pid-1];
       uint32_t r_t=poly_fpos[pid];
       for (uint32_t k = r_f; k < r_t; k++) //for each ring
       {
           uint32_t m = (k==0)?0:poly_rpos[k-1];
           for (;m < poly_rpos[k]-1; m++) //for each line segment
           {
              T x0, x1, y0, y1;
              x0 = poly_x[m];
              y0 = poly_y[m];
              x1 = poly_x[m+1];
              y1 = poly_y[m+1];
              //printf("block=%2d thread=%2d tid=%2d r_f=%2d r_t=%2d x=%10.5f y=%10.5f x0=%10.5f y0=%10.5f x1=%10.5f y1=%10.5f\n",
              //	blockIdx.x,threadIdx.x,tid,r_f,r_t,x,y,x0,y0,x1,y1);

              if ((((y0 <= y) && (y < y1)) ||
                   ((y1 <= y) && (y < y0))) &&
                       (x < (x1 - x0) * (y - y0) / (y1 - y0) + x0))
                 in_polygon = !in_polygon;
            }//m
         }//k
      }
      __syncthreads();

      unsigned mask = __ballot_sync(0xFFFFFFFF, threadIdx.x < num_point);
      uint32_t vote=__ballot_sync(mask,in_polygon);
      //printf("p1: block=%d thread=%d tid=%d in_polygon=%d mask=%08x vote=%08x\n",blockIdx.x,threadIdx.x,tid,in_polygon,mask,vote);
      
      if(threadIdx.x%num_threads_per_warp==0)
      	data[threadIdx.x/num_threads_per_warp]=__popc(vote);  
      __syncthreads();
      
      /*if(threadIdx.x<max_warps_per_block)
      	printf("p1: block=%d thread=%d data=%d\n",blockIdx.x,threadIdx.x,data[threadIdx.x]);
      __syncthreads();*/
      
      if(threadIdx.x<max_warps_per_block)
      {
      	uint32_t num=data[threadIdx.x];
        for (uint32_t offset = max_warps_per_block/2; offset > 0; offset /= 2) 
            num += __shfl_xor_sync(0xFFFFFFFF,num, offset);  	
        if(threadIdx.x==0)
            num_hits[blockIdx.x]=num;
      }
      __syncthreads();
}

template <typename T>
__global__ void quad_pip_phase2_kernel(const uint32_t * pq_poly_id,const uint32_t *pq_quad_id,
	const uint32_t *pnt_length,const uint32_t *pnt_fpos, const T*  pnt_x,const T*  pnt_y, const uint32_t* pnt_id,
	const uint32_t *poly_id,const uint32_t* poly_fpos,const uint32_t*  poly_rpos,const T*  poly_x,const T*  poly_y,
        uint32_t *d_num_hits,uint32_t *d_res_poly_id,uint32_t *d_res_pnt_id)        
{
    __shared__ uint32_t qid,pid,num_point,first_pos,mem_offset,num_adjusted;
    
    //assume #of points/threads no more than num_threads_per_warp*max_warps_per_block (32*32)
    __shared__ uint16_t temp[max_warps_per_block],sums[max_warps_per_block+1];

    //assuming 1d 
    if(threadIdx.x==0)
    {
    	qid=pq_quad_id[blockIdx.x];
    	pid=pq_poly_id[blockIdx.x];
    	num_point=pnt_length[qid];
    	first_pos=pnt_fpos[qid]; 
    	mem_offset=d_num_hits[blockIdx.x];
    	sums[0]=0;
    	num_adjusted=((num_point-1)/num_threads_per_warp+1)*num_threads_per_warp;
     	//printf("block=%d qid=%d pid=%d num_point=%d first_pos=%d mem_offset=%d\n",
    	//	blockIdx.x,qid,pid,num_point,first_pos,mem_offset);
    		
    }
    __syncthreads();

     if(threadIdx.x<max_warps_per_block+1)
    	temp[threadIdx.x]=0;
    __syncthreads();
   
    uint32_t tid = first_pos+threadIdx.x;    	
    bool in_polygon = false;
    if(threadIdx.x<num_point)
    {   
       T x = pnt_x[tid];
       T y = pnt_y[tid];
     
       uint32_t r_f = (0 == pid) ? 0 : poly_fpos[pid-1];
       uint32_t r_t=poly_fpos[pid];
       for (uint16_t k = r_f; k < r_t; k++) //for each ring
       {
           uint16_t m = (k==0)?0:poly_rpos[k-1];
           for (;m < poly_rpos[k]-1; m++) //for each line segment
           {
              T x0, x1, y0, y1;
              x0 = poly_x[m];
              y0 = poly_y[m];
              x1 = poly_x[m+1];
              y1 = poly_y[m+1];

              if ((((y0 <= y) && (y < y1)) ||
                   ((y1 <= y) && (y < y0))) &&
                       (x < (x1 - x0) * (y - y0) / (y1 - y0) + x0))
                 in_polygon = !in_polygon;
            }//m
          }//k
      }
      __syncthreads();    
  
      unsigned mask = __ballot_sync(0xFFFFFFFF, threadIdx.x < num_adjusted);
      uint32_t vote=__ballot_sync(mask,in_polygon);    
      if(threadIdx.x%num_threads_per_warp==0)
      	temp[threadIdx.x/num_threads_per_warp]=__popc(vote);  
      __syncthreads();
    
     //warp-level scan; only one warp is used
     if(threadIdx.x<num_threads_per_warp)
      {
          uint16_t num=temp[threadIdx.x];
          for (uint8_t i=1; i<=num_threads_per_warp; i*=2)
          {
            int n = __shfl_up_sync(0xFFFFFFF,num, i, num_threads_per_warp);
            if (threadIdx.x >= i) num += n;
          }
          sums[threadIdx.x+1]=num;
          __syncthreads();
      }
      //important!!!!!!!!!!!
      __syncthreads();
      
      /*if(threadIdx.x<num_point)
      	printf("after: block=%d thread=%d tid=%d %10.5f %10.5f in_polygon=%d val=%d\n",
      		blockIdx.x,threadIdx.x,tid,pnt_x[tid],pnt_y[tid],in_polygon,sums[threadIdx.x]);
      __syncthreads();*/
      
      if((threadIdx.x<num_point)&&(in_polygon))
      {
     	uint16_t num=sums[threadIdx.x/num_threads_per_warp];
     	uint16_t warp_offset=__popc(vote>>(threadIdx.x%num_threads_per_warp))-1;
     	uint16_t pos=num+warp_offset;
     	
     	//printf("block=%d thread=%d qid=%d pid=%d tid=%d mem_offset=%d num=%d warp_offset=%d pos=%d\n",
    	//	blockIdx.x,threadIdx.x,qid,pid,tid,mem_offset,num,warp_offset,pos); 
    		
        d_res_poly_id[mem_offset+pos]=poly_id[pid];
        d_res_pnt_id[mem_offset+pos]=pnt_id[tid];
      } 
      __syncthreads();
}

template<typename T>
std::vector<std::unique_ptr<cudf::column>> dowork(
	uint32_t num_pq_pair,const uint32_t * d_pq_polyid,const uint32_t * d_pq_quadid,
	uint32_t num_node,const uint32_t *d_qt_key,const uint8_t *d_qt_lev,
	const bool *d_qt_sign, const uint32_t *d_qt_length, const uint32_t *d_qt_fpos,
	const uint32_t num_pnt,const uint32_t * d_pnt_id,const T *d_pnt_x, const T *d_pnt_y,
	const uint32_t num_poly,const uint32_t * d_poly_id,const uint32_t * d_poly_fpos,
	const uint32_t * d_poly_rpos,const T *d_poly_x, const T *d_poly_y,
	rmm::mr::device_memory_resource* mr, hipStream_t stream)	
                                         
{
    auto exec_policy = rmm::exec_policy(stream)->on(stream);
    
    uint32_t *d_num_hits=NULL;
    RMM_TRY( RMM_ALLOC( &d_num_hits,num_pq_pair* sizeof(uint32_t), 0));
    assert(d_num_hits!=NULL);
    HANDLE_CUDA_ERROR( hipMemset(d_num_hits,0,num_pq_pair*sizeof(uint32_t)) ); 

    printf("running quad_pip_phase1_kernel\n");
    quad_pip_phase1_kernel<T> <<< num_pq_pair, 256 >>> (d_pq_polyid,d_pq_quadid,
    	d_qt_length,d_qt_fpos,d_pnt_x,d_pnt_y,d_poly_fpos,d_poly_rpos,d_poly_x,d_poly_y,d_num_hits);
    HANDLE_CUDA_ERROR( hipDeviceSynchronize() );	

    //remove poly-quad pair with zero hits
 
    uint32_t *d_tmp_polyid=NULL,*d_tmp_quadid=NULL;
    RMM_TRY( RMM_ALLOC( &d_tmp_polyid,num_pq_pair* sizeof(uint32_t), stream));
    RMM_TRY( RMM_ALLOC( &d_tmp_quadid,num_pq_pair* sizeof(uint32_t), stream));
    assert(d_tmp_polyid!=NULL && d_tmp_quadid!=NULL);
    HANDLE_CUDA_ERROR( hipMemcpy( (void *)d_tmp_polyid, (void *)d_pq_polyid, num_pq_pair * sizeof(uint32_t), hipMemcpyDeviceToDevice ) );
    HANDLE_CUDA_ERROR( hipMemcpy( (void *)d_tmp_quadid, (void *)d_pq_quadid, num_pq_pair * sizeof(uint32_t), hipMemcpyDeviceToDevice ) );

if(1)
{
	printf("phase1 results before remove:\n");	
	thrust::device_ptr<uint32_t> d_num_hits_ptr=thrust::device_pointer_cast(d_num_hits);		
	printf("d_num_hits: before reduce\n");
        thrust::copy(d_num_hits_ptr,d_num_hits_ptr+num_pq_pair,std::ostream_iterator<uint32_t>(std::cout, " "));std::cout<<std::endl; 
}
    auto valid_pq_pair_iter=thrust::make_zip_iterator(thrust::make_tuple(d_tmp_polyid,d_tmp_quadid,d_num_hits));    
    uint32_t num_valid_pair=thrust::remove_if(exec_policy,valid_pq_pair_iter,valid_pq_pair_iter+num_pq_pair,
    	valid_pq_pair_iter,pq_remove_zero())-valid_pq_pair_iter;   
    printf("num_valid_pair=%d\n",num_valid_pair);

if(1)
{
	printf("phase1 results after remove:\n");	
	thrust::device_ptr<uint32_t> d_num_hits_ptr=thrust::device_pointer_cast(d_num_hits);		
	printf("d_num_hits: before reduce\n");
        thrust::copy(d_num_hits_ptr,d_num_hits_ptr+num_valid_pair,std::ostream_iterator<uint32_t>(std::cout, " "));std::cout<<std::endl; 
}    
    uint32_t total_hits=thrust::reduce(exec_policy,d_num_hits,d_num_hits+num_valid_pair);
    printf("total_hits=%d\n",total_hits);
    thrust::exclusive_scan(exec_policy,d_num_hits,d_num_hits+num_valid_pair,d_num_hits);

if(1)
{
	printf("phase1 results:\n");	
	thrust::device_ptr<uint32_t> d_num_hits_ptr=thrust::device_pointer_cast(d_num_hits);		
	printf("d_num_hits: after reduce\n");
        thrust::copy(d_num_hits_ptr,d_num_hits_ptr+num_valid_pair,std::ostream_iterator<uint32_t>(std::cout, " "));std::cout<<std::endl; 
}
  
    std::unique_ptr<cudf::column> poly_id_col = cudf::make_numeric_column(
       cudf::data_type(cudf::type_id::INT32), total_hits,cudf::mask_state::UNALLOCATED,  stream, mr);      
    uint32_t *d_res_poly_id=cudf::mutable_column_device_view::create(poly_id_col->mutable_view(), stream)->data<uint32_t>();
    CUDF_EXPECTS(d_res_poly_id!=NULL,"poly_id can not be NULL"); 
   
    std::unique_ptr<cudf::column> pnt_id_col = cudf::make_numeric_column(
       cudf::data_type(cudf::type_id::INT32), total_hits,cudf::mask_state::UNALLOCATED,  stream, mr);      
    uint32_t *d_res_pnt_id=cudf::mutable_column_device_view::create(pnt_id_col->mutable_view(), stream)->data<uint32_t>();
    CUDF_EXPECTS(d_res_pnt_id!=NULL,"point_id can not be NULL"); 
        
    printf("running quad_pip_phase2_kernel\n");
    quad_pip_phase2_kernel<T> <<< num_valid_pair, 256 >>> 
       (d_tmp_polyid,d_tmp_quadid,
    	d_qt_length,d_qt_fpos,d_pnt_x,d_pnt_y,d_pnt_id,
    	d_poly_id,d_poly_fpos,d_poly_rpos,d_poly_x,d_poly_y,
    	d_num_hits,d_res_poly_id,d_res_pnt_id);   
    HANDLE_CUDA_ERROR( hipDeviceSynchronize() );
    
    RMM_TRY(RMM_FREE(d_num_hits,stream));d_num_hits=NULL;
    RMM_TRY(RMM_FREE(d_tmp_polyid,stream));d_tmp_polyid=NULL;
    RMM_TRY(RMM_FREE(d_tmp_quadid,stream));d_tmp_quadid=NULL;
    
   //thrust::stable_sort_by_key(exec_policy,d_res_pnt_id,d_res_pnt_id+total_hits,d_res_poly_id);
   
if(1)
{
	printf("phase2 results:\n");	

	thrust::device_ptr<uint32_t> d_res_poly_ptr=thrust::device_pointer_cast(d_res_poly_id);		
	printf("d_res_poly_id\n");
        thrust::copy(d_res_poly_ptr,d_res_poly_ptr+total_hits,std::ostream_iterator<uint32_t>(std::cout, " "));std::cout<<std::endl; 

	thrust::device_ptr<uint32_t> d_res_pnt_ptr=thrust::device_pointer_cast(d_res_pnt_id);		
	printf("d_res_pnt_id\n");
        thrust::copy(d_res_pnt_ptr,d_res_pnt_ptr+total_hits,std::ostream_iterator<uint32_t>(std::cout, " "));std::cout<<std::endl; 
}

   std::vector<std::unique_ptr<cudf::column>> pair_cols;
   pair_cols.push_back(std::move(poly_id_col));
   pair_cols.push_back(std::move(pnt_id_col));
   return pair_cols;    
}

struct pip_refine_processor {
  
  template<typename T, std::enable_if_t<std::is_floating_point<T>::value >* = nullptr>
  std::unique_ptr<cudf::experimental::table> operator()(
	cudf::table_view const& pq_pair,cudf::table_view const& quadtree,cudf::table_view const& pnt,
	cudf::column_view const& poly_id,cudf::column_view const& poly_fpos,cudf::column_view const& poly_rpos,
	cudf::column_view const& poly_x,cudf::column_view const& poly_y,
	rmm::mr::device_memory_resource* mr,
        hipStream_t stream)
   {            
       const uint32_t *d_poly_id=poly_id.data<uint32_t>();       
       const uint32_t *d_poly_fpos=poly_fpos.data<uint32_t>();
       const uint32_t *d_poly_rpos=poly_rpos.data<uint32_t>();
       const T *d_poly_x=poly_x.data<T>();
       const T *d_poly_y=poly_y.data<T>();
  
       const uint32_t *d_pnt_id=pnt.column(0).data<uint32_t>();       
       const T *d_pnt_x=pnt.column(1).data<T>();
       const T *d_pnt_y=pnt.column(2).data<T>();

       const uint32_t *d_qt_key=    quadtree.column(0).data<uint32_t>();
       const uint8_t  *d_qt_lev=    quadtree.column(1).data<uint8_t>();
       const bool     *d_qt_sign=   quadtree.column(2).data<bool>();
       const uint32_t *d_qt_length= quadtree.column(3).data<uint32_t>();
       const uint32_t *d_qt_fpos=   quadtree.column(4).data<uint32_t>();
       
       //column 0 and 1 of table pq_pair are for debug use only
       const uint32_t *d_pq_poly_id=   pq_pair.column(2).data<uint32_t>();
       const uint32_t *d_pq_quad_id=   pq_pair.column(3).data<uint32_t>();
             
       uint32_t num_pair=pq_pair.num_rows();
       uint32_t num_node=quadtree.num_rows();
       uint32_t num_poly=poly_id.size();
       uint32_t num_pnt=pnt.num_rows();

       std::vector<std::unique_ptr<cudf::column>> pair_cols=
       		dowork(num_pair,d_pq_poly_id,d_pq_quad_id,
       			num_node,d_qt_key,d_qt_lev,d_qt_sign,d_qt_length,d_qt_fpos,
       			num_pnt,d_pnt_id,d_pnt_x,d_pnt_y,
       			num_poly,d_poly_id,d_poly_fpos,d_poly_rpos,d_poly_x,d_poly_y,
       			mr,stream);
       	
      std::unique_ptr<cudf::experimental::table> destination_table = 
    	std::make_unique<cudf::experimental::table>(std::move(pair_cols));      
      
      return destination_table;
    }
  
  template<typename T, std::enable_if_t<!std::is_floating_point<T>::value >* = nullptr>
  std::unique_ptr<cudf::experimental::table> operator()(
	cudf::table_view const& pq_pair,cudf::table_view const& quadtree,cudf::table_view const& pnt,
	cudf::column_view const& poly_id,cudf::column_view const& poly_fpos,cudf::column_view const& poly_rpos,
	cudf::column_view const& poly_x,cudf::column_view const& poly_y,
	rmm::mr::device_memory_resource* mr, hipStream_t stream)       
    {
 	CUDF_FAIL("Non-floating point operation is not supported");
    }  
      
};
  
} //end anonymous namespace

namespace cuspatial {
std::unique_ptr<cudf::experimental::table> pip_refine(
	cudf::table_view const& pq_pair,cudf::table_view const& quadtree,cudf::table_view const& pnt,
	cudf::column_view const& poly_id,cudf::column_view const& poly_fpos,cudf::column_view const& poly_rpos,
	cudf::column_view const& poly_x,cudf::column_view const& poly_y)
	
	
{   
   cudf::data_type pnt_dtype=pnt.column(1).type();
   cudf::data_type poly_dtype=poly_x.type();
   CUDF_EXPECTS(pnt_dtype==poly_dtype,"point and polygon must have the same data type");
   
   hipStream_t stream=0;
   rmm::mr::device_memory_resource* mr=rmm::mr::get_default_resource();    
   
   return cudf::experimental::type_dispatcher(pnt_dtype,pip_refine_processor{}, 
    	pq_pair,quadtree,pnt,poly_id,poly_fpos,poly_rpos,poly_x,poly_y,mr,stream);   
    	    	
}

}// namespace cuspatial
