#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <thrust/gather.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/iterator/discard_iterator.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/reduce.h>

#include <vector>
#include <cudf/column/column.hpp>
#include <cudf/column/column_view.hpp>
#include <cudf/column/column_factories.hpp>
#include <cudf/column/column_device_view.cuh>
#include <cudf/table/table_view.hpp>
#include <cudf/table/table.hpp>

#include <utility/helper_thrust.cuh>
#include <utility/quadtree_thrust.cuh>
#include <utility/bbox_thrust.cuh>
#include <cuspatial/bounding_box.hpp>
#include <cuspatial/spatial_jion.hpp>


namespace
{
typedef thrust::tuple<double, double,double,double,double,uint32_t,uint32_t> quad_point_parameters;

template<typename T>
std::vector<std::unique_ptr<cudf::column>> dowork(
	uint32_t num_node,const uint32_t *d_p_qtkey,const uint8_t *d_p_qtlev,
	const bool *d_p_qtsign, const uint32_t *d_p_qtlength, const uint32_t *d_p_qtfpos,
	const uint32_t num_poly,const T *poly_x1, const T *poly_y1,T const *poly_x2, const T *poly_y2,
	const SBBox<double>& aoi_bbox, double scale,uint32_t num_level, uint32_t min_size, 
	rmm::mr::device_memory_resource* mr, hipStream_t stream)	
                                         
{
    double x1=thrust::get<0>(aoi_bbox.first);
    double y1=thrust::get<1>(aoi_bbox.first);
    double x2=thrust::get<0>(aoi_bbox.second);
    double y2=thrust::get<1>(aoi_bbox.second);
  
    std::cout<<"num_node="<<num_node<<std::endl;
    std::cout<<"num_poly="<<num_poly<<std::endl;
    
    std::cout<<"bounding box(x1,y1,x2,y2)=("<<x1<<","<<y1<<","<<x2<<","<<x2<<","<<y2<<std::endl;
    std::cout<<"scale="<<scale<<std::endl;
    std::cout<<"num_level="<<num_level<<std::endl;
    std::cout<<"match: min_size="<<min_size<<std::endl;
    
    auto exec_policy = rmm::exec_policy(stream)->on(stream);
    
    SBBox<T> * d_poly_sbbox=NULL;
    RMM_TRY( RMM_ALLOC( (void**)&(d_poly_sbbox),num_poly*sizeof(SBBox<T>), 0));
    assert(d_poly_sbbox!=NULL);
 
 if(0)
 {
    printf("x1\n");
    thrust::device_ptr<const T> d_x1_ptr = thrust::device_pointer_cast(poly_x1);
    thrust::copy(d_x1_ptr,d_x1_ptr+num_poly,std::ostream_iterator<T>(std::cout, " "));std::cout<<std::endl;  

    printf("y1\n");
    thrust::device_ptr<const T> d_y1_ptr = thrust::device_pointer_cast(poly_y1);
    thrust::copy(d_y1_ptr,d_y1_ptr+num_poly,std::ostream_iterator<T>(std::cout, " "));std::cout<<std::endl;  

    printf("x2\n");
    thrust::device_ptr<const T> d_x2_ptr = thrust::device_pointer_cast(poly_x2);
    thrust::copy(d_x2_ptr,d_x2_ptr+num_poly,std::ostream_iterator<T>(std::cout, " "));std::cout<<std::endl;  

    printf("y2\n");
    thrust::device_ptr<const T> d_y2_ptr = thrust::device_pointer_cast(poly_y2);
    thrust::copy(d_y2_ptr,d_y2_ptr+num_poly,std::ostream_iterator<T>(std::cout, " "));std::cout<<std::endl;   	
 }

if(0)
{
   printf("qt sign\n");
   thrust::device_ptr<const bool> d_sign_ptr=thrust::device_pointer_cast(d_p_qtsign); 
   thrust::copy(d_sign_ptr,d_sign_ptr+num_node,std::ostream_iterator<bool>(std::cout, " "));std::cout<<std::endl;
}
    auto ploy_bbox_iter=thrust::make_zip_iterator(
    	thrust::make_tuple(poly_x1,poly_y1,poly_x2,poly_y2));
    thrust::transform(exec_policy,ploy_bbox_iter,ploy_bbox_iter+num_poly,d_poly_sbbox,tuple2bbox<T>());
   
    uint32_t num_top_lev_children=thrust::count_if(exec_policy,d_p_qtlev,
    	d_p_qtlev+num_node,thrust::placeholders::_1==0);  
    uint32_t init_len=1000;
    uint32_t curr_len=init_len;
    
    uint8_t *d_pq_lev_out=NULL,*d_pq_type_out=NULL;
    RMM_TRY( RMM_ALLOC( &d_pq_lev_out,curr_len* sizeof(uint8_t), stream));
    assert(d_pq_lev_out!=NULL);
    RMM_TRY( RMM_ALLOC( &d_pq_type_out,curr_len* sizeof(uint8_t), stream));
    assert(d_pq_type_out!=NULL);
  
    uint32_t *d_quad_idx_out=NULL,*d_poly_idx_out=NULL;
    RMM_TRY( RMM_ALLOC( &d_quad_idx_out,curr_len* sizeof(uint32_t), stream));
    assert(d_quad_idx_out!=NULL);
    RMM_TRY( RMM_ALLOC( &d_poly_idx_out,curr_len* sizeof(uint32_t), stream));
    assert(d_poly_idx_out!=NULL);
   
    auto pair_output_iter=thrust::make_zip_iterator(
    	thrust::make_tuple(d_pq_lev_out,d_pq_type_out,d_poly_idx_out,d_quad_idx_out));
    uint32_t b_pos=0;
  
  
    uint32_t  num_pair=num_top_lev_children*num_poly;  
    printf("num_top_lev_children=%d num_poly=%d num_pair=%d\n",num_top_lev_children,num_poly,num_pair);

    uint8_t *d_pq_lev_temp=NULL,*d_pq_type_temp=NULL;
    uint32_t *d_quad_idx_temp=NULL,*d_poly_idx_temp=NULL;
    RMM_TRY( RMM_ALLOC( &d_quad_idx_temp,num_pair* sizeof(uint32_t), stream));
    assert(d_quad_idx_temp!=NULL);
    
    RMM_TRY( RMM_ALLOC( &d_poly_idx_temp,num_pair* sizeof(uint32_t), stream));
    assert(d_poly_idx_temp!=NULL);
    RMM_TRY( RMM_ALLOC( &d_pq_lev_temp,num_pair* sizeof(uint8_t), stream));
    assert(d_pq_lev_temp!=NULL);
    RMM_TRY( RMM_ALLOC( &d_pq_type_temp,num_pair* sizeof(uint8_t), stream));
    assert(d_pq_type_temp!=NULL);
    
    auto pair_counting_iter=thrust::make_counting_iterator(0);
    auto pair_output_temp_iter=thrust::make_zip_iterator(thrust::make_tuple(d_pq_lev_temp,d_pq_type_temp,d_poly_idx_temp,d_quad_idx_temp));

    thrust::transform(exec_policy,pair_counting_iter,pair_counting_iter+num_pair,pair_output_temp_iter,
       		pairwise_test_intersection<T>(num_level,num_top_lev_children,aoi_bbox,scale,d_p_qtkey,d_p_qtlev,d_p_qtsign,d_poly_sbbox)); 		       

    uint32_t num_leaf_pair=thrust::copy_if(exec_policy,pair_output_temp_iter,pair_output_temp_iter+num_pair,
    	pair_output_iter+b_pos,qt_is_type(0))-(pair_output_iter+b_pos);

    uint32_t num_nonleaf_pair=thrust::remove_if(exec_policy,pair_output_temp_iter,pair_output_temp_iter+num_pair,
        pair_output_temp_iter,qt_not_type(1))-pair_output_temp_iter;
    std::cout<<"num_leaf_pair="<<num_leaf_pair<<" ,num_nonleaf_pair="<<num_nonleaf_pair<<std::endl;
    
    b_pos+=num_leaf_pair; 
    for(uint32_t i=1;i<num_level;i++)
    {
        uint32_t *d_quad_nchild=NULL;
        RMM_TRY( RMM_ALLOC( &d_quad_nchild,num_nonleaf_pair* sizeof(uint32_t), stream));
        assert(d_quad_nchild!=NULL);
              
        thrust::transform(exec_policy,d_quad_idx_temp,d_quad_idx_temp+num_nonleaf_pair,
              d_quad_nchild,get_vec_element<const uint32_t>(d_p_qtlength));
               
        num_pair=thrust::reduce(exec_policy,d_quad_nchild,d_quad_nchild+num_nonleaf_pair);
        printf("num_pair after gathering child nodes=%d\n",num_pair);
        
        uint32_t *d_expand_pos=NULL;
        RMM_TRY( RMM_ALLOC( &d_expand_pos,num_pair* sizeof(uint32_t), stream));
        assert(d_expand_pos!=NULL);
 	HANDLE_CUDA_ERROR( hipMemset(d_expand_pos,0,num_pair*sizeof(uint32_t)) ); 
 	
        uint32_t *d_quad_idx_new=NULL,*d_poly_idx_new=NULL;
        RMM_TRY( RMM_ALLOC( &d_quad_idx_new,num_pair* sizeof(uint32_t), stream));
        assert(d_quad_idx_new!=NULL);      
        RMM_TRY( RMM_ALLOC( &d_poly_idx_new,num_pair* sizeof(uint32_t), stream));
        assert(d_poly_idx_new!=NULL);
     
        uint8_t *d_pq_lev_new=NULL,*d_pq_type_new=NULL;
        RMM_TRY( RMM_ALLOC( &d_pq_lev_new,num_pair* sizeof(uint8_t), stream));
        assert(d_pq_lev_new!=NULL);
        RMM_TRY( RMM_ALLOC( &d_pq_type_new,num_pair* sizeof(uint8_t),stream));
        assert(d_pq_type_new!=NULL);
        
        auto pair_output_new_iter=thrust::make_zip_iterator(thrust::make_tuple(d_pq_lev_new,d_pq_type_new,d_poly_idx_new,d_quad_idx_new));   
        auto counting_iter=thrust::make_counting_iterator(0);
        thrust::exclusive_scan(exec_policy,d_quad_nchild,d_quad_nchild+num_nonleaf_pair,d_quad_nchild);
        thrust::scatter(exec_policy,counting_iter,counting_iter+num_nonleaf_pair,d_quad_nchild,d_expand_pos);        
        RMM_TRY(RMM_FREE(d_quad_nchild,stream));d_quad_nchild=NULL;    

        thrust::inclusive_scan(exec_policy,d_expand_pos,d_expand_pos+num_pair,d_expand_pos,thrust::maximum<int>());        
        thrust::gather(exec_policy,d_expand_pos,d_expand_pos+num_pair,pair_output_temp_iter,pair_output_new_iter);       
        
        uint32_t *d_seq_pos=NULL;
        RMM_TRY( RMM_ALLOC( &d_seq_pos,num_pair* sizeof(uint32_t), stream));
        assert(d_seq_pos!=NULL);
     
        thrust::exclusive_scan_by_key(exec_policy,d_expand_pos,d_expand_pos+num_pair,
       		thrust::constant_iterator<int>(1),d_seq_pos);
             	
        RMM_TRY(RMM_FREE(d_expand_pos,stream));d_expand_pos=NULL;        	
        auto update_quad_iter=thrust::make_zip_iterator(thrust::make_tuple(d_quad_idx_new,thrust::make_counting_iterator(0)));
        thrust::transform(exec_policy,update_quad_iter,update_quad_iter+num_pair,d_quad_idx_new,
       		update_quad(d_p_qtfpos,d_seq_pos));
       
        RMM_TRY(RMM_FREE(d_seq_pos,stream));d_seq_pos=NULL;      
        
        auto pq_pair_iterator=thrust::make_zip_iterator(thrust::make_tuple(d_poly_idx_new,d_quad_idx_new));
        uint32_t n=thrust::transform(exec_policy,pq_pair_iterator,pq_pair_iterator+num_pair,pair_output_new_iter,
       		twolist_test_intersection<T>(num_level,aoi_bbox,scale,d_p_qtkey,d_p_qtlev,d_p_qtsign,d_poly_sbbox))-pair_output_new_iter;
        printf("n=%d\n",n);

        num_leaf_pair=thrust::copy_if(exec_policy,pair_output_new_iter,pair_output_new_iter+num_pair,
     	        pair_output_iter+b_pos,qt_is_type(0))-(pair_output_iter+b_pos);
        num_nonleaf_pair=thrust::remove_if(exec_policy,pair_output_new_iter,pair_output_new_iter+num_pair,
                pair_output_new_iter,qt_not_type(1))-pair_output_new_iter;
        
        printf("level=%d num_leaf_pair=%d num_nonleaf_pair=%d\n",i,num_leaf_pair,num_nonleaf_pair);      
        b_pos+=num_leaf_pair;
 
     RMM_TRY( RMM_FREE(d_pq_lev_temp,stream) ); d_pq_lev_temp=d_pq_lev_new;  
     RMM_TRY( RMM_FREE(d_pq_type_temp,stream) ); d_pq_type_temp=d_pq_type_new;  
     RMM_TRY( RMM_FREE(d_poly_idx_temp,stream) ); d_poly_idx_temp=d_poly_idx_new;  
     RMM_TRY( RMM_FREE(d_quad_idx_temp,stream) ); d_quad_idx_temp=d_quad_idx_new;
     
     if(num_nonleaf_pair==0) 
      	break;

     pair_output_temp_iter=thrust::make_zip_iterator(thrust::make_tuple(d_pq_lev_temp,d_pq_type_temp,d_poly_idx_temp,d_quad_idx_temp));    

    //expand child nodes and get ready for the next round
    uint32_t max_num=b_pos+num_nonleaf_pair*4;
    if((i<num_level-1)&&(max_num>curr_len))
    {
       	 curr_len*=((max_num/curr_len)+1);
      	 printf("increasing capacity: level=%d bpos=%d max_num=%d curr_len=%d\n",i,b_pos,max_num,curr_len);
  
         uint8_t *d_pq_lev_new=NULL,*d_pq_type_new=NULL;
         RMM_TRY( RMM_ALLOC( &d_pq_lev_new,curr_len* sizeof(uint8_t), stream));
         assert(d_pq_lev_new!=NULL);
 	 HANDLE_CUDA_ERROR( hipMemcpy( (void *)d_pq_lev_new, (void *)d_pq_lev_out, b_pos * sizeof(uint8_t), hipMemcpyDeviceToDevice ) );                
         RMM_TRY( RMM_FREE (d_pq_lev_out,stream));      
         d_pq_lev_out=d_pq_lev_new;
         
         RMM_TRY( RMM_ALLOC( &d_pq_type_new,curr_len* sizeof(uint8_t), stream));
         assert(d_pq_type_new!=NULL);
   	 HANDLE_CUDA_ERROR( hipMemcpy( (void *)d_pq_type_new, (void *)d_pq_type_out, b_pos * sizeof(uint8_t), hipMemcpyDeviceToDevice ) );              
         RMM_TRY( RMM_FREE (d_pq_type_out,stream));
         d_pq_type_out=d_pq_type_new;
            	      	 
       	 uint32_t *d_quad_idx_new=NULL,*d_poly_idx_new=NULL;
   	 RMM_TRY( RMM_ALLOC( &d_quad_idx_new,curr_len* sizeof(uint32_t), stream));
         assert(d_quad_idx_new!=NULL);
         HANDLE_CUDA_ERROR( hipMemcpy( (void *)d_quad_idx_new, (void *)d_quad_idx_out, b_pos * sizeof(uint32_t), hipMemcpyDeviceToDevice ) );
         RMM_TRY( RMM_FREE (d_quad_idx_out,stream));
         d_quad_idx_out=d_quad_idx_new;
         
         RMM_TRY( RMM_ALLOC( &d_poly_idx_new,curr_len* sizeof(uint32_t), stream));
         assert(d_poly_idx_new!=NULL);
         HANDLE_CUDA_ERROR( hipMemcpy( (void *)d_poly_idx_new, (void *)d_poly_idx_out, b_pos * sizeof(uint32_t), hipMemcpyDeviceToDevice ) );
         RMM_TRY( RMM_FREE (d_poly_idx_out,stream));
         d_poly_idx_out=d_poly_idx_new;
   
         pair_output_iter=thrust::make_zip_iterator(thrust::make_tuple(d_pq_lev_out,d_pq_type_out,d_poly_idx_out,d_quad_idx_out));
       }  
       printf("level=%d b_pos=%d curr_len=%d\n",i,b_pos,curr_len);
    }
    printf("final: b_pos=%d\n",b_pos);
    CUDF_EXPECTS(b_pos<=curr_len,"out of boundary"); 
    RMM_TRY(RMM_FREE(d_poly_sbbox,stream));d_poly_sbbox=NULL;    

    /*std::unique_ptr<cudf::column> lev_col = cudf::make_numeric_column(
       cudf::data_type(cudf::type_id::INT8), b_pos,cudf::mask_state::UNALLOCATED,  stream, mr);      
    uint8_t *d_pq_lev=cudf::mutable_column_device_view::create(lev_col->mutable_view(), stream)->data<uint8_t>();
    CUDF_EXPECTS(d_pq_lev!=NULL,"lev"); 
    thrust::copy(exec_policy,d_pq_lev_out,d_pq_lev_out+b_pos,d_pq_lev);
 
    std::unique_ptr<cudf::column> type_col = cudf::make_numeric_column(
       cudf::data_type(cudf::type_id::INT8), b_pos,cudf::mask_state::UNALLOCATED,  stream, mr);      
    uint8_t *d_pq_type=cudf::mutable_column_device_view::create(type_col->mutable_view(), stream)->data<uint8_t>();
    CUDF_EXPECTS(d_pq_type!=NULL,"type"); 
    thrust::copy(exec_policy,d_pq_type_out,d_pq_type_out+b_pos,d_pq_type);*/
  
    std::unique_ptr<cudf::column> poly_idx_col = cudf::make_numeric_column(
       cudf::data_type(cudf::type_id::INT32), b_pos,cudf::mask_state::UNALLOCATED,  stream, mr);      
    uint32_t *d_pq_poly_idx=cudf::mutable_column_device_view::create(poly_idx_col->mutable_view(), stream)->data<uint32_t>();
    CUDF_EXPECTS(d_pq_poly_idx!=NULL,"poly_id"); 
    thrust::copy(exec_policy,d_poly_idx_out,d_poly_idx_out+b_pos,d_pq_poly_idx);
   
    std::unique_ptr<cudf::column> quad_idx_col = cudf::make_numeric_column(
       cudf::data_type(cudf::type_id::INT32), b_pos,cudf::mask_state::UNALLOCATED,  stream, mr);      
    uint32_t *d_pq_quad_idx=cudf::mutable_column_device_view::create(quad_idx_col->mutable_view(), stream)->data<uint32_t>();
    CUDF_EXPECTS(d_pq_quad_idx!=NULL,"quid_id"); 
    thrust::copy(exec_policy,d_quad_idx_out,d_quad_idx_out+b_pos,d_pq_quad_idx);
  
    RMM_FREE(d_pq_lev_out,stream);d_pq_lev_out=NULL;
    RMM_FREE(d_pq_type_out,stream);d_pq_lev_out=NULL;
    RMM_FREE(d_poly_idx_out,stream);d_poly_idx_out=NULL;
    RMM_FREE(d_quad_idx_out,stream);d_quad_idx_out=NULL;
  
  if(0)
 {
    printf("total pairs =%d\n",b_pos);

    /*thrust::device_ptr<uint8_t> d_pq_lev_ptr=thrust::device_pointer_cast(d_pq_lev);		
    printf("lev\n");
    thrust::copy(d_pq_lev_ptr,d_pq_lev_ptr+b_pos,std::ostream_iterator<uint32_t>(std::cout, " "));std::cout<<std::endl; 	

    thrust::device_ptr<uint8_t> d_pq_type_ptr=thrust::device_pointer_cast(d_pq_type);		
    printf("type\n");
    thrust::copy(d_pq_type_ptr,d_pq_type_ptr+b_pos,std::ostream_iterator<uint32_t>(std::cout, " "));std::cout<<std::endl;*/ 	

    thrust::device_ptr<uint32_t> d_poly_idx_ptr=thrust::device_pointer_cast(d_pq_poly_idx);		
    printf("d_ply_idx\n");
    thrust::copy(d_poly_idx_ptr,d_poly_idx_ptr+b_pos,std::ostream_iterator<uint32_t>(std::cout, " "));std::cout<<std::endl; 	

    thrust::device_ptr<uint32_t> d_quad_idx_ptr=thrust::device_pointer_cast(d_pq_quad_idx);		
    printf("d_quad_idx\n");
    thrust::copy(d_quad_idx_ptr,d_quad_idx_ptr+b_pos,std::ostream_iterator<uint32_t>(std::cout, " "));std::cout<<std::endl; 	   
 }     
 

  
   std::vector<std::unique_ptr<cudf::column>> pair_cols;
   //pair_cols.push_back(std::move(lev_col));
   //pair_cols.push_back(std::move(type_col));
   pair_cols.push_back(std::move(poly_idx_col));
   pair_cols.push_back(std::move(quad_idx_col));
   return pair_cols;    
}

struct quad_bbox_processor {
  
  template<typename T, std::enable_if_t<std::is_floating_point<T>::value >* = nullptr>
  std::unique_ptr<cudf::experimental::table> operator()(
	const cudf::table_view& quadtree,const cudf::table_view& poly_bbox,	
	quad_point_parameters qpi,
	rmm::mr::device_memory_resource* mr,
        hipStream_t stream)
   {    
       double x1=thrust::get<0>(qpi);
       double y1=thrust::get<1>(qpi);
       double x2=thrust::get<2>(qpi);
       double y2=thrust::get<3>(qpi);
       SBBox<double> aoi_bbox(thrust::make_tuple(x1,y1),thrust::make_tuple(x2,y2));
       std::cout<<"quadtree_poly.aoi:"<<x1<<" "<<y1<<" "<<x2<<" "<<y2<<std::endl;
       double scale=thrust::get<4>(qpi);
       uint32_t num_level=thrust::get<5>(qpi);
       uint32_t min_size=thrust::get<6>(qpi);    
       
       const uint32_t *d_p_qtkey=    quadtree.column(0).data<uint32_t>();
       const uint8_t  *d_p_qtlev=    quadtree.column(1).data<uint8_t>();
       const bool     *d_p_qtsign=   quadtree.column(2).data<bool>();
       const uint32_t *d_p_qtlength= quadtree.column(3).data<uint32_t>();
       const uint32_t *d_p_qtfpos=   quadtree.column(4).data<uint32_t>();
   
       const T *poly_x1=poly_bbox.column(0).data<T>();
       const T *poly_y1=poly_bbox.column(1).data<T>();
       const T *poly_x2=poly_bbox.column(2).data<T>();
       const T *poly_y2=poly_bbox.column(3).data<T>();     
       
       uint32_t num_node=quadtree.num_rows();
       uint32_t num_poly=poly_bbox.num_rows();
       std::vector<std::unique_ptr<cudf::column>> pair_cols=
       		dowork(num_node,d_p_qtkey,d_p_qtlev,d_p_qtsign,d_p_qtlength,d_p_qtfpos,
       		num_poly,poly_x1,poly_y1,poly_x2,poly_y2,
       		aoi_bbox,scale,num_level,min_size,mr,stream);
       	
      std::unique_ptr<cudf::experimental::table> destination_table = 
    	std::make_unique<cudf::experimental::table>(std::move(pair_cols));      
      
      return destination_table;
    }
  
  template<typename T, std::enable_if_t<!std::is_floating_point<T>::value >* = nullptr>
  std::unique_ptr<cudf::experimental::table> operator()(
	const cudf::table_view& quadtree,const cudf::table_view& bbox,	
	quad_point_parameters qpi,
	rmm::mr::device_memory_resource* mr,
        hipStream_t stream)
    {
 	CUDF_FAIL("Non-floating point operation is not supported");
    }  
      
};
  
} //end anonymous namespace

namespace cuspatial {
std::unique_ptr<cudf::experimental::table> quad_bbox_join(
	cudf::table_view const& quadtree,cudf::table_view const& poly_bbox,
	double x1,double y1,double x2,double y2, double scale, uint32_t num_level, uint32_t min_size)
{   
    hipStream_t stream=0;
    rmm::mr::device_memory_resource* mr=rmm::mr::get_default_resource();
    quad_point_parameters qpi=thrust::make_tuple(x1,y1,x2,y2,scale,num_level,min_size);
    cudf::data_type dtype=poly_bbox.column(0).type();
    
    return cudf::experimental::type_dispatcher(dtype,quad_bbox_processor{}, 
    	quadtree,poly_bbox,qpi, mr,stream);   
    	    	
}

}// namespace cuspatial
