#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <thrust/gather.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/iterator/discard_iterator.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/reduce.h>

#include <vector>
#include <cudf/column/column.hpp>
#include <cudf/column/column_view.hpp>
#include <cudf/column/column_factories.hpp>
#include <cudf/column/column_device_view.cuh>
#include <cudf/table/table_view.hpp>
#include <cudf/table/table.hpp>

#include <utility/helper_thrust.cuh>
#include <utility/quadtree_thrust.cuh>
#include <utility/bbox_thrust.cuh>
#include <cuspatial/bounding_box.hpp>
#include <cuspatial/spatial_jion.hpp>

namespace
{

typedef thrust::tuple<double, double,double,double,double,uint32_t,uint32_t> quad_point_parameters;

template<typename T>
std::vector<std::unique_ptr<cudf::column>> dowork(
    uint32_t num_node,const uint32_t *d_p_qtkey,const uint8_t *d_p_qtlev,
    const bool *d_p_qtsign, const uint32_t *d_p_qtlength, const uint32_t *d_p_qtfpos,
    const uint32_t num_poly,const T *poly_x1, const T *poly_y1,T const *poly_x2, const T *poly_y2,
    const SBBox<double>& aoi_bbox, double scale,uint32_t num_level, uint32_t min_size, 
    rmm::mr::device_memory_resource* mr, hipStream_t stream)    
                                         
{
    double x1=thrust::get<0>(aoi_bbox.first);
    double y1=thrust::get<1>(aoi_bbox.first);
    double x2=thrust::get<0>(aoi_bbox.second);
    double y2=thrust::get<1>(aoi_bbox.second);

    std::cout<<"num_node="<<num_node<<std::endl;
    std::cout<<"num_poly="<<num_poly<<std::endl;
    std::cout<<"bounding box(x1,y1,x2,y2)=("<<x1<<","<<y1<<","<<x2<<","<<x2<<","<<y2<<std::endl;
    std::cout<<"scale="<<scale<<std::endl;
    std::cout<<"num_level="<<num_level<<std::endl;
    std::cout<<"match: min_size="<<min_size<<std::endl;

    auto exec_policy = rmm::exec_policy(stream);

    rmm::device_buffer *db_poly_bbox=new rmm::device_buffer(num_poly* sizeof(SBBox<T>),stream,mr);
    CUDF_EXPECTS(db_poly_bbox!=nullptr, "Error allocating memory for polygon bounding boxes on device");
    SBBox<T> *d_poly_sbbox=static_cast<SBBox<T> *>(db_poly_bbox->data());

 if(0)
 {
    std::cout<<"x1"<<std::endl;
    thrust::device_ptr<const T> d_x1_ptr = thrust::device_pointer_cast(poly_x1);
    thrust::copy(d_x1_ptr,d_x1_ptr+num_poly,std::ostream_iterator<T>(std::cout, " "));std::cout<<std::endl;

    std::cout<<"y1"<<std::endl;
    thrust::device_ptr<const T> d_y1_ptr = thrust::device_pointer_cast(poly_y1);
    thrust::copy(d_y1_ptr,d_y1_ptr+num_poly,std::ostream_iterator<T>(std::cout, " "));std::cout<<std::endl;

    std::cout<<"x2"<<std::endl;
    thrust::device_ptr<const T> d_x2_ptr = thrust::device_pointer_cast(poly_x2);
    thrust::copy(d_x2_ptr,d_x2_ptr+num_poly,std::ostream_iterator<T>(std::cout, " "));std::cout<<std::endl;

    std::cout<<"y2"<<std::endl;
    thrust::device_ptr<const T> d_y2_ptr = thrust::device_pointer_cast(poly_y2);
    thrust::copy(d_y2_ptr,d_y2_ptr+num_poly,std::ostream_iterator<T>(std::cout, " "));std::cout<<std::endl;
}

if(0)
{
   std::cout<<"qt lev"<<std::endl;
   thrust::device_ptr<const uint8_t> d_lev_ptr=thrust::device_pointer_cast(d_p_qtlev);
   thrust::copy(d_lev_ptr,d_lev_ptr+num_node,std::ostream_iterator<uint32_t>(std::cout, " "));std::cout<<std::endl;

   std::cout<<"qt sign"<<std::endl;
   thrust::device_ptr<const bool> d_sign_ptr=thrust::device_pointer_cast(d_p_qtsign);
   thrust::copy(d_sign_ptr,d_sign_ptr+num_node,std::ostream_iterator<uint32_t>(std::cout, " "));std::cout<<std::endl;
}

   //assemble arrays of columns to an array of bbox (x1,y1,x2,y2) 
    auto ploy_bbox_iter=thrust::make_zip_iterator(
        thrust::make_tuple(poly_x1,poly_y1,poly_x2,poly_y2));
    thrust::transform(exec_policy->on(stream),ploy_bbox_iter,ploy_bbox_iter+num_poly,d_poly_sbbox,tuple2bbox<T>());

    //couting the number of top level nodes to begin with
    //the number could be stored explicitly, but count_if should be fast enough
    uint32_t num_top_lev_children=thrust::count_if(exec_policy->on(stream),d_p_qtlev,
        d_p_qtlev+num_node,thrust::placeholders::_1==0);  

    //the matched quadrant-polygon pairs are dynamic and can not be pre-allocated in a fixed manner
    // relevant arrays are resized accordingly for memory efficiency 
    
    //{_lev,_type,_poly_idx,_quad_idx}_out are for outputs for matched paris with an initial capcity of init_len
    //{_lev,_type,_poly_idx,_quad_idx}_increased are for resized storage for outputs,
    //condering the maximum number of possible matched pairs at the next level. 
    //The *_increased arrays are only resized as necessary 

    //{_lev,_type,_poly_idx,_quad_idx}_temp are for temporal stroage at a level
    //{_lev,_type,_poly_idx,_quad_idx}_expanded are for expanded stroage at the next level
    //their size is computed precisely by retriving the numbers of child nodes for all non-leaf quadrants
    
    uint32_t init_len=1000;
    uint32_t curr_cap=init_len;

    rmm::device_buffer *db_pq_lev_out = new rmm::device_buffer(curr_cap* sizeof(uint8_t),stream,mr);
    CUDF_EXPECTS(db_pq_lev_out!=nullptr, "Error allocating memory for permanent level array on device");
    uint8_t *d_pq_lev_out=static_cast<uint8_t *>(db_pq_lev_out->data());

    rmm::device_buffer *db_pq_type_out = new rmm::device_buffer(curr_cap* sizeof(uint8_t),stream,mr);
    CUDF_EXPECTS(db_pq_type_out!=nullptr, "Error allocating memory for permanent type array on device");        
    uint8_t *d_pq_type_out=static_cast<uint8_t *>(db_pq_type_out->data());

    rmm::device_buffer *db_poly_idx_out = new rmm::device_buffer(curr_cap* sizeof(uint32_t),stream,mr);
    CUDF_EXPECTS(db_poly_idx_out!=nullptr, "Error allocating memory for permanent polygon index array on device");
    uint32_t *d_poly_idx_out=static_cast<uint32_t *>(db_poly_idx_out->data());

    rmm::device_buffer *db_quad_idx_out = new rmm::device_buffer(curr_cap* sizeof(uint32_t),stream,mr);
    CUDF_EXPECTS(db_quad_idx_out!=nullptr, "Error allocating memory for permanent quadrant index array on device");
    uint32_t *d_quad_idx_out=static_cast<uint32_t *>(db_quad_idx_out->data());

    auto pair_output_iter=thrust::make_zip_iterator(
        thrust::make_tuple(d_pq_lev_out,d_pq_type_out,d_poly_idx_out,d_quad_idx_out));

    uint32_t output_nodes_pos=0;

    uint32_t  num_pair=num_top_lev_children*num_poly;
    
    std::cout<<"num_top_lev_children="<<num_top_lev_children<<std::endl;
    std::cout<<"num_poly="<<num_poly<<std::endl;
    std::cout<<"num_pair="<<num_pair<<std::endl;

    rmm::device_buffer *db_pq_lev_temp = new rmm::device_buffer(num_pair* sizeof(uint8_t),stream,mr);
    CUDF_EXPECTS(db_pq_lev_temp!=nullptr, "Error allocating memory for temporal level array on device");
    uint8_t *d_pq_lev_temp=static_cast<uint8_t *>(db_pq_lev_temp->data());

    rmm::device_buffer *db_pq_type_temp = new rmm::device_buffer(num_pair* sizeof(uint8_t),stream,mr);
    CUDF_EXPECTS(db_pq_type_temp!=nullptr, "Error allocating memory for temporal type level array on device");
    uint8_t *d_pq_type_temp=static_cast<uint8_t *>(db_pq_type_temp->data());

    rmm::device_buffer *db_poly_idx_temp = new rmm::device_buffer(num_pair* sizeof(uint32_t),stream,mr);
    CUDF_EXPECTS(db_poly_idx_temp!=nullptr, "Error allocating memory for temporal polygon index array on device");
    uint32_t *d_poly_idx_temp=static_cast<uint32_t *>(db_poly_idx_temp->data());

    rmm::device_buffer *db_quad_idx_temp = new rmm::device_buffer(num_pair* sizeof(uint32_t),stream,mr);
    CUDF_EXPECTS(db_quad_idx_temp!=nullptr, "Error allocating memory for temporal quadrant index array on device");
    uint32_t *d_quad_idx_temp=static_cast<uint32_t *>(db_quad_idx_temp->data());
    
    auto pair_counting_iter=thrust::make_counting_iterator(0);
    auto pair_output_temp_iter=thrust::make_zip_iterator(
        thrust::make_tuple(d_pq_lev_temp,d_pq_type_temp,d_poly_idx_temp,d_quad_idx_temp));

    //paring up all top level quadrants and all polygons and store the result in pair_output_temp_iter
    thrust::transform(exec_policy->on(stream),pair_counting_iter,pair_counting_iter+num_pair,pair_output_temp_iter,
        pairwise_test_intersection<T>(num_level,num_top_lev_children,aoi_bbox,scale,d_p_qtkey,d_p_qtlev,d_p_qtsign,d_poly_sbbox));

    //copy intersected (quadrant,polygon) pairs that involve leaf qudrants to outputs directly (type 0)
    uint32_t num_leaf_pair=thrust::copy_if(exec_policy->on(stream),pair_output_temp_iter,pair_output_temp_iter+num_pair,
        pair_output_iter+output_nodes_pos,qt_is_type(0))-(pair_output_iter+output_nodes_pos);

    //remove all the (quadrant,polygon) pairs that quadrants do not intersect with polygon bboxes
    uint32_t num_nonleaf_pair=thrust::remove_if(exec_policy->on(stream),pair_output_temp_iter,pair_output_temp_iter+num_pair,
        pair_output_temp_iter,qt_not_type(1))-pair_output_temp_iter;

    std::cout<<"num_leaf_pair="<<num_leaf_pair<<" ,num_nonleaf_pair="<<num_nonleaf_pair<<std::endl;

    output_nodes_pos+=num_leaf_pair; 
    
    //loopin through all the rest of levels
    for(uint32_t i=1;i<num_level;i++)
    {
        //allocate memory for numbers of child nodes for all non-leaf quadrants
        rmm::device_buffer *db_quad_nchild = new rmm::device_buffer(num_nonleaf_pair* sizeof(uint32_t),stream,mr);
        CUDF_EXPECTS(db_quad_nchild!=nullptr, "Error allocating memory for number of child nodes array on device");
        uint32_t *d_quad_nchild=static_cast<uint32_t *>(db_quad_nchild->data());

        //retrieve the numbers of child quadrants and store them in d_quad_nchild
        thrust::transform(exec_policy->on(stream),d_quad_idx_temp,d_quad_idx_temp+num_nonleaf_pair,
              d_quad_nchild,get_vec_element<const uint32_t>(d_p_qtlength));

        //compute the total number of child nodes using a reduction
        num_pair=thrust::reduce(exec_policy->on(stream),d_quad_nchild,d_quad_nchild+num_nonleaf_pair);
        std::cout<<"num_pair after gathering child nodes="<<num_pair<<std::endl;

        //allocate memory for the next level 
        
        rmm::device_buffer *db_expand_pos = new rmm::device_buffer(num_pair* sizeof(uint32_t),stream,mr);
        CUDF_EXPECTS(db_expand_pos!=nullptr, "Error allocating memory for expanded offset array on device");
        uint32_t *d_expand_pos=static_cast<uint32_t *>(db_expand_pos->data());
        HANDLE_CUDA_ERROR( hipMemset(d_expand_pos,0,num_pair*sizeof(uint32_t)) );

        rmm::device_buffer *db_pq_lev_expanded = new rmm::device_buffer(num_pair* sizeof(uint8_t),stream,mr);
        CUDF_EXPECTS(db_pq_lev_expanded!=nullptr, "Error allocating memory for expanded lev array on device");
        uint8_t *d_pq_lev_expanded=static_cast<uint8_t *>(db_pq_lev_expanded->data());

        rmm::device_buffer *db_pq_type_expanded = new rmm::device_buffer(num_pair* sizeof(uint8_t),stream,mr);
        CUDF_EXPECTS(db_pq_type_expanded!=nullptr, "Error allocating memory for expanded type level array on device");
        uint8_t *d_pq_type_expanded=static_cast<uint8_t *>(db_pq_type_expanded->data());

        rmm::device_buffer *db_poly_idx_expanded = new rmm::device_buffer(num_pair* sizeof(uint32_t),stream,mr);
        CUDF_EXPECTS(db_poly_idx_expanded!=nullptr, "Error allocating memory for expanded polygon index array on device");
        uint32_t *d_poly_idx_expanded=static_cast<uint32_t *>(db_poly_idx_expanded->data());

        rmm::device_buffer *db_quad_idx_expanded = new rmm::device_buffer(num_pair* sizeof(uint32_t),stream,mr);
        CUDF_EXPECTS(db_quad_idx_expanded!=nullptr, "Error allocating memory for expanded quadrant index array on device");
        uint32_t *d_quad_idx_expanded=static_cast<uint32_t *>(db_quad_idx_expanded->data());

        //exclusive scan on the numbers to compute the offsets 
        auto counting_iter=thrust::make_counting_iterator(0);
            thrust::exclusive_scan(exec_policy->on(stream),d_quad_nchild,d_quad_nchild+num_nonleaf_pair,d_quad_nchild);

        //use the offset as the map to scatter sequential numbers 0..num_nonleaf_pair to d_expand_pos
        thrust::scatter(exec_policy->on(stream),counting_iter,counting_iter+num_nonleaf_pair,d_quad_nchild,d_expand_pos);

        //d_quad_nchild is no longer needed, so delete its asociated device_buffer and release memory
        delete db_quad_nchild; db_quad_nchild=nullptr;

        //inclusive scan with maximum functor to fill the empty elements with their left-most non-empty elements
        //d_expand_pos is now a full array with each element stores the sequene idx of a quadrant's parent
        thrust::inclusive_scan(exec_policy->on(stream),d_expand_pos,d_expand_pos+num_pair,d_expand_pos,thrust::maximum<int>());

        //assemble the {_lev,_type,_poly_id,_quad_id) arrays as a zipped iterator
        auto pair_output_expanded_iter=thrust::make_zip_iterator(thrust::make_tuple
            (d_pq_lev_expanded,d_pq_type_expanded,d_poly_idx_expanded,d_quad_idx_expanded));

        //use d_expand_pos as the map to gather info on non-leaf quadrants for their respective child quadrants
        thrust::gather(exec_policy->on(stream),d_expand_pos,d_expand_pos+num_pair,pair_output_temp_iter,pair_output_expanded_iter);
 
        //generate sequential idx within each parent quadrants; used with fpos array to retrieve child quadrants
        rmm::device_buffer *db_seq_pos = new rmm::device_buffer(num_pair* sizeof(uint32_t),stream,mr);
        CUDF_EXPECTS(db_seq_pos!=nullptr, "Error allocating memory for sequence index  array on device");
        uint32_t *d_seq_pos=static_cast<uint32_t *>(db_seq_pos->data());

        thrust::exclusive_scan_by_key(exec_policy->on(stream),d_expand_pos,d_expand_pos+num_pair,
            thrust::constant_iterator<int>(1),d_seq_pos);

        //d_expand_pos is no long needed; delete associated device_buffer and release memory
        delete db_expand_pos; db_expand_pos=nullptr;
        
        //retrieve child quadrants, given fpos of paranet quadrants (d_p_qtfpos) and offsets child quarants  
        auto update_quad_iter=thrust::make_zip_iterator(thrust::make_tuple(
            d_quad_idx_expanded,thrust::make_counting_iterator(0)));     
        thrust::transform(exec_policy->on(stream),update_quad_iter,update_quad_iter+num_pair,d_quad_idx_expanded,
            update_quad(d_p_qtfpos,d_seq_pos));

        //d_seq_pos is no long needed; delete related device_buffer 
        delete db_seq_pos; db_seq_pos=nullptr;

        //testing intersection of quadrnats and polygon bboxes, results stored in d_pq_type_expanded
        //three possible types: intersection and leaf nodes==>0, intersection and non-leaf nodes==>1, non-intersection==>2
        //pair_output_expanded_iter has four components; polygon/quadrant idx repeated to work with copy_if/remove_if next 
        auto pq_pair_iterator=thrust::make_zip_iterator(thrust::make_tuple(d_poly_idx_expanded,d_quad_idx_expanded));
        thrust::transform(exec_policy->on(stream),pq_pair_iterator,pq_pair_iterator+num_pair,pair_output_expanded_iter,
            twolist_test_intersection<T>(num_level,aoi_bbox,scale,d_p_qtkey,d_p_qtlev,d_p_qtsign,d_poly_sbbox));

        //copy type 0 (intersection and leaf nodes) to output directly 
        num_leaf_pair=thrust::copy_if(exec_policy->on(stream),pair_output_expanded_iter,pair_output_expanded_iter+num_pair,
            pair_output_iter+output_nodes_pos,qt_is_type(0))-(pair_output_iter+output_nodes_pos);

        //keep type 1(intersection and non-leaf nodes) only 
        num_nonleaf_pair=thrust::remove_if(exec_policy->on(stream),pair_output_expanded_iter,pair_output_expanded_iter+num_pair,
            pair_output_expanded_iter,qt_not_type(1))-pair_output_expanded_iter;

        std::cout<<"level="<<i<<std::endl;
        std::cout<<"num_leaf_pair="<<num_leaf_pair<<std::endl;
        std::cout<<"num_nonleaf_pair="<<num_nonleaf_pair<<std::endl;

        //update numbers of pairs in the output
        output_nodes_pos+=num_leaf_pair;

        //release device buffers for parent quadrants 
        //and update pointers to device buffers and arrays to point to child quadrants 
        delete db_pq_lev_temp; db_pq_lev_temp=db_pq_lev_expanded; 
        d_pq_lev_temp=d_pq_lev_expanded;
        delete db_pq_type_temp; db_pq_type_temp=db_pq_type_expanded; 
        d_pq_type_temp=d_pq_type_expanded;
        delete db_poly_idx_temp; db_poly_idx_temp=db_poly_idx_expanded; 
        d_poly_idx_temp=d_poly_idx_expanded;
        delete db_quad_idx_temp; db_quad_idx_temp=db_quad_idx_expanded; 
        d_quad_idx_temp=d_quad_idx_expanded;

        //stop level-wise iteration if no more non-leaf quadrants to expand
        if(num_nonleaf_pair==0) 
            break;

        //update pair_output_temp_iter to get ready for next level iteration
        pair_output_temp_iter=thrust::make_zip_iterator(thrust::make_tuple(
            d_pq_lev_temp,d_pq_type_temp,d_poly_idx_temp,d_quad_idx_temp));

        //resize device buffers for storing output and update the corresponding pointers
        //the next level will add no more than num_nonleaf_pair*4 pairs
        //as a parent quadrant has no more than 4 child quadrants
        uint32_t max_num=output_nodes_pos+num_nonleaf_pair*4;

        if((i<num_level-1)&&(max_num>curr_cap))
        {
            curr_cap*=((max_num/curr_cap)+1);
            std::cout<<"increasing output capacity: level="<<i<<" to "<<curr_cap<<std::endl;

            rmm::device_buffer *db_pq_lev_increased = new rmm::device_buffer(curr_cap* sizeof(uint8_t),stream,mr);
            CUDF_EXPECTS(db_pq_lev_increased!=nullptr, "Error allocating memory for increased lev array on device");
            uint8_t * d_pq_lev_increased=static_cast<uint8_t *>(db_pq_lev_increased->data());
            HANDLE_CUDA_ERROR( hipMemcpy( (void *)d_pq_lev_increased, (void *)d_pq_lev_out, 
                output_nodes_pos * sizeof(uint8_t), hipMemcpyDeviceToDevice ) ); 
            delete db_pq_lev_out;db_pq_lev_out=db_pq_lev_increased;
            d_pq_lev_out=d_pq_lev_increased;

            rmm::device_buffer *db_pq_type_increased = new rmm::device_buffer(curr_cap* sizeof(uint8_t),stream,mr);
            CUDF_EXPECTS(db_pq_type_increased!=nullptr,"Error allocating memory for increased type array on device");
            uint8_t * d_pq_type_increased=static_cast<uint8_t *>(db_pq_type_increased->data());
            HANDLE_CUDA_ERROR( hipMemcpy( (void *)d_pq_type_increased, (void *)d_pq_type_out, 
                output_nodes_pos * sizeof(uint8_t), hipMemcpyDeviceToDevice ) );
            delete db_pq_type_out;db_pq_type_out=db_pq_type_increased;
            d_pq_type_out=d_pq_type_increased;

            rmm::device_buffer *db_quad_idx_increased = new rmm::device_buffer(curr_cap* sizeof(uint32_t),stream,mr);
            CUDF_EXPECTS(db_quad_idx_increased!=nullptr,"Error allocating memory for increased quad index array on device");
            uint32_t * d_quad_idx_increased=static_cast<uint32_t *>(db_quad_idx_increased->data());
            HANDLE_CUDA_ERROR( hipMemcpy( (void *)d_quad_idx_increased, (void *)d_quad_idx_out, 
                output_nodes_pos * sizeof(uint32_t), hipMemcpyDeviceToDevice ) );
            delete db_quad_idx_out; db_quad_idx_out=db_quad_idx_increased;
            d_quad_idx_out=d_quad_idx_increased;

            rmm::device_buffer *db_poly_idx_increased = new rmm::device_buffer(curr_cap* sizeof(uint32_t),stream,mr);
            CUDF_EXPECTS(db_poly_idx_increased!=nullptr,"Error allocating memory for increased polygon index array on device");
            uint32_t * d_poly_idx_increased=static_cast<uint32_t *>(db_poly_idx_increased->data());
            HANDLE_CUDA_ERROR( hipMemcpy( (void *)d_poly_idx_increased, (void *)d_poly_idx_out, 
                 output_nodes_pos * sizeof(uint32_t), hipMemcpyDeviceToDevice ) );
            delete db_poly_idx_out; db_poly_idx_out=db_poly_idx_increased;
            d_poly_idx_out=d_poly_idx_increased;

            //update pair_output_iter
            pair_output_iter=thrust::make_zip_iterator(thrust::make_tuple(
                d_pq_lev_out,d_pq_type_out,d_poly_idx_out,d_quad_idx_out));
        }
        std::cout<<"level="<<i<<" output_nodes_pos="<<output_nodes_pos<<" curr_cap="<<curr_cap<<std::endl;
    }
    std::cout<<"final: output_nodes_pos="<<output_nodes_pos<<std::endl;
    CUDF_EXPECTS(output_nodes_pos<=curr_cap,"output arrays: out of boundary"); 

    //d_poly_bbox is no longer needed, delete the associated device buffer and relase memory 
    delete db_poly_bbox; db_poly_bbox=nullptr;

    //allocate columns for paris of polygon offsets and quadrant offsets as the final output
    //lev and type are not needed in the output
    //note only the first output_nodes_pos elements are copied to output columns
    std::unique_ptr<cudf::column> poly_idx_col = cudf::make_numeric_column(
       cudf::data_type(cudf::type_id::INT32), output_nodes_pos,cudf::mask_state::UNALLOCATED,  stream, mr);
    uint32_t *d_pq_poly_idx=cudf::mutable_column_device_view::create(poly_idx_col->mutable_view(), stream)->data<uint32_t>();
    CUDF_EXPECTS(d_pq_poly_idx!=nullptr,"Error in accessing data array of polygon index column"); 
    thrust::copy(exec_policy->on(stream),d_poly_idx_out,d_poly_idx_out+output_nodes_pos,d_pq_poly_idx);

    std::unique_ptr<cudf::column> quad_idx_col = cudf::make_numeric_column(
       cudf::data_type(cudf::type_id::INT32), output_nodes_pos,cudf::mask_state::UNALLOCATED,  stream, mr);
    uint32_t *d_pq_quad_idx=cudf::mutable_column_device_view::create(quad_idx_col->mutable_view(), stream)->data<uint32_t>();
    CUDF_EXPECTS(d_pq_quad_idx!=nullptr,"Error in accessing data array of quadrant index column"); 
    thrust::copy(exec_policy->on(stream),d_quad_idx_out,d_quad_idx_out+output_nodes_pos,d_pq_quad_idx);

    //the output arrays are no longer needed; delete device buffers and release memory
    delete db_pq_lev_out; db_pq_lev_out=nullptr;
    delete db_pq_type_out; db_pq_lev_out=nullptr;
    delete db_poly_idx_out; db_poly_idx_out=nullptr;
    delete db_quad_idx_out; db_quad_idx_out=nullptr;

if(0)
{
    std::cout<<"total pairs="<<output_nodes_pos<<std::endl;

    thrust::device_ptr<uint32_t> d_poly_idx_ptr=thrust::device_pointer_cast(d_pq_poly_idx);
    std::cout<<"poly id of poly-quad pairs"<<std::endl;
    thrust::copy(d_poly_idx_ptr,d_poly_idx_ptr+output_nodes_pos,std::ostream_iterator<uint32_t>(std::cout, " "));std::cout<<std::endl;

    thrust::device_ptr<uint32_t> d_quad_idx_ptr=thrust::device_pointer_cast(d_pq_quad_idx);
    std::cout<<"quadrant id of poly-quad pairs"<<std::endl;
    thrust::copy(d_quad_idx_ptr,d_quad_idx_ptr+output_nodes_pos,std::ostream_iterator<uint32_t>(std::cout, " "));std::cout<<std::endl;
}

    std::vector<std::unique_ptr<cudf::column>> pair_cols;
    pair_cols.push_back(std::move(poly_idx_col));
    pair_cols.push_back(std::move(quad_idx_col));
    return pair_cols;
}

struct quad_bbox_processor {
    template<typename T, std::enable_if_t<std::is_floating_point<T>::value >* = nullptr>
    std::unique_ptr<cudf::experimental::table> operator()(
       const cudf::table_view& quadtree,const cudf::table_view& poly_bbox,    
       quad_point_parameters qpi,
       rmm::mr::device_memory_resource* mr,
        hipStream_t stream)
    {
        double x1=thrust::get<0>(qpi);
        double y1=thrust::get<1>(qpi);
        double x2=thrust::get<2>(qpi);
        double y2=thrust::get<3>(qpi);
        SBBox<double> aoi_bbox(thrust::make_tuple(x1,y1),thrust::make_tuple(x2,y2));
        std::cout<<"quadtree_poly.aoi:"<<x1<<" "<<y1<<" "<<x2<<" "<<y2<<std::endl;
        double scale=thrust::get<4>(qpi);
        uint32_t num_level=thrust::get<5>(qpi);
        uint32_t min_size=thrust::get<6>(qpi);

        const uint32_t *d_p_qtkey=    quadtree.column(0).data<uint32_t>();
        const uint8_t  *d_p_qtlev=    quadtree.column(1).data<uint8_t>();
        const bool     *d_p_qtsign=   quadtree.column(2).data<bool>();
        const uint32_t *d_p_qtlength= quadtree.column(3).data<uint32_t>();
        const uint32_t *d_p_qtfpos=   quadtree.column(4).data<uint32_t>();

        const T *poly_x1=poly_bbox.column(0).data<T>();
        const T *poly_y1=poly_bbox.column(1).data<T>();
        const T *poly_x2=poly_bbox.column(2).data<T>();
        const T *poly_y2=poly_bbox.column(3).data<T>();

        uint32_t num_node=quadtree.num_rows();
        uint32_t num_poly=poly_bbox.num_rows();

        std::vector<std::unique_ptr<cudf::column>> pair_cols=dowork(
            num_node,d_p_qtkey,d_p_qtlev,d_p_qtsign,d_p_qtlength,d_p_qtfpos,
            num_poly,poly_x1,poly_y1,poly_x2,poly_y2,
            aoi_bbox,scale,num_level,min_size,mr,stream);

        std::unique_ptr<cudf::experimental::table> destination_table =
        std::make_unique<cudf::experimental::table>(std::move(pair_cols));

        return destination_table;
    }

  template<typename T, std::enable_if_t<!std::is_floating_point<T>::value >* = nullptr>
  std::unique_ptr<cudf::experimental::table> operator()(
    const cudf::table_view& quadtree,const cudf::table_view& bbox,
    quad_point_parameters qpi,
    rmm::mr::device_memory_resource* mr,
        hipStream_t stream)
    {
     CUDF_FAIL("Non-floating point operation is not supported");
    }  

};
  
} //end anonymous namespace

namespace cuspatial {

std::unique_ptr<cudf::experimental::table> quad_bbox_join(
    cudf::table_view const& quadtree,cudf::table_view const& poly_bbox,
    double x1,double y1,double x2,double y2, double scale, uint32_t num_level, uint32_t min_size)
{   
 
    CUDF_EXPECTS(quadtree.num_columns()==5,"quadtree table must have 5 columns");  
    
    CUDF_EXPECTS(poly_bbox.num_columns()==4,"polygon bbox table must have 4 columns");  
    
    CUDF_EXPECTS(quadtree.num_rows()>0 && poly_bbox.num_rows()>0,
        "neither quadtree table nor polygon bbox table can be empty");
    
    CUDF_EXPECTS(x1<x2 && y1<y2, "invalid bounding box (x1,y1,x2,y2)");
    
    CUDF_EXPECTS(scale>0, "scale must be positive");
    
    CUDF_EXPECTS(num_level<16, "maximum of levels might be in [0,16)");
    
    CUDF_EXPECTS(min_size>0, "minimum number of points for a non-leaf node must be larger than zero"); 
   
    hipStream_t stream=0;
    rmm::mr::device_memory_resource* mr=rmm::mr::get_default_resource();

    quad_point_parameters qpi=thrust::make_tuple(x1,y1,x2,y2,scale,num_level,min_size);

    cudf::data_type dtype=poly_bbox.column(0).type();

    return cudf::experimental::type_dispatcher(dtype,quad_bbox_processor{}, 
        quadtree,poly_bbox,qpi, mr,stream);
}

}// namespace cuspatial
