#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <vector>
#include <cudf/column/column.hpp>
#include <cudf/column/column_view.hpp>
#include <cudf/column/column_factories.hpp>
#include <cudf/column/column_device_view.cuh>
#include <cudf/table/table_view.hpp>
#include <cudf/table/table.hpp>
#include <cudf/utilities/legacy/type_dispatcher.hpp>
#include <cudf/utilities/type_dispatcher.hpp>
#include <cuspatial/quadtree.hpp>
#include <utility/helper_thrust.cuh>
#include <utility/quadtree_thrust.cuh>


typedef thrust::tuple<double, double,double,double,double,uint32_t,uint32_t> quad_point_parameters;

namespace { //anonymous

/*
 *quadtree indexing on points using the bottom-up algorithm described at ref.
 *http://www.adms-conf.org/2019-camera-ready/zhang_adms19.pdf
*/

template<typename T>
std::vector<std::unique_ptr<cudf::column>> dowork(cudf::size_type point_len,
    T *d_pnt_x,T *d_pnt_y,SBBox<double> bbox, double scale,
    uint32_t num_level, uint32_t min_size, rmm::mr::device_memory_resource* mr, hipStream_t stream)
                                         
{
    double x1=thrust::get<0>(bbox.first);
    double y1=thrust::get<1>(bbox.first);
    double x2=thrust::get<0>(bbox.second);
    double y2=thrust::get<1>(bbox.second);
    
    std::cout<<"indexing space bounding box(x1,y1,x2,y2)=("<<x1<<","<<y1<<","<<x2<<","<<y2<<")"<<std::endl;
    std::cout<<"scale="<<scale<<std::endl;
    std::cout<<"point_len="<<point_len<<std::endl;
    std::cout<<"num_level="<<num_level<<std::endl;
    std::cout<<"min_size="<<min_size<<std::endl;
    
    auto exec_policy = rmm::exec_policy(stream)->on(stream);
    
//debugging: make sure the inputs are correct
if(0)
{
    thrust::device_ptr<T> d_x_ptr=thrust::device_pointer_cast(d_pnt_x);	
    thrust::device_ptr<T> d_y_ptr=thrust::device_pointer_cast(d_pnt_y);   
    
    std::cout<<"x:"<<std::endl;
    thrust::copy(d_x_ptr,d_x_ptr+point_len,std::ostream_iterator<T>(std::cout, " "));std::cout<<std::endl;
    std::cout<<"x:"<<std::endl;
    thrust::copy(d_y_ptr,d_y_ptr+point_len,std::ostream_iterator<T>(std::cout, " "));std::cout<<std::endl;
}    

    auto d_pnt_iter=thrust::make_zip_iterator(thrust::make_tuple(d_pnt_x,d_pnt_y));       
    uint32_t *d_pnt_pntkey=NULL;
    RMM_TRY( RMM_ALLOC( (void**)&(d_pnt_pntkey),point_len* sizeof(uint32_t),stream));
    assert(d_pnt_pntkey!=NULL); 
        
    //computing Morton code (Z-order) 
    thrust::transform(exec_policy,d_pnt_iter,d_pnt_iter+point_len, d_pnt_pntkey,xytoz<T>(bbox,num_level,scale));   

if(0)
{
    thrust::device_ptr<uint32_t> d_pntkey_ptr=thrust::device_pointer_cast(d_pnt_pntkey);	
    thrust::copy(d_pntkey_ptr,d_pntkey_ptr+point_len,std::ostream_iterator<uint32_t>(std::cout, " "));std::cout<<std::endl;
}
    size_t free_mem = 0, total_mem = 0;
    hipMemGetInfo(&free_mem, &total_mem);
    std::cout<<"before sort: free memory:"<<free_mem<<"("<<total_mem<<")"<<std::endl;   
    //sorting points based on Morton code according to the first paragrah of Section 4.2 of ref. 
    thrust::sort_by_key(exec_policy,d_pnt_pntkey, d_pnt_pntkey+point_len,d_pnt_iter);
    hipMemGetInfo(&free_mem, &total_mem);
    std::cout<<"after sort: free memory:"<<free_mem<<"("<<total_mem<<")"<<std::endl;        

    uint32_t *d_pnt_runkey=NULL, *d_pnt_runlen=NULL;
    RMM_TRY( RMM_ALLOC( (void**)&(d_pnt_runkey),point_len* sizeof(uint32_t),stream));
    RMM_TRY( RMM_ALLOC( (void**)&(d_pnt_runlen),point_len* sizeof(uint32_t),stream));
    assert(d_pnt_runkey!=NULL && d_pnt_runlen!=NULL);

    size_t num_run = thrust::reduce_by_key(exec_policy,d_pnt_pntkey,d_pnt_pntkey+point_len,
    	thrust::constant_iterator<int>(1),d_pnt_runkey,d_pnt_runlen).first -d_pnt_runkey;
    RMM_FREE(d_pnt_pntkey,stream);d_pnt_pntkey=NULL;
    std::cout<<"num_run"<<num_run<<std::endl;

    //allocate sufficient GPU memory for "full quadrants" (Secection 4.1 of ref.)
    uint32_t *d_pnt_parentkey=NULL,*d_pnt_numchild=NULL,*d_pnt_pntlen=NULL;    
    RMM_TRY( RMM_ALLOC( (void**)&(d_pnt_parentkey),num_level*num_run* sizeof(uint32_t),stream));
    HANDLE_CUDA_ERROR( hipMemcpy( (void *)d_pnt_parentkey, (void *)d_pnt_runkey, num_run * sizeof(uint32_t), hipMemcpyDeviceToDevice ) );
    assert(d_pnt_parentkey!=NULL);
    RMM_FREE(d_pnt_runkey,stream);d_pnt_runkey=NULL;
    
    RMM_TRY( RMM_ALLOC( (void**)&(d_pnt_pntlen),num_level*num_run* sizeof(uint32_t),stream));    
    HANDLE_CUDA_ERROR( hipMemcpy( (void *)d_pnt_pntlen, (void *)d_pnt_runlen, num_run * sizeof(uint32_t), hipMemcpyDeviceToDevice ) );
    assert(d_pnt_pntlen!=NULL);
    RMM_FREE(d_pnt_runlen,stream);d_pnt_runlen=NULL;
     
    RMM_TRY( RMM_ALLOC( (void**)&(d_pnt_numchild),num_level*num_run* sizeof(uint32_t),stream));
    assert(d_pnt_numchild!=NULL);
    HANDLE_CUDA_ERROR( hipMemset(d_pnt_numchild,0,num_run*sizeof(uint32_t)) ); 
    
    //generating keys of paraent quadrants and numbers of child quadrants of "full quadrants" 
    //based on the second of paragraph of Section 4.2 of ref. 
    //keeping track of the number of quadrants, their begining/ending positions for each level 
    int lev_num[num_level],lev_bpos[num_level],lev_epos[num_level];
    lev_num[num_level-1]=num_run;
    uint32_t begin_pos=0, end_pos=num_run;
    for(int k=num_level-1;k>=0;k--)
    {
        uint32_t nk=thrust::reduce_by_key(exec_policy,
        thrust::make_transform_iterator(d_pnt_parentkey+begin_pos,get_parent(2)),
	    thrust::make_transform_iterator(d_pnt_parentkey+end_pos,get_parent(2)),
	    thrust::constant_iterator<int>(1),
	    d_pnt_parentkey+end_pos,d_pnt_numchild+end_pos).first-(d_pnt_parentkey+end_pos);
        uint32_t nn=thrust::reduce_by_key(exec_policy,
            thrust::make_transform_iterator(d_pnt_parentkey+begin_pos,get_parent(2)),
	    thrust::make_transform_iterator(d_pnt_parentkey+end_pos,get_parent(2)),
	    d_pnt_pntlen+begin_pos,
	    d_pnt_parentkey+end_pos,d_pnt_pntlen+end_pos).first-(d_pnt_parentkey+end_pos);
	assert(nk==nn);	
	std::cout<<"lev="<<k<<" begin_pos="<<begin_pos<<" end_pos="<<end_pos<<" nk="<<nk<<" nn="<<nn<<std::endl;
        lev_num[k]=nk; lev_bpos[k]=begin_pos; lev_epos[k]=end_pos; 	  	
        begin_pos=end_pos; end_pos+=nk; 
 }
            
/*
 *allocate three temporal arrays for parent key,number of children,
 *and the number of points in each quadrant, respectively
 *d_pnt_fullkey will be copied to the data array of the key column after revmoing invlaid quadtree ndoes
 *d_pnt_qtclen and d_pnt_qtnlen will be combined to generate the final length array
 *see fig.1 of ref. 
*/
    uint32_t *d_pnt_fullkey=NULL,*d_pnt_qtclen=NULL,*d_pnt_qtnlen=NULL;
    RMM_TRY( RMM_ALLOC( (void**)&(d_pnt_fullkey),end_pos* sizeof(uint32_t),stream));
    assert(d_pnt_fullkey!=NULL);
    RMM_TRY( RMM_ALLOC( (void**)&(d_pnt_qtclen),end_pos* sizeof(uint32_t),stream));
    assert(d_pnt_qtclen!=NULL);
    RMM_TRY( RMM_ALLOC( (void**)&(d_pnt_qtnlen),end_pos* sizeof(uint32_t),stream));
    assert(d_pnt_qtnlen!=NULL);
    uint8_t *d_pnt_fulllev=NULL;
    RMM_TRY( RMM_ALLOC( (void**)&(d_pnt_fulllev),end_pos* sizeof(uint8_t),0));
    assert(d_pnt_fulllev!=NULL);

    //reverse the order of quadtree nodes for easier manipulation; skip the root node 
    int num_count_nodes=0;
    for(uint32_t k=0;k<num_level;k++)
    {	
        thrust::fill(thrust::device,d_pnt_fulllev+num_count_nodes,d_pnt_fulllev+num_count_nodes+(lev_epos[k]-lev_bpos[k]),k);
        int nq1=thrust::copy(exec_policy,d_pnt_parentkey+lev_bpos[k],d_pnt_parentkey+lev_epos[k],d_pnt_fullkey+num_count_nodes)-(d_pnt_fullkey+num_count_nodes);   	
        int nq2=thrust::copy(exec_policy,d_pnt_numchild+lev_bpos[k],d_pnt_numchild+lev_epos[k],d_pnt_qtclen+num_count_nodes)-(d_pnt_qtclen+num_count_nodes); 
        int nq3=thrust::copy(exec_policy,d_pnt_pntlen+lev_bpos[k],d_pnt_pntlen+lev_epos[k],d_pnt_qtnlen+num_count_nodes)-(d_pnt_qtnlen+num_count_nodes);   	
        int nq4=thrust::reduce(exec_policy,d_pnt_pntlen+lev_bpos[k],d_pnt_pntlen+lev_epos[k]);
        assert(nq1==nq2 && nq2==nq3 && nq4==point_len);
        num_count_nodes+=nq1;
    }
    //Note: root node not counted
    assert(num_count_nodes==begin_pos);

/*
 *delete oversized nodes for memroy efficiency
 *num_count_nodes should be typically much smaller than num_level*num_run 
*/
    RMM_FREE(d_pnt_parentkey,stream);d_pnt_parentkey=NULL;
    RMM_FREE(d_pnt_numchild,stream);d_pnt_numchild=NULL;
    RMM_FREE(d_pnt_pntlen,stream);d_pnt_pntlen=NULL;

    int num_parent_nodes=0;
    for(uint32_t k=1;k<num_level;k++) num_parent_nodes+=lev_num[k];

    //temporal device memory for vector expansion
    uint32_t *d_pnt_tmppos=NULL;
    RMM_TRY( RMM_ALLOC( (void**)&(d_pnt_tmppos),num_parent_nodes* sizeof(uint32_t),stream));
    assert(d_pnt_tmppos!=NULL);

    //line 1 of algorithm in Fig. 5 in ref. 
    thrust::exclusive_scan(exec_policy,d_pnt_qtclen,d_pnt_qtclen+num_parent_nodes,d_pnt_tmppos);
    size_t num_child_nodes=thrust::reduce(exec_policy,d_pnt_qtclen,d_pnt_qtclen+num_parent_nodes);
    std::cout<<"num_child_nodes="<<num_child_nodes<<std::endl;
    uint32_t *d_pnt_parentpos=NULL;
    RMM_TRY( RMM_ALLOC( (void**)&(d_pnt_parentpos),num_child_nodes* sizeof(uint32_t),stream));
    assert(d_pnt_parentpos!=NULL); 
    HANDLE_CUDA_ERROR( hipMemset(d_pnt_parentpos,0,num_child_nodes*sizeof(uint32_t)) );
    
    //line 2 of algorithm in Fig. 5 in ref. 
    thrust::scatter(exec_policy,thrust::make_counting_iterator(0),
        thrust::make_counting_iterator(0)+num_parent_nodes,d_pnt_tmppos,d_pnt_parentpos);
    RMM_FREE(d_pnt_tmppos,stream);d_pnt_tmppos=NULL;

    //line 3 of algorithm in Fig. 5 in ref. 
    thrust::inclusive_scan(exec_policy,d_pnt_parentpos,d_pnt_parentpos+num_child_nodes,d_pnt_parentpos,thrust::maximum<int>()); 
/*
 *counting the number of nodes whose children have numbers of points no less than min_size;
 *note that we start at level 2 as level nodes (whose parents are the root node -level 0) need to be kept
*/
    auto iter_in=thrust::make_zip_iterator(thrust::make_tuple(d_pnt_fullkey+lev_num[1],d_pnt_fulllev+lev_num[1],
        d_pnt_qtclen+lev_num[1],d_pnt_qtnlen+lev_num[1],d_pnt_parentpos));
    int num_invalid_parent_nodes = thrust::count_if(exec_policy,iter_in,iter_in+(num_parent_nodes-lev_num[1]),
        remove_discard(d_pnt_qtnlen,min_size));
    assert(num_invalid_parent_nodes<=num_parent_nodes);
    num_parent_nodes-=num_invalid_parent_nodes;
 
    uint32_t *d_pnt_templen=NULL;
    RMM_TRY( RMM_ALLOC( (void**)&(d_pnt_templen),end_pos* sizeof(uint32_t),stream));
    assert(d_pnt_templen!=NULL);
   
    //line 4 of algorithm in Fig. 5 in ref. 
    HANDLE_CUDA_ERROR( hipMemcpy( (void *)d_pnt_templen, (void *)d_pnt_qtnlen, end_pos * sizeof(uint32_t), hipMemcpyDeviceToDevice ) );

    //line 5 of algorithm in Fig. 5 in ref. 
    int num_valid_nodes = thrust::remove_if(exec_policy,iter_in,iter_in+num_child_nodes,remove_discard(d_pnt_templen,min_size))-iter_in;
    RMM_FREE(d_pnt_templen,stream);d_pnt_templen=NULL;
    RMM_FREE(d_pnt_parentpos,stream);d_pnt_parentpos=NULL;
   
    //add back level 1 nodes
    num_valid_nodes+=lev_num[1];
    std::cout<<"num_invalid_parent_nodes="<<num_invalid_parent_nodes<<std::endl;
    std::cout<<"num_valid_nodes="<<num_valid_nodes<<std::endl;
     
/*
 *preparing the key column for output 
 *Note: only the first num_valid_nodes elements should in the output array
*/
    std::unique_ptr<cudf::column> key_col = cudf::make_numeric_column(
       cudf::data_type(cudf::type_id::INT32), num_valid_nodes,cudf::mask_state::UNALLOCATED,  stream, mr);      
    uint32_t *d_pnt_qtkey=cudf::mutable_column_device_view::create(key_col->mutable_view(), stream)->data<uint32_t>();
    assert(d_pnt_qtkey!=NULL);
  
    thrust::copy(exec_policy,d_pnt_fullkey,d_pnt_fullkey+num_valid_nodes,d_pnt_qtkey);
    RMM_FREE(d_pnt_fullkey,stream);d_pnt_fullkey=NULL;

    std::unique_ptr<cudf::column> lev_col = cudf::make_numeric_column(
       cudf::data_type(cudf::type_id::INT8), num_valid_nodes,cudf::mask_state::UNALLOCATED,  stream, mr);      
    uint8_t *d_pnt_qtlev=cudf::mutable_column_device_view::create(lev_col->mutable_view(), stream)->data<uint8_t>();
    assert(d_pnt_qtlev!=NULL);
  
    thrust::copy(exec_policy,d_pnt_fulllev,d_pnt_fulllev+num_valid_nodes,d_pnt_qtlev);
    RMM_FREE(d_pnt_fulllev,stream);d_pnt_fulllev=NULL;   
   
    //preparing the indicator array for output
    std::unique_ptr<cudf::column> sign_col = cudf::make_numeric_column(
           cudf::data_type(cudf::type_id::BOOL8), num_valid_nodes,cudf::mask_state::UNALLOCATED,  stream, mr);
    bool *d_pnt_qtsign=cudf::mutable_column_device_view::create(sign_col->mutable_view(), stream)->data<bool>();
    assert(d_pnt_qtsign!=NULL);
    HANDLE_CUDA_ERROR( hipMemset(d_pnt_qtsign,0,num_valid_nodes*sizeof(bool)) );

    //line 6 of algorithm in Fig. 5 in ref. 
    thrust::transform(exec_policy,d_pnt_qtnlen,d_pnt_qtnlen+num_parent_nodes,d_pnt_qtsign,thrust::placeholders::_1 > min_size);

    //line 7 of algorithm in Fig. 5 in ref. 
    thrust::replace_if(exec_policy,d_pnt_qtnlen,d_pnt_qtnlen+num_parent_nodes,d_pnt_qtsign,thrust::placeholders::_1,0);
 
    std::cout<<"total point"<<thrust::reduce(exec_policy,d_pnt_qtnlen,d_pnt_qtnlen+num_valid_nodes)<<std::endl;
    std::cout<<"non-last-level points="<<thrust::reduce(exec_policy,d_pnt_qtnlen,d_pnt_qtnlen+num_parent_nodes)<<std::endl;

    //allocating two temporal array:the first child position array and first point position array,respectively
    //later they will be used to generate the final position array 
    uint32_t *d_pnt_qtnpos=NULL,*d_pnt_qtcpos=NULL;
    RMM_TRY( RMM_ALLOC( (void**)&(d_pnt_qtnpos),num_valid_nodes* sizeof(uint32_t),stream));
    RMM_TRY( RMM_ALLOC(  (void**)&(d_pnt_qtcpos),num_valid_nodes* sizeof(uint32_t),stream));
    assert(d_pnt_qtnpos!=NULL && d_pnt_qtcpos!=NULL);
   
/*
 *revision to line 8 of algorithm in Fig. 5 in ref. 
 *ajust nlen and npos based on last-level z-order code
*/
    uint32_t *d_pnt_tmp_key=NULL;
    RMM_TRY( RMM_ALLOC( (void**)&(d_pnt_tmp_key),num_valid_nodes* sizeof(uint32_t),stream));
    assert(d_pnt_tmp_key!=NULL);
    HANDLE_CUDA_ERROR( hipMemcpy( (void *)d_pnt_tmp_key, (void *)d_pnt_qtkey, 
        num_valid_nodes * sizeof(uint32_t), hipMemcpyDeviceToDevice ) );
    uint32_t *d_pnt_tmp_pos=NULL;
    RMM_TRY( RMM_ALLOC( (void**)&(d_pnt_tmp_pos),num_valid_nodes* sizeof(uint32_t),stream));
    assert(d_pnt_tmp_pos!=NULL);

    auto key_lev_iter=thrust::make_zip_iterator(thrust::make_tuple(d_pnt_qtkey,d_pnt_qtlev,d_pnt_qtsign));
    thrust::transform(exec_policy,key_lev_iter,key_lev_iter+num_valid_nodes,d_pnt_tmp_key,flatten_z_code(num_level));
    uint32_t num_leaf_nodes=thrust::copy_if(exec_policy,thrust::make_counting_iterator(0),
   	thrust::make_counting_iterator(0)+num_valid_nodes,d_pnt_qtsign,d_pnt_tmp_pos,!thrust::placeholders::_1)-d_pnt_tmp_pos;

    uint32_t *d_pnt_tmp_seq=NULL;
    RMM_TRY( RMM_ALLOC( (void**)&(d_pnt_tmp_seq),num_valid_nodes* sizeof(uint32_t),stream));
    assert(d_pnt_tmp_seq!=NULL);

    uint32_t *d_pnt_tmp_neln=NULL;
    RMM_TRY( RMM_ALLOC( (void**)&(d_pnt_tmp_neln),num_valid_nodes* sizeof(uint32_t),stream));
    assert(d_pnt_tmp_neln!=NULL);

    uint32_t *d_pnt_tmp_npos=NULL;
    RMM_TRY( RMM_ALLOC( (void**)&(d_pnt_tmp_npos),num_valid_nodes* sizeof(uint32_t),stream));
    assert(d_pnt_tmp_npos!=NULL);

    thrust::sequence(exec_policy,d_pnt_tmp_seq,d_pnt_tmp_seq+num_valid_nodes);
    thrust::copy(exec_policy,d_pnt_qtnlen,d_pnt_qtnlen+num_valid_nodes,d_pnt_tmp_neln);   
    auto seq_len_pos=thrust::make_zip_iterator(thrust::make_tuple(d_pnt_tmp_seq,d_pnt_tmp_neln));
    thrust::stable_sort_by_key(exec_policy,d_pnt_tmp_key,d_pnt_tmp_key+num_valid_nodes,seq_len_pos);

if(0)
{
   printf("d_pnt_tmp_key:after sort\n");
   thrust::device_ptr<uint> d_tmpkey_ptr=thrust::device_pointer_cast(d_pnt_tmp_key);
   thrust::copy(d_tmpkey_ptr,d_tmpkey_ptr+num_valid_nodes,std::ostream_iterator<uint>(std::cout, " "));std::cout<<std::endl;
   
   printf("d_pnt_tmp_seq:after sort\n");
   thrust::device_ptr<uint> d_tmpseq_ptr=thrust::device_pointer_cast(d_pnt_tmp_seq);
   thrust::copy(d_tmpseq_ptr,d_tmpseq_ptr+num_valid_nodes,std::ostream_iterator<uint>(std::cout, " "));std::cout<<std::endl; 
   
   printf("d_pnt_tmp_neln:after sort\n");
   thrust::device_ptr<uint> d_tmplen_ptr=thrust::device_pointer_cast(d_pnt_tmp_neln);
   thrust::copy(d_tmplen_ptr,d_tmplen_ptr+num_valid_nodes,std::ostream_iterator<uint>(std::cout, " "));std::cout<<std::endl;  
}   

    thrust::remove_if(exec_policy,d_pnt_tmp_neln,d_pnt_tmp_neln+num_valid_nodes,d_pnt_tmp_neln,thrust::placeholders::_1==0);
    //only the first num_leaf_nodes are needed after the above removal (copy_if and remove_if should return the same numbers
    thrust::exclusive_scan(exec_policy,d_pnt_tmp_neln,d_pnt_tmp_neln+num_leaf_nodes,d_pnt_tmp_npos);
    auto len_pos_iter=thrust::make_zip_iterator(thrust::make_tuple(d_pnt_tmp_neln,d_pnt_tmp_npos));
    thrust::stable_sort_by_key(thrust::device,d_pnt_tmp_seq,d_pnt_tmp_seq+num_leaf_nodes,len_pos_iter);
    
    RMM_TRY(RMM_FREE(d_pnt_tmp_seq,stream));d_pnt_tmp_seq=NULL; 
    HANDLE_CUDA_ERROR( hipMemset(d_pnt_qtnlen,0,num_valid_nodes*sizeof(uint32_t)) );
    HANDLE_CUDA_ERROR( hipMemset(d_pnt_qtnpos,0,num_valid_nodes*sizeof(uint32_t)) );
   
    auto in_len_pos_iter=thrust::make_zip_iterator(thrust::make_tuple(d_pnt_tmp_neln,d_pnt_tmp_npos));
    auto out_len_pos_iter=thrust::make_zip_iterator(thrust::make_tuple(d_pnt_qtnlen,d_pnt_qtnpos));
    thrust::scatter(thrust::device,in_len_pos_iter,in_len_pos_iter+num_leaf_nodes,d_pnt_tmp_pos,out_len_pos_iter);
    
    RMM_TRY(RMM_FREE(d_pnt_tmp_pos,stream));d_pnt_tmp_pos=NULL;
    RMM_TRY(RMM_FREE(d_pnt_tmp_neln,stream));d_pnt_tmp_neln=NULL;
    RMM_TRY(RMM_FREE(d_pnt_tmp_npos,stream));d_pnt_tmp_npos=NULL;
  
  
    //line 9 of algorithm in Fig. 5 in ref. 
    thrust::replace_if(exec_policy,d_pnt_qtclen,d_pnt_qtclen+num_valid_nodes,d_pnt_qtsign,!thrust::placeholders::_1,0);
   
    //line 10 of algorithm in Fig. 5 in ref. 
    thrust::exclusive_scan(exec_policy,d_pnt_qtclen,d_pnt_qtclen+num_valid_nodes,d_pnt_qtcpos,lev_num[1]);   

if(0)
{
   std::cout<<"length0:"<<std::endl;
   thrust::device_ptr<uint32_t> d_qtclen_ptr=thrust::device_pointer_cast(d_pnt_qtclen);
   thrust::device_ptr<uint32_t> d_qtnlen_ptr=thrust::device_pointer_cast(d_pnt_qtnlen);
   thrust::copy(d_qtclen_ptr,d_qtclen_ptr+num_valid_nodes,std::ostream_iterator<uint32_t>(std::cout, " "));std::cout<<std::endl;
   thrust::copy(d_qtnlen_ptr,d_qtnlen_ptr+num_valid_nodes,std::ostream_iterator<uint32_t>(std::cout, " "));std::cout<<std::endl;

   std::cout<<"pos0:"<<std::endl;
   thrust::device_ptr<uint32_t> d_qtcpos_ptr=thrust::device_pointer_cast(d_pnt_qtcpos);
   thrust::device_ptr<uint32_t> d_qtnpos_ptr=thrust::device_pointer_cast(d_pnt_qtnpos);
   thrust::copy(d_qtcpos_ptr,d_qtcpos_ptr+num_valid_nodes,std::ostream_iterator<uint32_t>(std::cout, " "));std::cout<<std::endl;
   thrust::copy(d_qtnpos_ptr,d_qtnpos_ptr+num_valid_nodes,std::ostream_iterator<uint32_t>(std::cout, " "));std::cout<<std::endl;
}
 
   //preparing the length and fpos array for output 
   std::unique_ptr<cudf::column> length_col = cudf::make_numeric_column(
   cudf::data_type(cudf::type_id::INT32), num_valid_nodes,cudf::mask_state::UNALLOCATED,  stream, mr);
   uint32_t *d_pnt_qtlength=cudf::mutable_column_device_view::create(length_col->mutable_view(), stream)->data<uint32_t>();
   assert(d_pnt_qtlength!=NULL);

   std::unique_ptr<cudf::column> fpos_col = cudf::make_numeric_column(
   cudf::data_type(cudf::type_id::INT32), num_valid_nodes,cudf::mask_state::UNALLOCATED,  stream, mr);
   uint32_t *d_pnt_qtfpos=cudf::mutable_column_device_view::create(fpos_col->mutable_view(), stream)->data<uint32_t>();
   assert(d_pnt_qtfpos!=NULL);

   //line 11 of algorithm in Fig. 5 in ref. 
   auto iter_len_in=thrust::make_zip_iterator(thrust::make_tuple(d_pnt_qtclen,d_pnt_qtnlen,d_pnt_qtsign));
   auto iter_pos_in=thrust::make_zip_iterator(thrust::make_tuple(d_pnt_qtcpos,d_pnt_qtnpos,d_pnt_qtsign));
   thrust::transform(exec_policy,iter_len_in,iter_len_in+num_valid_nodes,d_pnt_qtlength,what2output());
   thrust::transform(exec_policy,iter_pos_in,iter_pos_in+num_valid_nodes,d_pnt_qtfpos,what2output());
   
   RMM_FREE(d_pnt_qtnpos,stream);d_pnt_qtnpos=NULL;
   RMM_FREE(d_pnt_qtcpos,stream);d_pnt_qtcpos=NULL;
   RMM_FREE(d_pnt_qtnlen,stream);d_pnt_qtnlen=NULL;
   RMM_FREE(d_pnt_qtclen,stream);d_pnt_qtclen=NULL;

if(0)
{

    thrust::device_ptr<uint32_t> d_key_ptr=thrust::device_pointer_cast(d_pnt_qtkey);
    thrust::device_ptr<uint8_t> d_lev_ptr=thrust::device_pointer_cast(d_pnt_qtlev);   
    thrust::device_ptr<bool> d_sign_ptr=thrust::device_pointer_cast(d_pnt_qtsign);   
    thrust::device_ptr<uint32_t> d_len_ptr=thrust::device_pointer_cast(d_pnt_qtlength);	
    thrust::device_ptr<uint32_t> d_fpos_ptr=thrust::device_pointer_cast(d_pnt_qtfpos);   
 
    printf("key\n");
    thrust::copy(d_key_ptr,d_key_ptr+num_valid_nodes,std::ostream_iterator<uint32_t>(std::cout, " "));std::cout<<std::endl;
    
    printf("lev\n");
    //change from uint8_t to uint32_t in ostream_iterator to output numbers instead of special chars
    thrust::copy(d_lev_ptr,d_lev_ptr+num_valid_nodes,std::ostream_iterator<uint32_t>(std::cout, " "));std::cout<<std::endl;
   
    printf("sign\n");
    thrust::copy(d_sign_ptr,d_sign_ptr+num_valid_nodes,std::ostream_iterator<bool>(std::cout, " "));std::cout<<std::endl;
    
    printf("length\n");
    thrust::copy(d_len_ptr,d_len_ptr+num_valid_nodes,std::ostream_iterator<uint32_t>(std::cout, " "));std::cout<<std::endl;
    
    printf("fpos\n");
    thrust::copy(d_fpos_ptr,d_fpos_ptr+num_valid_nodes,std::ostream_iterator<uint32_t>(std::cout, " "));std::cout<<std::endl;
}   

   std::vector<std::unique_ptr<cudf::column>> quad_cols;
   quad_cols.push_back(std::move(key_col));
   quad_cols.push_back(std::move(lev_col));
   quad_cols.push_back(std::move(sign_col));
   quad_cols.push_back(std::move(length_col));
   quad_cols.push_back(std::move(fpos_col));
   return quad_cols;
}

struct quadtree_point_processor {

    template<typename T, std::enable_if_t<std::is_floating_point<T>::value >* = nullptr>
    std::unique_ptr<cudf::experimental::table> operator()(
        cudf::mutable_column_view& x,
        cudf::mutable_column_view& y,
        quad_point_parameters qpi,
        rmm::mr::device_memory_resource* mr,
        hipStream_t stream)
    {
        T *d_pnt_x=cudf::mutable_column_device_view::create(x, stream)->data<T>();
        T *d_pnt_y=cudf::mutable_column_device_view::create(y, stream)->data<T>();
        double x1=thrust::get<0>(qpi);
        double y1=thrust::get<1>(qpi);
        double x2=thrust::get<2>(qpi);
        double y2=thrust::get<3>(qpi);
        SBBox<double> bbox(thrust::make_tuple(x1,y1),thrust::make_tuple(x2,y2));
        double scale=thrust::get<4>(qpi);
        uint32_t num_level=thrust::get<5>(qpi);
        uint32_t min_size=thrust::get<6>(qpi);
       
        std::vector<std::unique_ptr<cudf::column>> quad_cols=
            dowork<T>(x.size(),d_pnt_x,d_pnt_y,bbox,scale, num_level,min_size,mr,stream);

        std::unique_ptr<cudf::experimental::table> destination_table= 
            std::make_unique<cudf::experimental::table>(std::move(quad_cols));
        
        return destination_table;
    }

    template<typename T, std::enable_if_t<!std::is_floating_point<T>::value >* = nullptr>
    std::unique_ptr<cudf::experimental::table> operator()(
        cudf::mutable_column_view& x,
        cudf::mutable_column_view& y,
        quad_point_parameters qpi,
        rmm::mr::device_memory_resource* mr,
        hipStream_t stream)
        {
             CUDF_FAIL("Non-floating point operation is not supported");
        }

    };

} //end anonymous namespace

namespace cuspatial
{

std::unique_ptr<cudf::column> nested_column_test(cudf::column_view x,cudf::column_view y)
{
    std::vector<std::unique_ptr<cudf::column>> children;

    std::unique_ptr<cudf::column> key_col=cudf::make_numeric_column(cudf::data_type{cudf::INT32}, 1);
    children.push_back(std::move(key_col));

    std::unique_ptr<cudf::column> indicator_col=cudf::make_numeric_column(cudf::data_type{cudf::BOOL8}, 1);
    children.push_back(std::move(indicator_col));

    std::unique_ptr<cudf::column> fpos_col=cudf::make_numeric_column(cudf::data_type{cudf::INT32}, 1);
    children.push_back(std::move(fpos_col));

    std::unique_ptr<cudf::column> len_col=cudf::make_numeric_column(cudf::data_type{cudf::INT32}, 1);
    children.push_back(std::move(len_col));

    
    cudf::data_type type=cudf::data_type{cudf::INT32};
    cudf::size_type size=1;
    cudf::mask_state state=cudf::mask_state::ALL_NULL;
    hipStream_t stream=0;
    rmm::mr::device_memory_resource* mr=rmm::mr::get_default_resource();
    rmm::device_buffer  buffer{size * cudf::size_of(type), stream, mr};
    rmm::device_buffer nmask=create_null_mask(size, state, stream, mr);
    cudf::size_type ncount=state_null_count(state, size);

    std::unique_ptr<cudf::column> ret=std::make_unique<cudf::column>(type,size,buffer,nmask,ncount,std::move(children));
    
    return ret;
}

std::unique_ptr<cudf::experimental::table> quadtree_on_points(
  cudf::mutable_column_view& x,cudf::mutable_column_view& y,
	double x1,double y1,double x2,double y2, double scale, int num_level, int min_size)
{
    hipStream_t stream=0;
    rmm::mr::device_memory_resource* mr=rmm::mr::get_default_resource();

    quad_point_parameters qpi=thrust::make_tuple(x1,y1,x2,y2,scale,num_level,min_size);
    return cudf::experimental::type_dispatcher(x.type(),quadtree_point_processor{},x,y, qpi, mr,stream);
}

}// namespace cuspatial
